#include "hip/hip_runtime.h"
#include "convolutionSeparable31.hpp"
#include "defines.hpp"

#define ROW_BLOCK_DIM_X31 32
#define ROW_BLOCK_DIM_Y31 16

__constant__ float c_Kernel31[256];

void setConvolutionKernel31(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel31), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter31(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y31][(ROW_RESULT_STEP + 2*ROW_HALO_STEP) * ROW_BLOCK_DIM_X31];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X31 - ROW_HALO_STEP * ROW_BLOCK_DIM_X31 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y31 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X31] = (baseX + i * ROW_BLOCK_DIM_X31) < imageW ? d_Src[i*ROW_BLOCK_DIM_X31] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X31] = (baseX + i * ROW_BLOCK_DIM_X31) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X31] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP + ROW_RESULT_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP + ROW_HALO_STEP; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X31] = (baseX + i * ROW_BLOCK_DIM_X31) < imageW ? d_Src[i * ROW_BLOCK_DIM_X31] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X31 < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS31; j <= KERNEL_RADIUS31; j++) {

                sum += c_Kernel31[KERNEL_RADIUS31 - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X31 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X31] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X31 16
#define COL_BLOCK_DIM_Y31 32

__global__ void colConvolutionFilter31(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X31][(COL_RESULT_STEP + 2 * COL_HALO_STEP) * COL_BLOCK_DIM_Y31 + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X31 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y31 * COL_RESULT_STEP - COL_HALO_STEP * COL_BLOCK_DIM_Y31 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y31] = (baseY + i * COL_BLOCK_DIM_Y31) < imageH ? d_Src[i * COL_BLOCK_DIM_Y31 * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y31] = (baseY + i * COL_BLOCK_DIM_Y31) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y31 * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP + COL_RESULT_STEP; i < COL_HALO_STEP + COL_RESULT_STEP + COL_HALO_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y31] = (baseY + i * COL_BLOCK_DIM_Y31) < imageH ? d_Src[i * COL_BLOCK_DIM_Y31 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y31) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS31; j <= KERNEL_RADIUS31; j++) {
                
                sum += c_Kernel31[KERNEL_RADIUS31 - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y31 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y31 * pitch] = sum;
        }
    }
}

void rowConvolve31(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X31) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X31), (imageH + ROW_BLOCK_DIM_Y31 - 1)  / ROW_BLOCK_DIM_Y31);
        dim3 threads(ROW_BLOCK_DIM_X31, ROW_BLOCK_DIM_Y31);

        rowConvolutionFilter31<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve31(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X31 - 1) / COL_BLOCK_DIM_X31, (imageH + COL_BLOCK_DIM_Y31 * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y31 * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X31, COL_BLOCK_DIM_Y31);
        
        colConvolutionFilter31<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }