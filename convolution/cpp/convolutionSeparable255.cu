#include "hip/hip_runtime.h"
#include "convolutionSeparable255.hpp"
#include "defines.hpp"

#define ROW_BLOCK_DIM_X255 128
#define ROW_BLOCK_DIM_Y255 4

__constant__ float c_Kernel255[256];

void setConvolutionKernel255(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel255), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter255(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y255][(ROW_RESULT_STEP + 2*ROW_HALO_STEP) * ROW_BLOCK_DIM_X255];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X255 - ROW_HALO_STEP * ROW_BLOCK_DIM_X255 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y255 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) < imageW ? d_Src[i*ROW_BLOCK_DIM_X255] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X255] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP + ROW_RESULT_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP + ROW_HALO_STEP; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) < imageW ? d_Src[i * ROW_BLOCK_DIM_X255] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X255 < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS255; j <= KERNEL_RADIUS255; j++) {

                sum += c_Kernel255[KERNEL_RADIUS255 - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X255 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X255] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X255 4
#define COL_BLOCK_DIM_Y255 128

__global__ void colConvolutionFilter255(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X255][(COL_RESULT_STEP + 2 * COL_HALO_STEP) * COL_BLOCK_DIM_Y255 + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X255 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y255 * COL_RESULT_STEP - COL_HALO_STEP * COL_BLOCK_DIM_Y255 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) < imageH ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP + COL_RESULT_STEP; i < COL_HALO_STEP + COL_RESULT_STEP + COL_HALO_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) < imageH ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y255) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS255; j <= KERNEL_RADIUS255; j++) {
                
                sum += c_Kernel255[KERNEL_RADIUS255 - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y255 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y255 * pitch] = sum;
        }
    }
}

void rowConvolve255(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X255) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X255), (imageH + ROW_BLOCK_DIM_Y255 - 1)  / ROW_BLOCK_DIM_Y255);
        dim3 threads(ROW_BLOCK_DIM_X255, ROW_BLOCK_DIM_Y255);

        rowConvolutionFilter255<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve255(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X255 - 1) / COL_BLOCK_DIM_X255, (imageH + COL_BLOCK_DIM_Y255 * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y255 * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X255, COL_BLOCK_DIM_Y255);
        
        colConvolutionFilter255<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }