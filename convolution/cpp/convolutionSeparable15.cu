#include "hip/hip_runtime.h"
#include "convolutionSeparable15.hpp"
#include "defines.hpp"

#define ROW_BLOCK_DIM_X15 32
#define ROW_BLOCK_DIM_Y15 16

__constant__ float c_Kernel15[256];

void setConvolutionKernel15(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel15), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter15(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y15][(ROW_RESULT_STEP + 2*ROW_HALO_STEP15) * ROW_BLOCK_DIM_X15];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X15 - ROW_HALO_STEP15 * ROW_BLOCK_DIM_X15 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y15 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP15; i < ROW_HALO_STEP15 + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X15] = (baseX + i * ROW_BLOCK_DIM_X15) < imageW ? d_Src[i*ROW_BLOCK_DIM_X15] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP15; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X15] = (baseX + i * ROW_BLOCK_DIM_X15) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X15] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP15 + ROW_RESULT_STEP; i < ROW_HALO_STEP15 + ROW_RESULT_STEP + ROW_HALO_STEP15; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X15] = (baseX + i * ROW_BLOCK_DIM_X15) < imageW ? d_Src[i * ROW_BLOCK_DIM_X15] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP15; i < ROW_HALO_STEP15+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X15 < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS15; j <= KERNEL_RADIUS15; j++) {

                sum += c_Kernel15[KERNEL_RADIUS15 - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X15 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X15] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X15 16
#define COL_BLOCK_DIM_Y15 32

__global__ void colConvolutionFilter15(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X15][(COL_RESULT_STEP + 2 * COL_HALO_STEP15) * COL_BLOCK_DIM_Y15 + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X15 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y15 * COL_RESULT_STEP - COL_HALO_STEP15 * COL_BLOCK_DIM_Y15 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP15; i < COL_HALO_STEP15 + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y15] = (baseY + i * COL_BLOCK_DIM_Y15) < imageH ? d_Src[i * COL_BLOCK_DIM_Y15 * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP15; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y15] = (baseY + i * COL_BLOCK_DIM_Y15) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y15 * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP15 + COL_RESULT_STEP; i < COL_HALO_STEP15 + COL_RESULT_STEP + COL_HALO_STEP15; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y15] = (baseY + i * COL_BLOCK_DIM_Y15) < imageH ? d_Src[i * COL_BLOCK_DIM_Y15 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP15; i < COL_HALO_STEP15 + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y15) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS15; j <= KERNEL_RADIUS15; j++) {
                
                sum += c_Kernel15[KERNEL_RADIUS15 - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y15 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y15 * pitch] = sum;
        }
    }
}

void rowConvolve15(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X15) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X15), (imageH + ROW_BLOCK_DIM_Y15 - 1)  / ROW_BLOCK_DIM_Y15);
        dim3 threads(ROW_BLOCK_DIM_X15, ROW_BLOCK_DIM_Y15);

        rowConvolutionFilter15<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve15(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X15 - 1) / COL_BLOCK_DIM_X15, (imageH + COL_BLOCK_DIM_Y15 * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y15 * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X15, COL_BLOCK_DIM_Y15);
        
        colConvolutionFilter15<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }