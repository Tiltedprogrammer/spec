#include "hip/hip_runtime.h"
#include "convolutionSeparable127.hpp"
#include "defines.hpp"

#define ROW_BLOCK_DIM_X127 64
#define ROW_BLOCK_DIM_Y127 8

__constant__ float c_Kernel127[256];

void setConvolutionKernel127(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel127), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter127(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y127][(ROW_RESULT_STEP + 2*ROW_HALO_STEP) * ROW_BLOCK_DIM_X127];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X127 - ROW_HALO_STEP * ROW_BLOCK_DIM_X127 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y127 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X127] = (baseX + i * ROW_BLOCK_DIM_X127) < imageW ? d_Src[i*ROW_BLOCK_DIM_X127] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X127] = (baseX + i * ROW_BLOCK_DIM_X127) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X127] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP + ROW_RESULT_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP + ROW_HALO_STEP; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X127] = (baseX + i * ROW_BLOCK_DIM_X127) < imageW ? d_Src[i * ROW_BLOCK_DIM_X127] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X127 < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS127; j <= KERNEL_RADIUS127; j++) {

                sum += c_Kernel127[KERNEL_RADIUS127 - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X127 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X127] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X127 8
#define COL_BLOCK_DIM_Y127 64

__global__ void colConvolutionFilter127(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X127][(COL_RESULT_STEP + 2 * COL_HALO_STEP) * COL_BLOCK_DIM_Y127 + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X127 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y127 * COL_RESULT_STEP - COL_HALO_STEP * COL_BLOCK_DIM_Y127 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y127] = (baseY + i * COL_BLOCK_DIM_Y127) < imageH ? d_Src[i * COL_BLOCK_DIM_Y127 * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y127] = (baseY + i * COL_BLOCK_DIM_Y127) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y127 * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP + COL_RESULT_STEP; i < COL_HALO_STEP + COL_RESULT_STEP + COL_HALO_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y127] = (baseY + i * COL_BLOCK_DIM_Y127) < imageH ? d_Src[i * COL_BLOCK_DIM_Y127 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y127) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS127; j <= KERNEL_RADIUS127; j++) {
                
                sum += c_Kernel127[KERNEL_RADIUS127 - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y127 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y127 * pitch] = sum;
        }
    }
}

void rowConvolve127(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X127) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X127), (imageH + ROW_BLOCK_DIM_Y127 - 1)  / ROW_BLOCK_DIM_Y127);
        dim3 threads(ROW_BLOCK_DIM_X127, ROW_BLOCK_DIM_Y127);

        rowConvolutionFilter127<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve127(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X127 - 1) / COL_BLOCK_DIM_X127, (imageH + COL_BLOCK_DIM_Y127 * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y127 * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X127, COL_BLOCK_DIM_Y127);
        
        colConvolutionFilter127<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }