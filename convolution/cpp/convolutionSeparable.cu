#include "hip/hip_runtime.h"
#include "convolutionSeparable.hpp"

#define ROW_BLOCK_DIM_X 32
#define ROW_BLOCK_DIM_Y 32

//how many pixels an individual thread would proccess
#define ROW_RESULT_STEP 8
//borders length of size @ROW_BLOCK_DIM to satisfy correct alignment
#define ROW_HALO_STEP 1

__constant__ float c_Kernel[KERNEL_LENGTH];

void setConvolutionKernel(float* h_Kernel)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}

__global__ void rowConvolutionFilter(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y][(ROW_RESULT_STEP + 2*ROW_HALO_STEP) * ROW_BLOCK_DIM_X];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X - ROW_HALO_STEP * ROW_BLOCK_DIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] = (baseX + i * ROW_BLOCK_DIM_X) < imageW ? d_Src[i*ROW_BLOCK_DIM_X] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] = (baseX + i*ROW_BLOCK_DIM_X) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP + ROW_RESULT_STEP; i < ROW_HALO_STEP + ROW_RESULT_STEP + ROW_HALO_STEP; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X] = (baseX + i*ROW_BLOCK_DIM_X) < imageW ? d_Src[i * ROW_BLOCK_DIM_X] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP; i < ROW_HALO_STEP+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j ++) {

                sum += c_Kernel[KERNEL_RADIUS - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X 32
#define COL_BLOCK_DIM_Y 32
#define COL_RESULT_STEP 8
#define COL_HALO_STEP 1

__global__ void colConvolutionFilter(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X][(COL_RESULT_STEP + 2 * COL_HALO_STEP) * COL_BLOCK_DIM_Y + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y * COL_RESULT_STEP - COL_HALO_STEP * COL_BLOCK_DIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y] = (baseY + i * COL_BLOCK_DIM_Y) < imageH ? d_Src[i * COL_BLOCK_DIM_Y * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y] = (baseY + i * COL_BLOCK_DIM_Y) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP + COL_RESULT_STEP; i < COL_HALO_STEP + COL_RESULT_STEP + COL_HALO_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y] = (baseY + i * COL_BLOCK_DIM_Y) < imageH ? d_Src[i * COL_BLOCK_DIM_Y * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP; i < COL_HALO_STEP + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) {
                
                sum += c_Kernel[KERNEL_RADIUS - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y * pitch] = sum;
        }
    }
}

void rowConvolve(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X), (imageH + ROW_BLOCK_DIM_Y - 1)  / ROW_BLOCK_DIM_Y);
        dim3 threads(ROW_BLOCK_DIM_X, ROW_BLOCK_DIM_Y);

        rowConvolutionFilter<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X - 1) / ROW_BLOCK_DIM_X, (imageH + COL_BLOCK_DIM_Y * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X, COL_BLOCK_DIM_Y);
        
        colConvolutionFilter<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }