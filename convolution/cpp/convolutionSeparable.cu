#include "hip/hip_runtime.h"
#include "convolutionSeparable.hpp"
#include "defines.hpp"


__constant__ float c_Kernel[256];

void setConvolutionKernel(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int radius,
    int blockX,
    int blockY,
    int step,
    int halo
){
    int ROW_BLOCK_DIM_X255 = blockX;
    int ROW_BLOCK_DIM_Y255 = blockY;
    extern __shared__ float sData[];
    int sDataWidth = blockX * (step + 2 * halo);

    //offset to left halo edge
    const int baseX = (blockIdx.x * step) * ROW_BLOCK_DIM_X255 - halo * ROW_BLOCK_DIM_X255 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y255 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
    for (int i = halo; i < halo + step; i++) {
     
        sData[threadIdx.y * sDataWidth + threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) < imageW ? d_Src[i*ROW_BLOCK_DIM_X255] : 0;
    
    }

    //load left halo
    
    for (int i = 0; i < halo; i++) {

        sData[threadIdx.y * sDataWidth + threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X255] : 0;
    }

    //load right halo

    for (int i = halo + step; i < halo + step + halo; i++) {
        
        sData[threadIdx.y * sDataWidth + threadIdx.x + i * ROW_BLOCK_DIM_X255] = (baseX + i * ROW_BLOCK_DIM_X255) < imageW ? d_Src[i * ROW_BLOCK_DIM_X255] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve

    for (int i = halo; i < halo+step; i++){

        if(baseX + i * ROW_BLOCK_DIM_X255 < imageW){

            float sum = 0;
            
            for (int j = -radius; j <= radius; j++) {

                sum += c_Kernel[radius - j] * sData[threadIdx.y * sDataWidth + threadIdx.x + i * ROW_BLOCK_DIM_X255 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X255] = sum;
        }

    }
}


__global__ void colConvolutionFilter(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int radius,
    int blockX,
    int blockY,
    int step,
    int halo
)
{
    int COL_BLOCK_DIM_X255 = blockX;
    int COL_BLOCK_DIM_Y255 = blockY;

    extern __shared__ float sData[]; //+1 to avoid shared mem bank conflicts
    int sDataWidth = (blockY * (step + 2 * halo) + 1);
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X255 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y255 * step - halo * COL_BLOCK_DIM_Y255 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data

    for (int i = halo; i < halo + step; i++) {
        
        sData[threadIdx.x * sDataWidth + threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) < imageH ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;
    
    }

    //load top halo
    
    for (int i = 0; i < halo; i ++) {

        sData[threadIdx.x * sDataWidth + threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;

    }
    //load bottom halo
    
    for (int i = halo + step; i < halo + step + halo; i++) {
        
        sData[threadIdx.x * sDataWidth + threadIdx.y + i * COL_BLOCK_DIM_Y255] = (baseY + i * COL_BLOCK_DIM_Y255) < imageH ? d_Src[i * COL_BLOCK_DIM_Y255 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
    
    for (int i = halo; i < halo + step; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y255) < imageH) {

            float sum = 0;
            
            for (int j = -radius; j <= radius; j++) {
                
                sum += c_Kernel[radius - j] * sData[threadIdx.x * sDataWidth + threadIdx.y + i * COL_BLOCK_DIM_Y255 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y255 * pitch] = sum;
        }
    }
}

void rowConvolve(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int radius,
    int blockX,
    int blockY,
    int step,
    int halo
    ){

        dim3 blocks((imageW + (step * blockX) - 1) / (step * blockX), (imageH + blockY - 1)  / blockY);
        dim3 threads(blockX, blockY);

        rowConvolutionFilter<<<blocks,threads,blockY * blockX * (step + 2 * halo) * sizeof(float) >>>(d_Dst,d_Src,imageW,imageH,pitch,radius,blockX,blockY,step,halo);

    }

void colConvolve(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch,
    int radius,
    int blockX,
    int blockY,
    int step,
    int halo
    ){

        dim3 blocks((imageW + blockX - 1) / blockX, (imageH + blockY * step - 1)  / (blockY * step));
        dim3 threads(blockX, blockY);
        
        colConvolutionFilter<<<blocks,threads,blockX * (blockY * (step + 2 * halo) + 1) * sizeof(float)>>>(d_Dst,d_Src,imageW,imageH,pitch,radius,blockX,blockY,step,halo);

    }