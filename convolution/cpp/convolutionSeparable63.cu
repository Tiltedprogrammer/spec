#include "hip/hip_runtime.h"
#include "convolutionSeparable63.hpp"
#include "defines.hpp"

#define ROW_BLOCK_DIM_X63 32
#define ROW_BLOCK_DIM_Y63 16

__constant__ float c_Kernel63[256];

void setConvolutionKernel63(float* h_Kernel, int k_length)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel63), h_Kernel, k_length * sizeof(float));
}

__global__ void rowConvolutionFilter63(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
){
    __shared__ float sData[ROW_BLOCK_DIM_Y63][(ROW_RESULT_STEP + 2*ROW_HALO_STEP63) * ROW_BLOCK_DIM_X63];

    //offset to left halo edge
    const int baseX = (blockIdx.x * ROW_RESULT_STEP) * ROW_BLOCK_DIM_X63 - ROW_HALO_STEP63 * ROW_BLOCK_DIM_X63 + threadIdx.x;
    const int baseY = blockIdx.y * ROW_BLOCK_DIM_Y63 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;
    
    //load main data
    
#pragma unroll

    for (int i = ROW_HALO_STEP63; i < ROW_HALO_STEP63 + ROW_RESULT_STEP; i++) {
     
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X63] = (baseX + i * ROW_BLOCK_DIM_X63) < imageW ? d_Src[i*ROW_BLOCK_DIM_X63] : 0;
    
    }

    //load left halo
#pragma unroll
    
    for (int i = 0; i < ROW_HALO_STEP63; i++) {

        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X63] = (baseX + i * ROW_BLOCK_DIM_X63) >= 0 ? d_Src[i*ROW_BLOCK_DIM_X63] : 0;
    }

    //load right halo
#pragma unroll

    for (int i = ROW_HALO_STEP63 + ROW_RESULT_STEP; i < ROW_HALO_STEP63 + ROW_RESULT_STEP + ROW_HALO_STEP63; i++) {
        
        sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X63] = (baseX + i * ROW_BLOCK_DIM_X63) < imageW ? d_Src[i * ROW_BLOCK_DIM_X63] : 0;
    
    }

    __syncthreads();

    if (baseY >= imageH) {
        return;
    }

    //convolve
#pragma unroll

    for (int i = ROW_HALO_STEP63; i < ROW_HALO_STEP63+ROW_RESULT_STEP; i++){

        if(baseX + i * ROW_BLOCK_DIM_X63 < imageW){

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS63; j <= KERNEL_RADIUS63; j++) {

                sum += c_Kernel63[KERNEL_RADIUS63 - j] * sData[threadIdx.y][threadIdx.x + i * ROW_BLOCK_DIM_X63 + j];    
            
            }

            d_Dst[i*ROW_BLOCK_DIM_X63] = sum;
        }

    }
}

#define COL_BLOCK_DIM_X63 16
#define COL_BLOCK_DIM_Y63 32

__global__ void colConvolutionFilter63(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{

    __shared__ float sData[COL_BLOCK_DIM_X63][(COL_RESULT_STEP + 2 * COL_HALO_STEP63) * COL_BLOCK_DIM_Y63 + 1]; //+1 to avoid shared mem bank conflicts
    
    const int baseX = blockIdx.x * COL_BLOCK_DIM_X63 + threadIdx.x;
    const int baseY = blockIdx.y * COL_BLOCK_DIM_Y63 * COL_RESULT_STEP - COL_HALO_STEP63 * COL_BLOCK_DIM_Y63 + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //load main data
#pragma unroll

    for (int i = COL_HALO_STEP63; i < COL_HALO_STEP63 + COL_RESULT_STEP; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y63] = (baseY + i * COL_BLOCK_DIM_Y63) < imageH ? d_Src[i * COL_BLOCK_DIM_Y63 * pitch] : 0;
    
    }

    //load top halo
#pragma unroll
    
    for (int i = 0; i < COL_HALO_STEP63; i ++) {

        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y63] = (baseY + i * COL_BLOCK_DIM_Y63) >= 0 ? d_Src[i * COL_BLOCK_DIM_Y63 * pitch] : 0;

    }
    //load bottom halo
#pragma unroll
    
    for (int i = COL_HALO_STEP63 + COL_RESULT_STEP; i < COL_HALO_STEP63 + COL_RESULT_STEP + COL_HALO_STEP63; i++) {
        
        sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y63] = (baseY + i * COL_BLOCK_DIM_Y63) < imageH ? d_Src[i * COL_BLOCK_DIM_Y63 * pitch] : 0;
    
    }

    __syncthreads();

    if (baseX >= imageW) {
        return;
    }

    //convolve
#pragma unroll
    
    for (int i = COL_HALO_STEP63; i < COL_HALO_STEP63 + COL_RESULT_STEP; i++) {

        if ((baseY + i * COL_BLOCK_DIM_Y63) < imageH) {

            float sum = 0;

        #pragma unroll
            
            for (int j = -KERNEL_RADIUS63; j <= KERNEL_RADIUS63; j++) {
                
                sum += c_Kernel63[KERNEL_RADIUS63 - j] * sData[threadIdx.x][threadIdx.y + i * COL_BLOCK_DIM_Y63 + j];
            }

            d_Dst[i * COL_BLOCK_DIM_Y63 * pitch] = sum;
        }
    }
}

void rowConvolve63(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + (ROW_RESULT_STEP * ROW_BLOCK_DIM_X63) - 1) / (ROW_RESULT_STEP * ROW_BLOCK_DIM_X63), (imageH + ROW_BLOCK_DIM_Y63 - 1)  / ROW_BLOCK_DIM_Y63);
        dim3 threads(ROW_BLOCK_DIM_X63, ROW_BLOCK_DIM_Y63);

        rowConvolutionFilter63<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }

void colConvolve63(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
    ){

        dim3 blocks((imageW + COL_BLOCK_DIM_X63 - 1) / COL_BLOCK_DIM_X63, (imageH + COL_BLOCK_DIM_Y63 * COL_RESULT_STEP - 1)  / (COL_BLOCK_DIM_Y63 * COL_RESULT_STEP));
        dim3 threads(COL_BLOCK_DIM_X63, COL_BLOCK_DIM_Y63);
        
        colConvolutionFilter63<<<blocks,threads>>>(d_Dst,d_Src,imageW,imageH,pitch);

    }