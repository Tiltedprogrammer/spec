#include <iostream>
#include <cstdlib>
#include <cassert>

// CUDA runtime
// #include <hip/hip_runtime.h>

#define cimg_use_jpeg

#include "../cimg/CImg-2.8.3/CImg.h"
#include "../cpp/convolutionSeparable_gold.hpp"


#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from convolutionSeparable.impala
#include "convolutionSeparable.inc"

//timers
#include "timer.h"
#include "cxxopts.hpp"


int main(int argc, char** argv) {

    if (argc < 4) {
        std::cout << "Image path and #iterations required" << "\n";
        return 0;
    }

    int KERNEL_LENGTH = std::atoi(argv[3]);
    std::string img_path(argv[1]);
    int iterations = std::atoi(argv[2]);
    std::cout << "# of iterations set to " << iterations << "\n";
    // cimg_library::CImg<float> img1("/home/alekseytyurinspb_gmail_com/specialization/spec/convolution/images/graytussaint100.jpg");
    cimg_library::CImg<float> img1(img_path.c_str());

    float* h_Kernel = new float[KERNEL_LENGTH];
    float* h_Output = new float[img1.width() * img1.height()];
    float* h_OutputGold = new float[img1.width() * img1.height()];
    float* h_BufferGold = new float[img1.width() * img1.height()];

    float  *d_Input,
           *d_Buffer,
           *d_Output;


    size_t pitch;
    hipMallocPitch((void**)&d_Input,&pitch,img1.width() * sizeof(float),img1.height());
    hipMallocPitch((void**)&d_Buffer,&pitch,img1.width() * sizeof(float),img1.height());
    hipMallocPitch((void**)&d_Output,&pitch,img1.width() * sizeof(float),img1.height());

    hipMemcpy2D(d_Input, pitch, img1.data(), img1.width()*sizeof(float), img1.width()*sizeof(float), img1.height(), hipMemcpyHostToDevice);

    srand(200);
    
    for (unsigned int i = 0; i < KERNEL_LENGTH; i++) {
        
        h_Kernel[i] = (float)(rand() % 16);
        
    }

    for (int i = 0; i < KERNEL_LENGTH; i++) {
        std::cout << h_Kernel[i] << " ";
    }
    std::cout << "\n";

    std::string kernel_string;

    for (int i = 0; i < KERNEL_LENGTH - 1; i++) {
        kernel_string += std::to_string(h_Kernel[i]);
        kernel_string += "f32, ";
    }
    kernel_string += std::to_string(h_Kernel[KERNEL_LENGTH - 1]) + "f32";

    int block_sizeX = 32;
    int block_sizeY = 16;
    int result_step = 8;

    if(KERNEL_LENGTH <= 63 ){ //radius < 31
        block_sizeX = 32;
        block_sizeY = 16;
    }else if(KERNEL_LENGTH <= 127){ //radius is 63
        block_sizeX = 64;
        block_sizeY = 8;
    }else if (KERNEL_LENGTH <= 255){
        block_sizeX = 128;
        block_sizeY = 4;
    }else{
        std::cout << "Too huge kernel length, maximum supported is 255" << "\n";
        return 0;
    }
    

    std::string dummy = "extern fn dummy(d_Src: &[f32],d_Buf : &mut[f32],d_Dst: &mut[f32])-> (){\n";
    dummy += "   convolveImpala(d_Src, d_Buf, d_Dst, [" +
            kernel_string + "], " +
            std::to_string((KERNEL_LENGTH - 1) / 2) + "i32, " +
            std::to_string(img1.height()) + "i32, " +
            std::to_string(img1.width()) + "i32, " +
            std::to_string(pitch / sizeof(float)) + "i32, " +
            std::to_string(block_sizeX) + "i32, " +
            std::to_string(block_sizeY) + "i32, " +
            std::to_string(result_step) + "i32)\n }";

    std::string program = std::string((char*)convolutionSeparable_impala) + dummy;

    std::cout << "Compiling ..." << "\n";
    am::timer time;
    time.start();
    
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    
    time.stop();
    std::cout << "compilation time " << time.milliseconds() << std::endl;
    time.reset();

    typedef void (*function) (const float*,const float* ,const float *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compilation failed\n";
        return 0;
    } else {
        std::cout << "succesfully compiled\n";
    }

    
    for (int j = 0; j < iterations; j++){
        call(d_Input,d_Buffer,d_Output);
    }
    hipDeviceSynchronize();

    hipMemcpy2D(h_Output, img1.width() * sizeof(float), d_Output, pitch, img1.width()*sizeof(float), img1.height(), hipMemcpyDeviceToHost);


    //gold

    convolutionRowCPU(h_BufferGold,img1.data(),h_Kernel,img1.width(),img1.height(),(KERNEL_LENGTH - 1) /2);
    convolutionColumnCPU(h_OutputGold,h_BufferGold,h_Kernel,img1.width(),img1.height(),(KERNEL_LENGTH - 1) /2);
    

    cimg_library::CImg<float> output(h_Output,img1.width(),img1.height(),1,1);
    cimg_library::CImg<float> convolved(h_OutputGold,img1.width(),img1.height(),1,1);

    //Tests whether convolution is correct
    assert(convolved == output);
    output.save("impala-convolved.jpg");
    convolved.save("manually-convolved.jpg");
    std::cout << "pitch = " << pitch << "\n";

    delete[] (h_Kernel);
    delete[] (h_Output);
    delete[] (h_OutputGold);
    delete[] (h_BufferGold);

    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_Output);

}