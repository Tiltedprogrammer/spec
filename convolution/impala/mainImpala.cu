#include <iostream>
#include <cstdlib>
#include <cassert>

// CUDA runtime
// #include <hip/hip_runtime.h>

#define cimg_use_jpeg

#include "../cimg/CImg-2.8.3/CImg.h"
#include "../cpp/convolutionSeparable_gold.hpp"


#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from convolutionSeparable.impala
#include "convolutionSeparable.inc"

//timers
#include "../include/timer.h"
//arg parsing
#include "../include/cxxopts.hpp"


int main(int argc, char** argv) {

    cxxopts::Options options("as", " - example command line options");
    options.add_options()("f,filename","path to image to convolve",cxxopts::value<std::string>())
                         ("o,outfile","path to save convolved image",cxxopts::value<std::string>())
                         ("i,isize","size of the image to generate : isize x isize",cxxopts::value<int>())
                         ("s,fsize","size of the filter to convolve with",cxxopts::value<int>())
                         ("c,static","whether to use static filters or not : 0 for not, 1 is default",cxxopts::value<int>())
                         ("t,test","assert correctness of the filter",cxxopts::value<int>());

    auto result = options.parse(argc, argv);
    int flag = 0;
    int test = 0;
    int KERNEL_LENGTH = 3;
    int imageH = 0;
    int imageW = 0;
    if(result.count("fsize")){
        KERNEL_LENGTH = result["fsize"].as<int>();
        assert(KERNEL_LENGTH % 2 == 1);
    }else{
        std::cout << "filter size if required" << "\n";
        return 0;
    }

    std::string img_path;
    int KERNEL_RADIUS = (KERNEL_LENGTH - 1) / 2;
    int image = 0;
    if(result.count("filename")){
        img_path = result["filename"].as<std::string>();
        image = 1;
    }else if(result.count("isize")){
        imageH = imageW = result["isize"].as<int>();
    }else{
        std::cout << "Either input image or its size is required" << "\n";
        return 0;
    }

    if(result.count("static")){
        flag = result["static"].as<int>();
    }

    if(result.count("test")){
        test = result["test"].as<int>();
    }

    int iterations = 1;

    // cimg_library::CImg<float> img1("/home/alekseytyurinspb_gmail_com/specialization/spec/convolution/images/graytussaint100.jpg");
    srand(200);
    
    float* h_Input;
    
    if(image){
        cimg_library::CImg<float> img1(img_path.c_str());
        imageW = img1.width();
        imageH = img1.height();
        h_Input = new float [imageH * imageW];
        for (int i = 0; i < imageW * imageH; i++)
        {
            h_Input[i] = img1.data()[i];
        }
    }else{
        long size = imageH * imageW;
        h_Input = new float [size];
        for (long i = 0; i < imageW * imageH; i++)
        {
            h_Input[i] = (float)(rand() % 16);
        }
    }


    std::cout << "image size is " << imageW << "x" << imageH <<"\n";

    float* h_Kernel = new float[KERNEL_LENGTH];
    
    float* h_Output = new float[imageW * imageH];
    

    float  *d_Input,
           *d_Buffer,
           *d_Output;

    size_t pitch;
    hipMallocPitch((void**)&d_Input,&pitch,imageW * sizeof(float),imageH);
    hipMallocPitch((void**)&d_Buffer,&pitch,imageW * sizeof(float),imageH);
    hipMallocPitch((void**)&d_Output,&pitch,imageW * sizeof(float),imageH);

    hipMemcpy2D(d_Input, pitch, h_Input, imageW*sizeof(float), imageW*sizeof(float), imageH, hipMemcpyHostToDevice);
    
    for (unsigned int i = 0; i < KERNEL_LENGTH; i++) {
        
        h_Kernel[i] = (float)(rand() % 16);
        
    }

    std::string kernel_string;

    for (int i = 0; i < KERNEL_LENGTH - 1; i++) {
        kernel_string += std::to_string(h_Kernel[i]);
        kernel_string += "f32, ";
    }
    kernel_string += std::to_string(h_Kernel[KERNEL_LENGTH - 1]) + "f32";

    int block_sizeX = 32;
    int block_sizeY = 16;
    int result_step = 8;

    if(KERNEL_LENGTH <= 63 ){ //radius < 31
        block_sizeX = 32;
        block_sizeY = 16;
    }else if(KERNEL_LENGTH <= 127){ //radius is 63
        block_sizeX = 64;
        block_sizeY = 8;
    }else if (KERNEL_LENGTH <= 255){
        block_sizeX = 128;
        block_sizeY = 4;
    }else{
        std::cout << "Too huge kernel length, maximum supported is 255" << "\n";
        return 0;
    }
    

    std::string dummy = "extern fn dummy(d_Src: &[f32],d_Buf : &mut[f32],d_Dst: &mut[f32])-> (){\n";
    dummy += "   convolveImpala(d_Src, d_Buf, d_Dst, [" +
            kernel_string + "], " +
            std::to_string((KERNEL_LENGTH - 1) / 2) + "i32, " +
            std::to_string(imageH) + "i32, " +
            std::to_string(imageW) + "i32, " +
            std::to_string(pitch / sizeof(float)) + "i32, " +
            std::to_string(block_sizeX) + "i32, " +
            std::to_string(block_sizeY) + "i32, " +
            std::to_string(result_step) + "i32)\n }";

    std::string program = std::string((char*)convolutionSeparable_impala) + dummy;

    std::cout << "Compiling ..." << "\n";
    am::timer time;
    time.start();
    
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    
    time.stop();
    std::cout << "compilation time " << time.milliseconds() << std::endl;
    time.reset();

    typedef void (*function) (const float*,const float* ,const float *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compilation failed\n";
        return 0;
    } else {
        std::cout << "succesfully compiled\n";
    }

    
    for (int j = 0; j < iterations; j++){
        call(d_Input,d_Buffer,d_Output);
    }
    hipDeviceSynchronize();

    hipMemcpy2D(h_Output, imageW * sizeof(float), d_Output, pitch, imageW*sizeof(float), imageH, hipMemcpyDeviceToHost);


    //gold
    if(test){
        float* h_OutputGold = new float[imageW * imageH];
        float* h_BufferGold = new float[imageW * imageH];

        convolutionRowCPU(h_BufferGold,h_Input,h_Kernel,imageW,imageH,(KERNEL_LENGTH - 1) /2);
        convolutionColumnCPU(h_OutputGold,h_BufferGold,h_Kernel,imageW,imageH,(KERNEL_LENGTH - 1) /2);

        for (long i = 0; i < imageH * imageW; i++) {
                assert(h_OutputGold[i] == h_Output[i]);
        }
        
        delete[] (h_OutputGold);
        delete[] (h_BufferGold);
    }

    // cimg_library::CImg<float> output(h_Output,img1.width(),img1.height(),1,1);
    // cimg_library::CImg<float> convolved(h_OutputGold,img1.width(),img1.height(),1,1);

    //Tests whether convolution is correct
    // assert(convolved == output);

    // output.save("impala-convolved.jpg");
    // convolved.save("manually-convolved.jpg");
    // std::cout << "pitch = " << pitch << "\n";

    delete[] (h_Input);
    delete[] (h_Kernel);
    delete[] (h_Output);

    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_Output);

}