#include <iostream>
#include <cstdlib>
#include <cassert>

// CUDA runtime
// #include <hip/hip_runtime.h>

#define cimg_use_jpeg

#include "../cimg/CImg-2.8.3/CImg.h"
#include "../cpp/convolutionSeparable_gold.hpp"


#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from convolutionSeparable.impala
#include "convolutionSeparable.inc"

#define KERNEL_LENGTH 8 * 2 + 1

int main(int argc, char** argv) {

    cimg_library::CImg<float> img1("/home/alekseytyurinspb_gmail_com/specialization/spec/convolution/images/graytussaint100.jpg");

    float* h_Kernel = new float[KERNEL_LENGTH];
    float* h_Output = new float[img1.width() * img1.height()];
    float* h_OutputGold = new float[img1.width() * img1.height()];
    float* h_BufferGold = new float[img1.width() * img1.height()];

    float  *d_Input,
           *d_Buffer,
           *d_Output;


    size_t pitch;
    hipMallocPitch((void**)&d_Input,&pitch,img1.width() * sizeof(float),img1.height());
    // hipMallocPitch((void**)&d_Buffer,&pitch,img1.width() * sizeof(float),img1.height());
    hipMallocPitch((void**)&d_Output,&pitch,img1.width() * sizeof(float),img1.height());

    hipMemcpy2D(d_Input, pitch, img1.data(), img1.width()*sizeof(float), img1.width()*sizeof(float), img1.height(), hipMemcpyHostToDevice);

    srand(200);
    
    for (unsigned int i = 0; i < KERNEL_LENGTH; i++) {
        
        h_Kernel[i] = (float)(rand() % 16);
        
    }

    std::string kernel_string;

    for (int i = 0; i < KERNEL_LENGTH - 1; i++) {
        kernel_string += std::to_string(h_Kernel[i]);
        kernel_string += "f32, ";
    }
    kernel_string += std::to_string(h_Kernel[KERNEL_LENGTH - 1]) + "f32";

    int block_sizeX = 32;
    int block_sizeY = 32;
    int result_step = 1;

    std::string dummy = "extern fn dummy(d_Src : &[f32], d_Dst : &mut[f32])-> (){\n";
    dummy += "   rowConvolveImpala(d_Src,d_Dst, [" +
            kernel_string + "], " +
            std::to_string((KERNEL_LENGTH - 1) / 2) + "i32, " +
            std::to_string(img1.height()) + "i32, " +
            std::to_string(img1.width()) + "i32, " +
            std::to_string(pitch / sizeof(float)) + "i32, " +
            std::to_string(block_sizeX) + "i32, " +
            std::to_string(block_sizeY) + "i32, " +
            std::to_string(result_step) + "i32);\n }";

    std::string program = std::string((char*)convolutionSeparable_impala) + dummy;

    std::cout << "Compiling ..." << "\n";
    
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    
    typedef void (*function) (const float*, const float *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return 0;
    } else {
        std::cout << "succesfully compiled\n";
    }

    call(d_Input,d_Output);
    hipDeviceSynchronize();

    hipMemcpy2D(h_Output, img1.width() * sizeof(float), d_Output, pitch, img1.width()*sizeof(float), img1.height(), hipMemcpyDeviceToHost);


    //gold

    convolutionRowCPU(h_BufferGold,img1.data(),h_Kernel,img1.width(),img1.height(),(KERNEL_LENGTH - 1) /2);
    
    cimg_library::CImg<float> output(h_Output,img1.width(),img1.height(),1,1);
    cimg_library::CImg<float> convolved(h_BufferGold,img1.width(),img1.height(),1,1);

    //Tests whether convolution is correct
    assert(convolved == output);
    output.save("impala-convolved.jpg");
    convolved.save("manually-convolved.jpg");
    std::cout << "pitch = " << pitch << "\n";

    delete[] (h_Kernel);
    delete[] (h_Output);
    delete[] (h_OutputGold);
    delete[] (h_BufferGold);

    hipFree(d_Input);
    // hipFree(d_Buffer);
    hipFree(d_Output);

}