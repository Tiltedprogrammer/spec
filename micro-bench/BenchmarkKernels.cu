#include "hip/hip_runtime.h"
#include "BenchKernels.hpp"

#include <cassert>

__constant__ int mini_array [2];

__global__ void dummy_kernel(int* dst){
    
    int t_id = threadIdx.x;
    int i = 42;
    dst[t_id] = mini_array[0];
}

__global__ void mini_kernel(int* src, int* dst, int* clocks){
    
    int t_id = blockIdx.x * gridDim.x + threadIdx.x;
    __shared__ int mini_shared [3];
    mini_shared[0] = 14;
    mini_shared[1] = 25;
    mini_shared[2] = 44;
    
    clock_t start,end;
    start = clock();
    
    int val = src[t_id]; //load from global
    dst[t_id] = val; //write to global

    end = clock();

    clocks[t_id] = (int)(start - end); //takes 634 cycles
    int next = t_id + 1;

    start = clock();
    
    int val2 = src[next]; //should be from L1
    dst[t_id] = val2; //should be to L1
    
    end = clock();

    clocks[t_id + 1] = (int)(start - end); //takes 76 cycles

    start = clock();
    
    dst[t_id] = 42; //to L1
    
    end = clock();  
    clocks[t_id + 2] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[0]; //load from const
    end = clock();
    clocks[t_id + 3] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[1]; //load from const cache
    end = clock();
    clocks[t_id + 4] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id + 1] = mini_shared[0]; //load from shared
    end = clock();
    clocks[t_id + 5] = (int)(start - end); //takes 54 cycles

}

__global__ void mini_kernel_2(int* src, int* dst, int* clocks){

    int t_id = blockIdx.x * gridDim.x + threadIdx.x;

    int val1 = 1;
    int val2,val3;
    int start,end;

    int* ptr = mini_array;

    // start = clock();
    asm volatile("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

    // asm volatile("ld.const.u32 %0, [%1];": "=r"(val1) : "l"(mini_array));
    asm volatile("ld.global.u32 %0, [%1];": "=r"(val1) : "l"(src));
    
    asm volatile("mov.u32 %0, %%clock;" : "=r"(end) :: "memory");
    // end = clock();
    // val1 = 42; //
    // val2 = src[t_id]; // L1 miss
    // val3 = src[t_id + 1]; //L1
    // val1 = mini_array[0];

    clocks[0] = end - start; 
    dst[t_id] = val1;
}

void set_const_mem(int * host_mem, int size){
    hipMemcpyToSymbol(HIP_SYMBOL(mini_array), host_mem, size * sizeof(int));
}

void mini_kernel_wrap(dim3 grid,dim3 block,int* src, int* dst,int* clocks){
    dummy_kernel<<<1,1>>>(dst);
    mini_kernel<<<grid,block>>>(src, dst, clocks);
    mini_kernel_2<<<grid,block>>>(src, dst, clocks);
    
}