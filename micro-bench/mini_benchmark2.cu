// specialized naive

#include <iostream>
#include <cstdlib>
#include <cassert>

// CUDA runtime
#include <hip/hip_runtime.h>

#define RUNTIME_ENABLE_JIT
// #include <anydsl_runtime.h>

// #include "kernel.inc"


#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaCheckError( const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    return;
}

// extern "C" void kernel(int*,int*,int);

__constant__ int mini_array [2];

__global__ void dummy_kernel(int* dst,int* clocks){
    
    int i;// = dst[0];
    int start,stop;
    int effect = mini_array[1]; //warm up cache
    dst[1] = effect;
    // start = clock();
    asm volatile("mov.u32 %0, %%clock;": "=r"(start) :: "memory");
    // asm volatile("st.global.u32 [%0], %1;": "=r"(clocks[0]) :"r"(start): "memory");
    
    asm volatile(
                //  "add.u32 %0, %1, 12;\n\t"
                "add.u32 %0, %1, %2;\n\t"
                //  "st.global.u32 [%1], 12;"
                //  :"=r"(i) :"r"(i): "memory");
                :"=r"(i) :"r"(i),"r"(mini_array[0]): "memory");
    asm volatile("mov.u32 %0, %%clock;": "=r"(stop) :: "memory");
    // asm volatile("st.global.u32 [%0], %1;": "=r"(clocks[1]) : "r"(stop): "memory");
    // dst[1] = effect;
    // stop = clock();
    clocks[0] = stop - start;
    dst[0] = i;
    // clocks[0] = stop - start;
}

__global__ void mini_kernel(int* src, int* dst, int* clocks){
    
    int t_id = blockIdx.x * gridDim.x + threadIdx.x;
    __shared__ int mini_shared [3];
    mini_shared[0] = 14;
    mini_shared[1] = 25;
    mini_shared[2] = 44;
    
    clock_t start,end;
    start = clock();
    
    int val = src[t_id]; //load from global
    dst[t_id] = val; //write to global

    end = clock();

    clocks[t_id] = (int)(start - end); //takes 634 cycles
    int next = t_id + 1;

    start = clock();
    
    int val2 = src[next]; //should be from L1
    dst[t_id] = val2; //should be to L1
    
    end = clock();

    clocks[t_id + 1] = (int)(start - end); //takes 76 cycles

    start = clock();
    
    dst[t_id] = 42; //to L1
    
    end = clock();  
    clocks[t_id + 2] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[0]; //load from const
    end = clock();
    clocks[t_id + 3] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[1]; //load from const cache
    end = clock();
    clocks[t_id + 4] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id + 1] = mini_shared[0]; //load from shared
    end = clock();
    clocks[t_id + 5] = (int)(start - end); //takes 54 cycles

}

__global__ void mini_register(int* src){
     
    int i = src[1];
    int j = src[0];
    
    asm volatile(
                 "add.u32 %0, %1, %2;\n\t"
                //  "st.global.u32 [%1], 12;"
                 :"=r"(i) :"r"(i), "r"(j): "memory");

    src[0] = i;
}

__global__ void mini_kernel_2(int* src, int* dst, int* clocks){
    

    int i = src[0];
    int start,stop;
    int effect = mini_array[1]; //warm up cache
    dst[1] = effect;
    
    asm volatile("mov.u32 %0, %%clock;": "=r"(start) :: "memory");
    
    // asm volatile(
    //              "shf.r.wrap.b32 %0, %0, 0, 2;\n\t"
    //             //  "st.global.u32 [%1], 12;"
    //              :"=r"(i) :"r"(i), "r"(mini_array[0]): "memory");
    asm volatile(
                 "div.u32 %0, %0, 4;\n\t"
                //  "st.global.u32 [%1], 12;"
                 :"=r"(i) :"r"(i), "r"(mini_array[0]): "memory");
    asm volatile("mov.u32 %0, %%clock;": "=r"(stop) :: "memory");
    
    clocks[1] = stop - start;
    dst[0] = i;
    // clocks[0] = stop - start;
}

void set_const_mem(int * host_mem, int size){
    hipMemcpyToSymbol(HIP_SYMBOL(mini_array), host_mem, size * sizeof(int));
}

void mini_kernel_wrap(dim3 grid,dim3 block,int* src, int* dst,int* clocks){
    dummy_kernel<<<1,1>>>(dst,clocks);
    // mini_kernel<<<grid,block>>>(src, dst, clocks);
    // mini_kernel_2<<<grid,block>>>(src, dst, clocks);
    // mini_register<<<grid,block>>>(src);
    
}


int main(int argc, char **argv)
{

    int* h_src = new int[2];
    int* h_clocks = new int[7];
    int* h_dst = new int[2];

    int* d_src;
    int* d_dst;
    int* d_clocks;

    hipMalloc((void**)&d_src, 2 * sizeof(int));
    hipMalloc((void**)&d_dst, 2 * sizeof(int));
    hipMalloc((void**)&d_clocks, 7 * sizeof(int));

    h_src[0] = 16;
    h_src[1] = 34;

    hipMemcpy(d_src,h_src,2 * sizeof(int),hipMemcpyHostToDevice);
    set_const_mem(h_src,2);

    dim3 block;

    block.x = 1;

    dim3 grid;

    grid.x = 1;
    
    memset(h_clocks,0,7 * sizeof(int));

    mini_kernel_wrap(grid,block,d_src,d_dst,d_clocks);

    // std::string r_naive_spec;

    // r_naive_spec += "extern fn dummy(src: &[i32], dst : &mut[i32]) -> (){\n";

    // r_naive_spec += "  kernel(src,dst,2)}"; //;

    // std::string program = std::string((char*)kernel_impala) + r_naive_spec;
    // auto key = anydsl_compile(program.c_str(),program.size(),0);
    // typedef void (*function) (const int*, int *);
    // auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    // if (call == nullptr) {
    //     std::cout << "compiliacion failed\n";
    //     return;
    // } else {
    //     std::cout << "succesfully compiled\n";
    // }


    memset(h_clocks,0,7 * sizeof(int));
    // call(d_src,d_dst);

    // kernel(d_src,d_dst,2);

    hipMemcpy(h_clocks,d_clocks,7 * sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(h_dst,d_dst, 2 * sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(h_src,d_src, 2 * sizeof(int),hipMemcpyDeviceToHost);


    std::cout << "Clocks for global load/store " << h_clocks[0] << "\n";
    std::cout << "Clocks for L1 load/store " << h_clocks[1] << "\n";
    std::cout << "Clocks for Spec load/store " << h_clocks[2] << "\n";
    std::cout << "Clocks for const load/store " << h_clocks[3] << "\n";
    std::cout << "Clocks for const cache load/store " << h_clocks[4] << "\n";
    std::cout << "Clocks for shared load/store " << h_clocks[5] << "\n";
    
    std::cout << h_src[0] << std::endl;
    // assert(h_dst[0] == 33);
    // assert(h_dst[1] == 14);
    
    hipFree(d_dst);
    hipFree(d_clocks);
    hipFree(d_src);

    delete[] (h_src);
    delete[] (h_clocks);
    delete[] (h_dst);


    return 0;
}