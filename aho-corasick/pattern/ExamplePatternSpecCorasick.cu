
#include <hip/hip_runtime.h>
extern "C" {
__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_21950(int, int, int*, int*, int);

__global__ __launch_bounds__ (256 * 1 * 1) void lambda_21950(int _21953_26963, int _21954_26964, int* _21955_26965, int* _21956_26966, int _21957_26967) {
    int  t_id_26976;
    int pt_id_26976;
    int  blockIdx_y_26982;
    int pblockIdx_y_26982;
    int  gridDim_x_26988;
    int pgridDim_x_26988;
    int  blockIdx_x_26994;
    int pblockIdx_x_26994;
    int*  s_Input_27011;
    int* ps_Input_27011;
    int  lower_27038;
    int plower_27038;
    
    int start_27001_slot;
    int* start_27001;
    start_27001 = &start_27001_slot;
    int matching_27051_slot;
    int* matching_27051;
    matching_27051 = &matching_27051_slot;
    int state_27065_slot;
    int* state_27065;
    state_27065 = &state_27065_slot;
    int pos_27049_slot;
    int* pos_27049;
    pos_27049 = &pos_27049_slot;
    t_id_26976 = threadIdx_x();
    pt_id_26976 = t_id_26976;
    l26974: ;
        t_id_26976 = pt_id_26976;
        blockIdx_y_26982 = blockIdx_y();
        pblockIdx_y_26982 = blockIdx_y_26982;
    l26980: ;
        blockIdx_y_26982 = pblockIdx_y_26982;
        gridDim_x_26988 = gridDim_x();
        pgridDim_x_26988 = gridDim_x_26988;
    l26986: ;
        gridDim_x_26988 = pgridDim_x_26988;
        blockIdx_x_26994 = blockIdx_x();
        pblockIdx_x_26994 = blockIdx_x_26994;
    l26992: ;
        blockIdx_x_26994 = pblockIdx_x_26994;
        int _27003;
        _27003 = blockIdx_y_26982 * gridDim_x_26988;
        int b_id_27004;
        b_id_27004 = _27003 + blockIdx_x_26994;
        int _27005;
        _27005 = 256 * b_id_27004;
        int start_27006;
        start_27006 = _27005 + t_id_26976;
        *start_27001 = start_27006;
        __shared__ int reserver_s_Input_27011[384];
        ps_Input_27011 = reserver_s_Input_27011;
    l27009: ;
        s_Input_27011 = ps_Input_27011;
        bool _27012;
        _27012 = _21957_26967 < b_id_27004;
        if (_27012) goto l27013; else goto l27014;
    l27014: ;
        int _27015;
        _27015 = *start_27001;
        int _27016;
        _27016 = _27015;
        bool _27017;
        _27017 = _27016 < _21954_26964;
        if (_27017) goto l27018; else goto l27276;
    l27276: ;
        goto l27019;
    l27018: ;
        int* _27273;
        _27273 = s_Input_27011 + t_id_26976;
        int _27267;
        _27267 = *start_27001;
        int _27269;
        _27269 = _27267;
        int* _27270;
        _27270 = _21956_26966 + _27269;
        int _27271;
        _27271 = *_27270;
        int _27274;
        _27274 = _27271;
        *_27273 = _27274;
        goto l27019;
    l27019: ;
        int _27021;
        _27021 = *start_27001;
        int _27022;
        _27022 = _27021;
        int _27023;
        _27023 = 256 + _27022;
        bool _27024;
        _27024 = _27023 < _21954_26964;
        *start_27001 = _27023;
        if (_27024) goto l27025; else goto l27265;
    l27265: ;
        goto l27263;
    l27025: ;
        bool _27027;
        _27027 = t_id_26976 < 128;
        if (_27027) goto l27028; else goto l27262;
    l27262: ;
        goto l27263;
    l27263: ;
        goto l27029;
    l27028: ;
        int _27258;
        _27258 = 256 + t_id_26976;
        int* _27259;
        _27259 = s_Input_27011 + _27258;
        int* _27255;
        _27255 = _21956_26966 + _27023;
        int _27256;
        _27256 = *_27255;
        int _27260;
        _27260 = _27256;
        *_27259 = _27260;
        goto l27029;
    l27029: ;
        __syncthreads();
    l27034: ;
        unsigned char* c_Input_27054;
        c_Input_27054 = (unsigned char*)s_Input_27011;
        int _27045;
        _27045 = 1024 * b_id_27004;
        int bdy_27046;
        bdy_27046 = _21953_26963 - _27045;
        int _27251;
        _27251 = _27045 + t_id_26976;
        *start_27001 = _27251;
        plower_27038 = 0;
        goto l27036;
    l27036: ;
        lower_27038 = plower_27038;
        bool _27040;
        _27040 = lower_27038 < 4;
        if (_27040) goto l27041; else goto l27250;
    l27250: ;
        return ;
    l27041: ;
        int _27042;
        _27042 = 256 * lower_27038;
        int pos_27043;
        pos_27043 = t_id_26976 + _27042;
        bool _27047;
        _27047 = pos_27043 < bdy_27046;
        *pos_27049 = pos_27043;
        *matching_27051 = 0;
        if (_27047) goto l27048; else goto l27249;
    l27249: ;
        goto l27112;
    l27048: ;
        unsigned char* _27055;
        _27055 = c_Input_27054 + pos_27043;
        unsigned char _27056;
        _27056 = *_27055;
        unsigned char _27057;
        _27057 = _27056;
        *state_27065 = -1;
        int c_27058;
        union { int dst; unsigned char src; } uc_27058;
        uc_27058.src = _27057;
        c_27058 = uc_27058.dst;
        bool _27060;
        _27060 = c_27058 == 65;
        if (_27060) goto l27061; else goto l27240;
    l27240: ;
        bool _27241;
        _27241 = c_27058 == 66;
        if (_27241) goto l27242; else goto l27244;
    l27244: ;
        bool _27245;
        _27245 = c_27058 == 69;
        if (_27245) goto l27246; else goto l27248;
    l27248: ;
        goto l27062;
    l27246: ;
        *state_27065 = 10;
        goto l27062;
    l27242: ;
        *state_27065 = 7;
        goto l27062;
    l27061: ;
        *state_27065 = 6;
        goto l27062;
    l27062: ;
        int _27066;
        _27066 = *state_27065;
        int _27067;
        _27067 = _27066;
        bool _27068;
        _27068 = -1 != _27067;
        if (_27068) goto l27069; else goto l27236;
    l27236: ;
        goto l27110;
    l27069: ;
        int _27071;
        _27071 = *state_27065;
        int _27072;
        _27072 = _27071;
        bool _27073;
        _27073 = _27072 <= 4;
        if (_27073) goto l27074; else goto l27235;
    l27235: ;
        goto l27075;
    l27074: ;
        int _27231;
        _27231 = *state_27065;
        int _27233;
        _27233 = _27231;
        *matching_27051 = _27233;
        goto l27075;
    l27075: ;
        int _27225;
        _27225 = *pos_27049;
        int _27227;
        _27227 = _27225;
        int _27228;
        _27228 = 1 + _27227;
        *pos_27049 = _27228;
        goto l27077;
    l27077: ;
        int _27079;
        _27079 = *pos_27049;
        int _27080;
        _27080 = _27079;
        bool _27081;
        _27081 = _27080 < bdy_27046;
        if (_27081) goto l27082; else goto l27224;
    l27224: ;
        goto l27108;
    l27082: ;
        int _27084;
        _27084 = *pos_27049;
        int _27086;
        _27086 = _27084;
        unsigned char* _27087;
        _27087 = c_Input_27054 + _27086;
        unsigned char _27088;
        _27088 = *_27087;
        unsigned char _27095;
        _27095 = _27088;
        int _27096;
        union { int dst; unsigned char src; } u_27096;
        u_27096.src = _27095;
        _27096 = u_27096.dst;
        int _27090;
        _27090 = *state_27065;
        int tmp_state_27091;
        tmp_state_27091 = _27090;
        bool _27093;
        _27093 = tmp_state_27091 == 1;
        *state_27065 = -1;
        if (_27093) goto l27094; else goto l27158;
    l27158: ;
        bool _27159;
        _27159 = tmp_state_27091 == 2;
        if (_27159) goto l27160; else goto l27161;
    l27161: ;
        bool _27163;
        _27163 = tmp_state_27091 == 3;
        if (_27163) goto l27164; else goto l27165;
    l27165: ;
        bool _27166;
        _27166 = tmp_state_27091 == 4;
        if (_27166) goto l27167; else goto l27168;
    l27168: ;
        bool _27170;
        _27170 = tmp_state_27091 == 5;
        bool _27184;
        _27184 = _27096 == 69;
        bool _27178;
        _27178 = _27096 == 66;
        if (_27170) goto l27171; else goto l27189;
    l27189: ;
        bool _27190;
        _27190 = tmp_state_27091 == 6;
        if (_27190) goto l27191; else goto l27195;
    l27195: ;
        bool _27196;
        _27196 = tmp_state_27091 == 7;
        if (_27196) goto l27197; else goto l27202;
    l27202: ;
        bool _27206;
        _27206 = _27096 == 68;
        bool _27203;
        _27203 = tmp_state_27091 == 8;
        if (_27203) goto l27204; else goto l27211;
    l27211: ;
        bool _27212;
        _27212 = tmp_state_27091 == 9;
        if (_27212) goto l27213; else goto l27217;
    l27217: ;
        bool _27218;
        _27218 = tmp_state_27091 == 10;
        if (_27218) goto l27219; else goto l27223;
    l27223: ;
        goto l27102;
    l27219: ;
        if (_27206) goto l27220; else goto l27222;
    l27222: ;
        goto l27100;
    l27220: ;
        *state_27065 = 4;
        goto l27100;
    l27213: ;
        if (_27184) goto l27214; else goto l27216;
    l27216: ;
        goto l27100;
    l27214: ;
        *state_27065 = 3;
        goto l27100;
    l27204: ;
        if (_27206) goto l27207; else goto l27210;
    l27210: ;
        goto l27100;
    l27207: ;
        *state_27065 = 9;
        goto l27100;
    l27197: ;
        if (_27184) goto l27198; else goto l27201;
    l27201: ;
        goto l27100;
    l27198: ;
        *state_27065 = 8;
        goto l27100;
    l27191: ;
        if (_27178) goto l27192; else goto l27194;
    l27194: ;
        goto l27100;
    l27192: ;
        *state_27065 = 1;
        goto l27100;
    l27171: ;
        bool _27172;
        _27172 = _27096 == 65;
        if (_27172) goto l27173; else goto l27176;
    l27176: ;
        if (_27178) goto l27179; else goto l27182;
    l27182: ;
        if (_27184) goto l27185; else goto l27188;
    l27188: ;
        goto l27100;
    l27185: ;
        *state_27065 = 10;
        goto l27100;
    l27179: ;
        *state_27065 = 7;
        goto l27100;
    l27173: ;
        *state_27065 = 6;
        goto l27100;
    l27167: ;
        goto l27100;
    l27164: ;
        goto l27100;
    l27160: ;
        goto l27100;
    l27094: ;
        bool _27098;
        _27098 = _27096 == 71;
        if (_27098) goto l27099; else goto l27157;
    l27157: ;
        goto l27100;
    l27099: ;
        *state_27065 = 2;
        goto l27100;
    l27100: ;
        goto l27102;
    l27102: ;
        int _27104;
        _27104 = *state_27065;
        int _27105;
        _27105 = _27104;
        bool _27106;
        _27106 = _27105 == -1;
        if (_27106) goto l27107; else goto l27135;
    l27135: ;
        int _27136;
        _27136 = *state_27065;
        int _27137;
        _27137 = _27136;
        bool _27138;
        _27138 = _27137 <= 4;
        if (_27138) goto l27139; else goto l27152;
    l27152: ;
        goto l27140;
    l27139: ;
        int _27148;
        _27148 = *state_27065;
        int _27150;
        _27150 = _27148;
        *matching_27051 = _27150;
        goto l27140;
    l27140: ;
        int _27142;
        _27142 = *pos_27049;
        int _27144;
        _27144 = _27142;
        int _27145;
        _27145 = 1 + _27144;
        *pos_27049 = _27145;
        goto l27077;
    l27107: ;
        goto l27108;
    l27108: ;
        goto l27110;
    l27110: ;
        goto l27112;
    l27112: ;
        int _27114;
        _27114 = *start_27001;
        int _27115;
        _27115 = _27114;
        bool _27116;
        _27116 = _21953_26963 <= _27115;
        if (_27116) goto l27117; else goto l27119;
    l27119: ;
        int _27120;
        _27120 = *start_27001;
        int _27133;
        _27133 = 1 + lower_27038;
        int _27124;
        _27124 = _27120;
        int* _27125;
        _27125 = _21955_26965 + _27124;
        int _27122;
        _27122 = *matching_27051;
        int _27126;
        _27126 = _27122;
        *_27125 = _27126;
        int _27128;
        _27128 = *start_27001;
        int _27130;
        _27130 = _27128;
        int _27131;
        _27131 = 256 + _27130;
        *start_27001 = _27131;
        plower_27038 = _27133;
        goto l27036;
    l27117: ;
        return ;
    l27013: ;
        return ;
}

}