#include "hip/hip_runtime.h"
naive_spec_manual
__global__
void match_naive_opt_spec_manual_corasick_jit(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {
    const int THREAD_BLOCK_SIZE = 256;
    const int EXTRA_SIZE_PER_TB = 128;
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;
    int start = gbid * THREAD_BLOCK_SIZE + t_id;
    int inputChar;
    int pos;
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    unsigned char *s_char;
    if ( gbid > num_blocks_minus1 ){
        return ;
    }
    s_char = (unsigned char *)s_input;
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }
    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();
    int bdy = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;
    for (int j = 0; j < 4; j++) {
        int match = 0;
        pos = t_id + j * THREAD_BLOCK_SIZE;
        if (pos < bdy){
            inputChar = s_char[pos];
            if(inputChar == 65){
              if(++pos < bdy){
                inputChar = s_char[pos];
                if(inputChar == 66){
                  match = 1;
            if(++pos < bdy){
      inputChar = s_char[pos];
if(inputChar == 69){
  match = 2;
}
else if(inputChar == 71){
  match = 3;
}
}
}
}
}
else if(inputChar == 66){
  if(++pos < bdy){
      inputChar = s_char[pos];
if(inputChar == 69){
  if(++pos < bdy){
      inputChar = s_char[pos];
if(inputChar == 68){
  if(++pos < bdy){
      inputChar = s_char[pos];
if(inputChar == 69){
  match = 4;
}
}
}
}
}
}
}
else if(inputChar == 69){
  if(++pos < bdy){
      inputChar = s_char[pos];
if(inputChar == 68){
  match = 5;
}
}
}
}
if (gbid < num_blocks_minus1) {
    d_match_result[start] = match;
    start += THREAD_BLOCK_SIZE;
}else {
     if (start >= input_size){
         return;
     }
     d_match_result[start] = match;
     start += THREAD_BLOCK_SIZE;
}
}
}

Kernel runtime 0.0264624 Std dev: 0.00474862
Kernel runtime 0.00932 Std dev: 0.00264708
PFAC
5 5
