#include "hip/hip_runtime.h"
#include "../spec_match.hpp"
#include "ImpalaKernels.hpp"


#define  SUBSEG_MATCH_NOTEX(j) \
    unsigned char s_char[37] = {0};\
    pos = t_id + j * THREAD_BLOCK_SIZE;\
    if (pos < bdy - 37 + 1){\
        _Pragma("unroll")\
        for(int i = 0; i < 37; i++){\
            s_char[i] = s_char_i[pos + i];\
        }\
        if ((s_char[0] == 0x14)\
                              & (s_char[1] == 'f')\
                              & (s_char[2] == 't')\
                              & (s_char[3] == 'y')\
                              & (s_char[4] == 'p')\
                              & (s_char[5] == 'i')\
                              & (s_char[6] == 's')\
                              & (s_char[7] == 'o')\
                              & (s_char[8] == 'm')){\
                              match = 1;}\
        if ((s_char[0] == 0x18)\
                              & (s_char[1] == 'f')\
                              & (s_char[2] == 't')\
                              & (s_char[3] == 'y')\
                              & (s_char[4] == 'p')\
                              & (s_char[5] == '3')\
                              & (s_char[6] == 'g')\
                              & (s_char[7] == 'p')\
                              & (s_char[8] == '5')){\
                              match = 2;}\
        if ((s_char[0] == 0x1a)\
                               & (s_char[1] == 'E')\
                               & (s_char[2] == 0xdf)\
                               & (s_char[3] == 0xa3)\
                               & (s_char[4] == 0x93)\
                               & (s_char[5] == 'B')\
                               & (s_char[6] == 0x82)\
                               & (s_char[7] == 0x88)\
                               & (s_char[8] == 'm')\
                               & (s_char[9] == 'a')\
                               & (s_char[10] == 't')\
                               & (s_char[11] == 'r')\
                               & (s_char[12] == 'o')\
                               & (s_char[13] == 's')\
                               & (s_char[14] == 'k')\
                               & (s_char[15] == 'a')){\
                               match = 3;}\
        if ((s_char[0] == 0x1f)\
                              & (s_char[1] == 0x8b)\
                              & (s_char[2] == 0x08)){\
                              match = 4;}\
        if ((s_char[0] == '%')\
                              & (s_char[1] == 'P')\
                              & (s_char[2] == 'D')\
                              & (s_char[3] == 'F')){\
                              match = 5;}\
        if ((s_char[0] == 0x37)\
                              & (s_char[1] == 0x7a)\
                              & (s_char[2] == 0xbc)\
                              & (s_char[3] == 0xaf)\
                              & (s_char[4] == 0x27)\
                              & (s_char[5] == 0x1c)){\
                              match = 6;}\
        if ((s_char[0] == '8')\
                              & (s_char[1] == 'B')\
                              & (s_char[2] == 'P')\
                              & (s_char[3] == 'S')){\
                              match = 7;}\
        if ((s_char[0] == '<')\
                              & (s_char[1] == '!')\
                              & (s_char[2] == 'd')\
                              & (s_char[3] == 'o')\
                              & (s_char[4] == 'c')\
                              & (s_char[5] == 't')\
                              & (s_char[6] == 'y')\
                              & (s_char[7] == 'p')){\
                              match = 8;}\
        if ((s_char[0] == 'C')\
                              & (s_char[1] == 'W')\
                              & (s_char[2] == 'S')){\
                              match = 9;}\
        if ((s_char[0] == 'F')\
                              & (s_char[1] == 'W')\
                              & (s_char[2] == 'S')){\
                              match = 10;}\
        if ((s_char[0] == 'G')\
                              & (s_char[1] == 'I')\
                              & (s_char[2 ] == 'F')\
                              & (s_char[3] == '8')\
                              & (s_char[4] == '7')\
                              & (s_char[5] == 'a')){\
                              match = 11;}\
        if ((s_char[0] == 'G')\
                              & (s_char[1] == 'I')\
                              & (s_char[2] == 'F')\
                              & (s_char[3] == '8')\
                              & (s_char[4] == '9')\
                              & (s_char[5] == 'a')){\
                              match = 12;}\
        if ((s_char[0] == 'I')\
                              & (s_char[1] == ' ')\
                              & (s_char[2] == 'I')){\
                              match = 13;}\
        if ((s_char[0] == 'I')\
                              & (s_char[1] == 'D')\
                              & (s_char[2] == '3')){\
                              match = 14;}\
        if ((s_char[0] == 'M')\
                              & (s_char[1] == 'Z')){\
                              match = 15;}\
        if ((s_char[0] == 'M')\
                               & (s_char[1] == 'i')\
                               & (s_char[2] == 'c')\
                               & (s_char[3] == 'r')\
                               & (s_char[4] == 'o')\
                               & (s_char[5] == 's')\
                               & (s_char[6] == 'o')\
                               & (s_char[7] == 'f')\
                               & (s_char[8] == 't')\
                               & (s_char[9] == ' ')\
                               & (s_char[10] == 'V')\
                               & (s_char[11] == 'i')\
                               & (s_char[12] == 's')\
                               & (s_char[13] == 'u')\
                               & (s_char[14] == 'a')\
                               & (s_char[15] == 'l')\
                               & (s_char[16] == ' ')\
                               & (s_char[17] == 'S')\
                               & (s_char[18] == 't')\
                               & (s_char[19] == 'u')\
                               & (s_char[20] == 'd')\
                               & (s_char[21] == 'i')\
                               & (s_char[22] == 'o')\
                               & (s_char[23] == ' ')\
                               & (s_char[24] == 'S')\
                               & (s_char[25] == 'o')\
                               & (s_char[26] == 'l')\
                               & (s_char[27] == 'u')\
                               & (s_char[28] == 't')\
                               & (s_char[29] == 'i')\
                               & (s_char[30] == 'o')\
                               & (s_char[31] == 'n')\
                               & (s_char[32] == ' ')\
                               & (s_char[33] == 'F')\
                               & (s_char[34] == 'i')\
                               & (s_char[35] == 'l')\
                               & (s_char[36] == 'e')){\
                               match = 16;}\
        if ((s_char[0] == 'O')\
                               & (s_char[1] == 'P')\
                               & (s_char[2] == 'L')\
                               & (s_char[3] == 'D')\
                               & (s_char[4] == 'a')\
                               & (s_char[5] == 't')\
                               & (s_char[6] == 'a')\
                               & (s_char[7] == 'b')\
                               & (s_char[8] == 'a')\
                               & (s_char[9] == 's')\
                               & (s_char[10] == 'e')\
                               & (s_char[11] == 'F')\
                               & (s_char[12] == 'i')\
                               & (s_char[13] == 'l')\
                               & (s_char[14] == 'e')){\
                               match = 17;}\
        if ((s_char[0] == 'P')\
                              & (s_char[1] == 'A')\
                              & (s_char[2] == 'G')\
                              & (s_char[3]  == 'E')\
                              & (s_char[4] == 'D')\
                              & (s_char[5] == 'U')\
                              & (s_char[6] == '6')\
                              & (s_char[7] == '4')){\
                              match = 18;}\
        if ((s_char[0] == 'P')\
                              & (s_char[1] == 'K')\
                              & (s_char[2] == 0x03)\
                              & (s_char[3] == 0x04)){\
                              match = 19;}\
        if ((s_char[0] == 'P')\
                              & (s_char[1] == 'K')\
                              & (s_char[2] == 0x05)\
                              & (s_char[3] == 0x06)\
                              & (s_char[4] == 'P')\
                              & (s_char[5] == 'K')\
                              & (s_char[6] == 0x07)\
                              & (s_char[7] == 0x08)){\
                              match = 20;}\
        if ((s_char[0] == 'R')\
                               & (s_char[1] == 'e')\
                               & (s_char[2] == 't')\
                               & (s_char[3] == 'u')\
                               & (s_char[4] == 'r')\
                               & (s_char[5] == 'n')\
                               & (s_char[6] == '-')\
                               & (s_char[7] == 'P')\
                               & (s_char[8] == 'a')\
                               & (s_char[9] == 't')\
                               & (s_char[10] == 'h')\
                               & (s_char[11] == ':')\
                               & (s_char[12] == ' ')){\
                               match = 21;}\
        if ((s_char[0] == '[')\
                               & (s_char[1] == 'W')\
                               & (s_char[2] == 'i')\
                               & (s_char[3] == 'n')\
                               & (s_char[4] == 'd')\
                               & (s_char[5] == 'o')\
                               & (s_char[6] == 'w')\
                               & (s_char[7] == 's')\
                               & (s_char[8] == ' ')\
                               & (s_char[9] == 'L')\
                               & (s_char[10] == 'a')\
                               & (s_char[11] == 't')\
                               & (s_char[12] == 'i')\
                               & (s_char[13] == 'n')\
                               & (s_char[14] == ' ')){\
                               match = 22;}\
        if ((s_char[0] == 'f')\
                        & (s_char[1] == 't')\
                        & (s_char[2] == 'y')\
                        & (s_char[3] == 'p')\
                        & (s_char[4] == 'M')\
                        & (s_char[5] == 'S')\
                        & (s_char[6] == 'N')\
                        & (s_char[7] == 'V')){\
                        match = 23;}\
        if ((s_char[0] == 0x7c)\
                               & (s_char[1] == 0x4b)\
                               & (s_char[2] == 0xc3)\
                               & (s_char[3] == 0x74)\
                               & (s_char[4] == 0xe1)\
                               & (s_char[5] == 0xc8)\
                               & (s_char[6] == 0x53)\
                               & (s_char[7] == 0xa4)\
                               & (s_char[8] == 0x79)\
                               & (s_char[9] == 0xb9)\
                               & (s_char[10] == 0x01)\
                               & (s_char[11] == 0x1d)\
                               & (s_char[12] == 0xfc)\
                               & (s_char[13] == 0x4f)\
                               & (s_char[14] == 0xdd)\
                               & (s_char[15] == 0x13)){\
                               match = 24;}\
        if ((s_char[0] == 0x7e)\
                               & (s_char[1] == 'E')\
                               & (s_char[2] == 'S')\
                               & (s_char[3] == 'D')\
                               & (s_char[4] == 'w')\
                               & (s_char[5] == 0xf6)\
                               & (s_char[6] == 0x85)\
                               & (s_char[7] == '>')\
                               & (s_char[8] == 0xbf)\
                               & (s_char[9] == 'j')\
                               & (s_char[10] == 0xd2)\
                               & (s_char[11] == 0x11)\
                               & (s_char[12] == 'E')\
                               & (s_char[13] == 'a')\
                               & (s_char[14] == 's')\
                               & (s_char[15] == 'y')\
                               & (s_char[16] == ' ')\
                               & (s_char[17] == 'S')\
                               & (s_char[18] == 't')\
                               & (s_char[19] == 'r')\
                               & (s_char[20] == 'e')\
                               & (s_char[21] == 'e')\
                               & (s_char[22] == 't')\
                               & (s_char[23] == ' ')\
                               & (s_char[24] == 'D')\
                               & (s_char[25] == 'r')\
                               & (s_char[26] == 'a')\
                               & (s_char[27] == 'w')){\
                               match = 25;}\
        if ((s_char[0] == 0xbe)\
                               & (s_char[1] == 0xba)\
                               & (s_char[2] == 0xfe)\
                               & (s_char[3] == 0xca)\
                               & (s_char[4] == 0x0f)\
                               & (s_char[5] == 'P')\
                               & (s_char[6] == 'a')\
                               & (s_char[7] == 'l')\
                               & (s_char[8] == 'm')\
                               & (s_char[9] == 'S')\
                               & (s_char[10] == 'G')\
                               & (s_char[11] == ' ')\
                               & (s_char[12] == 'D')\
                               & (s_char[13] == 'a')\
                               & (s_char[14] == 't')\
                               & (s_char[15] == 'a')){\
                               match = 26;}\
        if ((s_char[0] == 0xd0)\
                              & (s_char[1] == 0xcf)\
                              & (s_char[2] == 0x11)\
                              & (s_char[3] == 0xe0)\
                              & (s_char[4] == 0xa1)\
                              & (s_char[5] == 0xb1)\
                              & (s_char[6] == 0x1a)\
                              & (s_char[7] == 0xe1)){\
                              match = 27;}\
    }else if (pos < bdy){\
        if ((pos < bdy - 9 + 1) & (s_char_i[pos + 0] == 0x14)\
                              & (s_char_i[pos + 1] == 'f')\
                              & (s_char_i[pos + 2] == 't')\
                              & (s_char_i[pos + 3] == 'y')\
                              & (s_char_i[pos + 4] == 'p')\
                              & (s_char_i[pos + 5] == 'i')\
                              & (s_char_i[pos + 6] == 's')\
                              & (s_char_i[pos + 7] == 'o')\
                              & (s_char_i[pos + 8] == 'm')){\
                              match = 1;}\
        if ((pos < bdy - 9 + 1) & (s_char_i[pos + 0] == 0x18)\
                              & (s_char_i[pos + 1] == 'f')\
                              & (s_char_i[pos + 2] == 't')\
                              & (s_char_i[pos + 3] == 'y')\
                              & (s_char_i[pos + 4] == 'p')\
                              & (s_char_i[pos + 5] == '3')\
                              & (s_char_i[pos + 6] == 'g')\
                              & (s_char_i[pos + 7] == 'p')\
                              & (s_char_i[pos + 8] == '5')){\
                              match = 2;}\
        if ((pos < bdy - 16 + 1) & (s_char_i[pos + 0] == 0x1a)\
                               & (s_char_i[pos + 1] == 'E')\
                               & (s_char_i[pos + 2] == 0xdf)\
                               & (s_char_i[pos + 3] == 0xa3)\
                               & (s_char_i[pos + 4] == 0x93)\
                               & (s_char_i[pos + 5] == 'B')\
                               & (s_char_i[pos + 6] == 0x82)\
                               & (s_char_i[pos + 7] == 0x88)\
                               & (s_char_i[pos + 8] == 'm')\
                               & (s_char_i[pos + 10] == 't')\
                               & (s_char_i[pos + 11] == 'r')\
                               & (s_char_i[pos + 12] == 'o')\
                               & (s_char_i[pos + 13] == 's')\
                               & (s_char_i[pos + 14] == 'k')\
                               & (s_char_i[pos + 15] == 'a')){\
                               match = 3;}\
        if ((pos < bdy - 3 + 1) & (s_char_i[pos + 0] == 0x1f)\
                              & (s_char_i[pos + 1] == 0x8b)\
                              & (s_char_i[pos + 2] == 0x08)){\
                              match = 4;}\
        if ((pos < bdy - 4 + 1) & (s_char_i[pos + 0] == '%')\
                              & (s_char_i[pos + 1] == 'P')\
                              & (s_char_i[pos + 2] == 'D')\
                              & (s_char_i[pos + 3] == 'F')){\
                              match = 5;}\
        if ((pos < bdy - 6 + 1) & (s_char_i[pos + 0] == 0x37)\
                              & (s_char_i[pos + 1] == 0x7a)\
                              & (s_char_i[pos + 2] == 0xbc)\
                              & (s_char_i[pos + 3] == 0xaf)\
                              & (s_char_i[pos + 4] == 0x27)\
                              & (s_char_i[pos + 5] == 0x1c)){\
                              match = 6;}\
        if ((pos < bdy - 4 + 1) & (s_char_i[pos] == '8')\
                              & (s_char_i[pos + 1] == 'B')\
                              & (s_char_i[pos + 2] == 'P')\
                              & (s_char_i[pos + 3] == 'S')){\
                              match = 7;}\
        if ((pos < bdy - 8 + 1) & (s_char_i[pos] == '<')\
                              & (s_char_i[pos + 1] == '!')\
                              & (s_char_i[pos + 2] == 'd')\
                              & (s_char_i[pos + 3] == 'o')\
                              & (s_char_i[pos + 4] == 'c')\
                              & (s_char_i[pos + 5] == 't')\
                              & (s_char_i[pos + 6] == 'y')\
                              & (s_char_i[pos + 7] == 'p')){\
                              match = 8;}\
        if ((pos < bdy - 3 + 1) & (s_char_i[pos] == 'C')\
                              & (s_char_i[pos + 1] == 'W')\
                              & (s_char_i[pos + 2] == 'S')){\
                              match = 9;}\
        if ((pos < bdy - 3 + 1) & (s_char_i[pos] == 'F')\
                              & (s_char_i[pos + 1] == 'W')\
                              & (s_char_i[pos + 2] == 'S')){\
                              match = 10;}\
        if ((pos < bdy - 6 + 1) & (s_char_i[pos] == 'G')\
                              & (s_char_i[pos + 1] == 'I')\
                              & (s_char_i[pos + 2 ] == 'F')\
                              & (s_char_i[pos + 3] == '8')\
                              & (s_char_i[pos + 4] == '7')\
                              & (s_char_i[pos + 5] == 'a')){\
                              match = 11;}\
        if ((pos < bdy - 6 + 1) & (s_char_i[pos] == 'G')\
                              & (s_char_i[pos + 1] == 'I')\
                              & (s_char_i[pos + 2 ] == 'F')\
                              & (s_char_i[pos + 3] == '8')\
                              & (s_char_i[pos + 4] == '9')\
                              & (s_char_i[pos + 5] == 'a')){\
                              match = 12;}\
        if ((pos < bdy - 3 + 1) & (s_char_i[pos] == 'I')\
                              & (s_char_i[pos + 1] == ' ')\
                              & (s_char_i[pos + 2] == 'I')){\
                              match = 13;}\
        if ((pos < bdy - 3 + 1) & (s_char_i[pos] == 'I')\
                              & (s_char_i[pos + 1] == 'D')\
                              & (s_char_i[pos + 2] == '3')){\
                              match = 14;}\
        if ((pos < bdy - 2 + 1) & (s_char_i[pos] == 'M')\
                              & (s_char_i[pos + 1] == 'Z')){\
                              match = 15;}\
        if ((pos < bdy - 37 + 1) & (s_char_i[pos] == 'M')\
                               & (s_char_i[pos + 1] == 'i')\
                               & (s_char_i[pos + 2] == 'c')\
                               & (s_char_i[pos + 3] == 'r')\
                               & (s_char_i[pos + 4] == 'o')\
                               & (s_char_i[pos + 5] == 's')\
                               & (s_char_i[pos + 6] == 'o')\
                               & (s_char_i[pos + 7] == 'f')\
                               & (s_char_i[pos + 8] == 't')\
                               & (s_char_i[pos + 9] == ' ')\
                               & (s_char_i[pos + 10] == 'V')\
                               & (s_char_i[pos + 11] == 'i')\
                               & (s_char_i[pos + 12] == 's')\
                               & (s_char_i[pos + 13] == 'u')\
                               & (s_char_i[pos + 14] == 'a')\
                               & (s_char_i[pos + 15] == 'l')\
                               & (s_char_i[pos + 16] == ' ')\
                               & (s_char_i[pos + 17] == 'S')\
                               & (s_char_i[pos + 18] == 't')\
                               & (s_char_i[pos + 19] == 'u')\
                               & (s_char_i[pos + 20] == 'd')\
                               & (s_char_i[pos + 21] == 'i')\
                               & (s_char_i[pos + 22] == 'o')\
                               & (s_char_i[pos + 23] == ' ')\
                               & (s_char_i[pos + 24] == 'S')\
                               & (s_char_i[pos + 25] == 'o')\
                               & (s_char_i[pos + 26] == 'l')\
                               & (s_char_i[pos + 27] == 'u')\
                               & (s_char_i[pos + 28] == 't')\
                               & (s_char_i[pos + 29] == 'i')\
                               & (s_char_i[pos + 30] == 'o')\
                               & (s_char_i[pos + 31] == 'n')\
                               & (s_char_i[pos + 32] == ' ')\
                               & (s_char_i[pos + 33] == 'F')\
                               & (s_char_i[pos + 34] == 'i')\
                               & (s_char_i[pos + 35] == 'l')\
                               & (s_char_i[pos + 36] == 'e')){\
                               match = 16;}\
        if ((pos < bdy - 15 + 1) & (s_char_i[pos] == 'O')\
                               & (s_char_i[pos + 1] == 'P')\
                               & (s_char_i[pos + 2] == 'L')\
                               & (s_char_i[pos + 3] == 'D')\
                               & (s_char_i[pos + 4] == 'a')\
                               & (s_char_i[pos + 5] == 't')\
                               & (s_char_i[pos + 6] == 'a')\
                               & (s_char_i[pos + 7] == 'b')\
                               & (s_char_i[pos + 8] == 'a')\
                               & (s_char_i[pos + 9] == 's')\
                               & (s_char_i[pos + 10] == 'e')\
                               & (s_char_i[pos + 11] == 'F')\
                               & (s_char_i[pos + 12] == 'i')\
                               & (s_char_i[pos + 13] == 'l')\
                               & (s_char_i[pos + 14] == 'e')){\
                               match = 17;}\
        if ((pos < bdy - 8 + 1) & (s_char_i[pos] == 'P')\
                              & (s_char_i[pos + 1] == 'A')\
                              & (s_char_i[pos + 2] == 'G')\
                              & (s_char_i[pos + 3]  == 'E')\
                              & (s_char_i[pos + 4] == 'D')\
                              & (s_char_i[pos + 5] == 'U')\
                              & (s_char_i[pos + 6] == '6')\
                              & (s_char_i[pos + 7] == '4')){\
                              match = 18;}\
        if ((pos < bdy - 4 + 1) & (s_char_i[pos] == 'P')\
                              & (s_char_i[pos + 1] == 'K')\
                              & (s_char_i[pos + 2] == 0x03)\
                              & (s_char_i[pos + 3] == 0x04)){\
                              match = 19;}\
        if ((pos < bdy - 8 + 1) & (s_char_i[pos] == 'P')\
                              & (s_char_i[pos + 1] == 'K')\
                              & (s_char_i[pos + 2] == 0x05)\
                              & (s_char_i[pos + 3] == 0x06)\
                              & (s_char_i[pos + 4] == 'P')\
                              & (s_char_i[pos + 5] == 'K')\
                              & (s_char_i[pos + 6] == 0x07)\
                              & (s_char_i[pos + 7] == 0x08)){\
                              match = 20;}\
        if ((pos < bdy - 13 + 1) & (s_char_i[pos] == 'R')\
                               & (s_char_i[pos + 1] == 'e')\
                               & (s_char_i[pos + 2] == 't')\
                               & (s_char_i[pos + 3] == 'u')\
                               & (s_char_i[pos + 4] == 'r')\
                               & (s_char_i[pos + 5] == 'n')\
                               & (s_char_i[pos + 6] == '-')\
                               & (s_char_i[pos + 7] == 'P')\
                               & (s_char_i[pos + 8] == 'a')\
                               & (s_char_i[pos + 9] == 't')\
                               & (s_char_i[pos + 10] == 'h')\
                               & (s_char_i[pos + 11] == ':')\
                               & (s_char_i[pos + 12] == ' ')){\
                               match = 21;}\
        if ((pos < bdy - 15 + 1) & (s_char_i[pos] == '[')\
                               & (s_char_i[pos + 1] == 'W')\
                               & (s_char_i[pos + 2] == 'i')\
                               & (s_char_i[pos + 3] == 'n')\
                               & (s_char_i[pos + 4] == 'd')\
                               & (s_char_i[pos + 5] == 'o')\
                               & (s_char_i[pos + 6] == 'w')\
                               & (s_char_i[pos + 7] == 's')\
                               & (s_char_i[pos + 8] == ' ')\
                               & (s_char_i[pos + 9] == 'L')\
                               & (s_char_i[pos + 10] == 'a')\
                               & (s_char_i[pos + 11] == 't')\
                               & (s_char_i[pos + 12] == 'i')\
                               & (s_char_i[pos + 13] == 'n')\
                               & (s_char_i[pos + 14] == ' ')){\
                               match = 22;}\
        if ((pos < bdy - 8 + 1) & (s_char_i[pos] == 'f')\
                        & (s_char_i[pos + 1] == 't')\
                        & (s_char_i[pos + 2] == 'y')\
                        & (s_char_i[pos + 3] == 'p')\
                        & (s_char_i[pos + 4] == 'M')\
                        & (s_char_i[pos + 5] == 'S')\
                        & (s_char_i[pos + 6] == 'N')\
                        & (s_char_i[pos + 7] == 'V')){\
                        match = 23;}\
        if ((pos < bdy - 16 + 1) & (s_char_i[pos] == 0x7c)\
                               & (s_char_i[pos + 1] == 0x4b)\
                               & (s_char_i[pos + 2] == 0xc3)\
                               & (s_char_i[pos + 3] == 0x74)\
                               & (s_char_i[pos + 4] == 0xe1)\
                               & (s_char_i[pos + 5] == 0xc8)\
                               & (s_char_i[pos + 6] == 0x53)\
                               & (s_char_i[pos + 7] == 0xa4)\
                               & (s_char_i[pos + 8] == 0x79)\
                               & (s_char_i[pos + 9] == 0xb9)\
                               & (s_char_i[pos + 10] == 0x01)\
                               & (s_char_i[pos + 11] == 0x1d)\
                               & (s_char_i[pos + 12] == 0xfc)\
                               & (s_char_i[pos + 13] == 0x4f)\
                               & (s_char_i[pos + 14] == 0xdd)\
                               & (s_char_i[pos + 15] == 0x13)){\
                               match = 24;}\
        if ((pos < bdy - 28 + 1) & (s_char_i[pos] == 0x7e)\
                               & (s_char_i[pos + 1] == 'E')\
                               & (s_char_i[pos + 2] == 'S')\
                               & (s_char_i[pos + 3] == 'D')\
                               & (s_char_i[pos + 4] == 'w')\
                               & (s_char_i[pos + 5] == 0xf6)\
                               & (s_char_i[pos + 6] == 0x85)\
                               & (s_char_i[pos + 7] == '>')\
                               & (s_char_i[pos + 8] == 0xbf)\
                               & (s_char_i[pos + 9] == 'j')\
                               & (s_char_i[pos + 10] == 0xd2)\
                               & (s_char_i[pos + 11] == 0x11)\
                               & (s_char_i[pos + 12] == 'E')\
                               & (s_char_i[pos + 13] == 'a')\
                               & (s_char_i[pos + 14] == 's')\
                               & (s_char_i[pos + 15] == 'y')\
                               & (s_char_i[pos + 16] == ' ')\
                               & (s_char_i[pos + 17] == 'S')\
                               & (s_char_i[pos + 18] == 't')\
                               & (s_char_i[pos + 19] == 'r')\
                               & (s_char_i[pos + 20] == 'e')\
                               & (s_char_i[pos + 21] == 'e')\
                               & (s_char_i[pos + 22] == 't')\
                               & (s_char_i[pos + 23] == ' ')\
                               & (s_char_i[pos + 24] == 'D')\
                               & (s_char_i[pos + 25] == 'r')\
                               & (s_char_i[pos + 26] == 'a')\
                               & (s_char_i[pos + 27] == 'w')){\
                               match = 25;}\
        if ((pos < bdy - 16 + 1) & (s_char_i[pos] == 0xbe)\
                               & (s_char_i[pos + 1] == 0xba)\
                               & (s_char_i[pos + 2] == 0xfe)\
                               & (s_char_i[pos + 3] == 0xca)\
                               & (s_char_i[pos + 4] == 0x0f)\
                               & (s_char_i[pos + 5] == 'P')\
                               & (s_char_i[pos + 6] == 'a')\
                               & (s_char_i[pos + 7] == 'l')\
                               & (s_char_i[pos + 8] == 'm')\
                               & (s_char_i[pos + 9] == 'S')\
                               & (s_char_i[pos + 10] == 'G')\
                               & (s_char_i[pos + 11] == ' ')\
                               & (s_char_i[pos + 12] == 'D')\
                               & (s_char_i[pos + 13] == 'a')\
                               & (s_char_i[pos + 14] == 't')\
                               & (s_char_i[pos + 15] == 'a')){\
                               match = 26;}\
        if ((pos < bdy - 8 + 1) & (s_char_i[pos] == 0xd0)\
                              & (s_char_i[pos + 1] == 0xcf)\
                              & (s_char_i[pos + 2] == 0x11)\
                              & (s_char_i[pos + 3] == 0xe0)\
                              & (s_char_i[pos + 4] == 0xa1)\
                              & (s_char_i[pos + 5] == 0xb1)\
                              & (s_char_i[pos + 6] == 0x1a)\
                              & (s_char_i[pos + 7] == 0xe1)){\
                              match = 27;}\
    }\
    if (gbid < num_blocks_minus1) {\
        d_match_result[start] = match;\
        start += THREAD_BLOCK_SIZE;\
    }else {\
        if (start >= input_size){\
            return;\
        }\
        d_match_result[start] = match;\
        start += THREAD_BLOCK_SIZE;\
    }


__global__ void match_naive_opt_spec_manual_nu_bw(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {
    
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    int start = gbid * THREAD_BLOCK_SIZE + t_id ;
    int pos;
    int inputChar;
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    
    unsigned char *s_char_i;
    
    if ( gbid > num_blocks_minus1 ){
        return ; // whole block is outside input stream
    }

    s_char_i = (unsigned char *)s_input;

    // read global data to shared memory
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }

    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();

    int bdy_ = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    int bdy = (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4 > bdy_ ? bdy_ : (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4;


    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;

    for (int j = 0; j < 4; j++) {
        
        int match = 0;
        SUBSEG_MATCH_NOTEX(j)
    
    }
    
    
}

void matchNaiveSpecManualOptNUBWWrapper(PFAC_handle_t handle,dim3 grid, dim3 block,const int* d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result){
    
    std::vector<std::string> vpatterns;
    
    for (int i = 0; i < handle->numOfPatterns; i++) {
        vpatterns.push_back(std::string(handle->rowPtr[i],handle->patternLen_table[i+1]));
    }

    std::string kernel;
    
    kernel += "naive_spec_manual_bw\n";
    kernel += "__global__\n";
    kernel += "void match_naive_opt_spec_manual_nu_bw_jit(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {\n";
    kernel += "    const int THREAD_BLOCK_SIZE = " + std::to_string(THREAD_BLOCK_SIZE) + ";\n";
    kernel += "    const int EXTRA_SIZE_PER_TB = " + std::to_string(EXTRA_SIZE_PER_TB) + ";\n";
    kernel += "    int t_id = threadIdx.x;\n"
               "    int gbid = blockIdx.y * gridDim.x + blockIdx.x;\n"
               "    int start = gbid * THREAD_BLOCK_SIZE + t_id;\n"
               "    int pos;\n"
               "    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];\n"
               "    unsigned char *s_char;\n"
               "    if ( gbid > num_blocks_minus1 ){\n"
               "        return ;\n"
               "    }\n"
               "    s_char = (unsigned char *)s_input;\n"
               "    if ( start < n_hat ){\n"
               "        s_input[t_id] = d_input_string[start];\n"
               "    }\n"
               "    start += THREAD_BLOCK_SIZE ;\n"
               "    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){\n"
               "        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];\n"
               "    }\n"
               "    __syncthreads();\n"
               "    int bdy = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );\n"
               "    int legal_size = (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4 > bdy ? bdy : (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4;\n"
               "    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;\n"
               "    for (int j = 0; j < 4; j++) {\n"
               "        int match = 0;\n"
               "        unsigned char prefetched_char[" + std::to_string(handle->maxPatternLen) + "] = {0};\n"
               "        pos = t_id + j * THREAD_BLOCK_SIZE;\n"
               "        if (pos < legal_size -" + std::to_string(handle->maxPatternLen) + " + 1){\n"
               "            #pragma unroll\n"
               "            for (int i = 0; i < " + std::to_string(handle->maxPatternLen) + "; i++){\n"
               "                prefetched_char[i] = s_char[pos + i];\n"
               "            }\n";
    for (int i = 0; i < vpatterns.size(); i++){
        auto pattern = vpatterns[i];
        std::string if_clause = "           if (";
        for (int j = 0; j < pattern.size() - 1; j++){
            if_clause += "(prefetched_char[" + std::to_string(j) + "] == " + std::to_string((int)((unsigned char)pattern[j])) + ") & ";
        }
        if_clause += "(prefetched_char[" + std::to_string(pattern.size() - 1) + "] == " + std::to_string((int)((unsigned char)pattern[pattern.size() - 1])) + ")){\n"
                     "              match = " + std::to_string(i + 1) + ";}\n";
        kernel += if_clause;
    }

    kernel += "} else if(pos < legal_size){\n";

    for (int i = 0; i < vpatterns.size(); i++){
        auto pattern = vpatterns[i];
        std::string if_clause = "           if ((pos < legal_size - " + std::to_string(pattern.size()) + " + 1) & ";
        for (int j = 0; j < pattern.size() - 1; j++){
            if_clause += "(s_char[pos + " + std::to_string(j) + "] == " + std::to_string((int)((unsigned char)pattern[j])) + ") & ";
        }
        if_clause += "(s_char[pos + " + std::to_string(pattern.size() - 1) + "] == " + std::to_string((int)((unsigned char)pattern[pattern.size() - 1])) + ")){\n"
                     "              match = " + std::to_string(i + 1) + ";}\n";
        kernel += if_clause;
    }
    kernel += "}\n"
              "if (gbid < num_blocks_minus1) {\n"
              "    d_match_result[start] = match;\n"
              "    start += THREAD_BLOCK_SIZE;\n"
              "}else {\n"
              "     if (start >= input_size){\n"
              "         return;\n"
              "     }\n"
              "     d_match_result[start] = match;\n"
              "     start += THREAD_BLOCK_SIZE;\n"
              "}\n";

    kernel += "}\n}\n";
    // std::cout << kernel << std::endl;

    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(kernel);
    using jitify::reflection::type_of;

    auto kernel_instance = program.kernel("match_naive_opt_spec_manual_nu_bw_jit")
       .instantiate()
       .configure(grid, block);
    RUN(kernel_instance.launch(d_input_string,input_size,n_hat,num_blocks_minus1,d_match_result))

    // RUN((match_naive_opt_spec_manual_nu_bw<<<grid,block>>>(d_input_string,input_size,n_hat,num_blocks_minus1,d_match_result)))
}