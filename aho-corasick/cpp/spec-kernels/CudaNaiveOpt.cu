#include "hip/hip_runtime.h"
#include "../spec_match.hpp"
#include "ImpalaKernels.hpp"


#define MANUAL_EXPAND_2( X )   { X ; X ; }
#define MANUAL_EXPAND_4( X )   { MANUAL_EXPAND_2( MANUAL_EXPAND_2( X ) )  }


#define  SUBSEG_MATCH_NOTEX( j, match ) \
    pos = t_id + j * THREAD_BLOCK_SIZE ;\
    if ( pos < bdy_ ){\
        if (pos < bdy - max_len + 1){\
            int offset = 0;\
            for (int i = 0; i < p_num; i++){\
                int pos_in = pos;\
                int matched = 1;\
                for(int ii = 0; ii < p_sizes[i]; ii++) {\
                        inputChar = s_char[pos_in];\
                        if (inputChar != d_patterns[offset + ii]){\
                            matched = 0;\
                            break;\
                        }\
                        pos_in += 1;\
                }\
                offset += p_sizes[i];\
                if (matched != 0) {\
                    match = i + 1;\
                }\
            }\
        }else{\
                int offset = 0;\
                for (int i = 0; i < p_num; i++){\
                    int pos_in = pos;\
                    int matched = 1;\
                    if (pos_in < bdy - p_sizes[i] + 1) {\
                        for(int ii = 0; ii < p_sizes[i]; ii++) {\
                            inputChar = s_char[pos_in];\
                            if (inputChar != d_patterns[offset + ii]){\
                                matched = 0;\
                                break;\
                            }\
                            pos_in += 1;\
                        }\
                    }else {\
                        matched = 0;\
                    }\
                    offset += p_sizes[i];\
                    if (matched != 0) {\
                        match = i + 1;\
                    }\
                }\
        }\
    }

__global__ void match_naive_opt(const char* __restrict__ d_patterns, int* p_sizes, int p_num, const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int max_len, int* d_match_result) {
    
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    int start = gbid * THREAD_BLOCK_SIZE + t_id ;
    int pos;
    int inputChar;
    int match[4] = {0,0,0,0};
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    
    char *s_char;
    
    if ( gbid > num_blocks_minus1 ){
        return ; // whole block is outside input stream
    }

    s_char = (char *)s_input;

    // read global data to shared memory
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }

    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();

    int bdy_ = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    int bdy = (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4 > bdy_ ? bdy_ : (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4;

    int j = 0 ;

    MANUAL_EXPAND_4( SUBSEG_MATCH_NOTEX(j, match[j]) ; j++ ;)
    

    // write 4 results  match[0:3] to global d_match_result[0:input_size)
    // one thread block processes (BLOCKSIZE * 4) substrings
    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;

    if ( gbid < num_blocks_minus1 ){
        #pragma unroll
        for (int j = 0 ; j < 4 ; j++ ){
            d_match_result[start] = match[j];
            start += THREAD_BLOCK_SIZE;
        }
    }else{
        int j = 0 ;
        MANUAL_EXPAND_4( if (start>=input_size) return ; d_match_result[start] = match[j]; \
        j++ ; start += THREAD_BLOCK_SIZE ; )
    } 


}

void matchNaiveOptWrapper(dim3 grid, dim3 block,const char* d_patterns, int* p_sizes, int p_num, const int* d_input_string, int input_size, int n_hat, int num_blocks_minus1,int max_len, int* d_match_result){
    RUN((match_naive_opt<<<grid,block>>>(d_patterns,p_sizes,p_num,d_input_string,input_size,n_hat,num_blocks_minus1,max_len,d_match_result)))
}