#include "hip/hip_runtime.h"
#include "../spec_match.hpp"
#include "BenchKernels.hpp"

__constant__ int mini_array [2];

__global__ void dummy_kernel(){
    
    int t_id = threadIdx.x;
    int i = 42;
}

__global__ void mini_kernel(int* src, int* dst, int* clocks){
    
    int t_id = blockIdx.x * gridDim.x + threadIdx.x;
    __shared__ int mini_shared [3];
    mini_shared[0] = 14;
    mini_shared[1] = 25;
    mini_shared[2] = 44;
    
    clock_t start,end;
    start = clock();
    
    int val = src[t_id]; //load from global
    dst[t_id] = val; //write to global

    end = clock();

    clocks[t_id] = (int)(start - end); //takes 634 cycles
    int next = t_id + 1;

    start = clock();
    
    int val2 = src[next]; //should be from L1
    dst[t_id] = val2; //should be to L1
    
    end = clock();

    clocks[t_id + 1] = (int)(start - end); //takes 76 cycles

    start = clock();
    
    dst[t_id] = 42; //to L1
    
    end = clock();  
    clocks[t_id + 2] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[0]; //load from const
    end = clock();
    clocks[t_id + 3] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id] = mini_array[1]; //load from const cache
    end = clock();
    clocks[t_id + 4] = (int)(start - end); //takes 54 cycles

    start = clock();
    dst[t_id + 1] = mini_shared[0]; //load from shared
    end = clock();
    clocks[t_id + 5] = (int)(start - end); //takes 54 cycles

}

__global__ void mini_kernel_2(int* src, int* dst, int* clocks){

    int t_id = blockIdx.x * gridDim.x + threadIdx.x;

    int val1,val2,val3;
    val1 = 42; //
    val2 = src[t_id]; // L1 miss
    val3 = src[t_id + 1]; //L1

    dst[t_id] = val3;
}

void set_const_mem(int * host_mem, int size){
    hipMemcpyToSymbol(HIP_SYMBOL(mini_array), host_mem, size * sizeof(int));
}

void mini_kernel_wrap(dim3 grid,dim3 block,int* src, int* dst,int* clocks){
    dummy_kernel<<<1,1>>>();
    mini_kernel_2<<<grid,block>>>(src, dst, clocks);
    mini_kernel<<<grid,block>>>(src, dst, clocks);
}