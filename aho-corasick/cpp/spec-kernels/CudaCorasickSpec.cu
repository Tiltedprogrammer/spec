#include "hip/hip_runtime.h"
#include "../spec_match.hpp"
#include "ImpalaKernels.hpp"

#include <stack>

#define MANUAL_EXPAND_2( X )   { X ; X ; }
#define MANUAL_EXPAND_4( X )   { MANUAL_EXPAND_2( MANUAL_EXPAND_2( X ) )  }

//check pos < bdy
#define  SUBSEG_MATCH_NOTEX( j, match ) \
    pos = t_id + j * THREAD_BLOCK_SIZE;\
    if(pos < bdy){\
        inputChar = s_char[pos++];\
        /*"0x14ftypisom"*/\
        if (inputChar == 0x14){\
            inputChar = s_char[pos++];\
            if (inputChar == 'f'){\
                inputChar = s_char[pos++];\
                if (inputChar == 't'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'y') {\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'p'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'i') {\
                                inputChar = s_char[pos++];\
                                if (inputChar == 's') {\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'o'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 'm') {\
                                            match = 1;\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
            /*"0x18ftyp3gp5"*/\
        }else if (inputChar == 0x18){/*state # 37*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'f'){\
                inputChar = s_char[pos++];\
                if (inputChar == 't') {\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'y') {\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'p') {\
                            inputChar = s_char[pos++];\
                            if (inputChar == '3') {\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'g'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'p'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == '5'){\
                                            match = 2;\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
            /*0x1aE0xdf0xa30x93B0x820x88matroska*/\
        } else if(inputChar == 0x1a){ /*state # 45*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'E'){\
               inputChar = s_char[pos++];\
               if (inputChar == 0xdf) {\
                inputChar = s_char[pos++];\
                if (inputChar == 0xa3) {\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0x93){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'B'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 0x82) {\
                                inputChar = s_char[pos++];\
                                if (inputChar == 0x88) {\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'm'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 'a') {\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 't'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'r') {\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 'o') {\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 's'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'k'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 'a'){\
                                                                    match = 3;\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
               }\
            }\
        /*0x1f0x8b0x08*/\
        }else if (inputChar == 0x1f){ /*state #60*/\
            inputChar = s_char[pos++];\
            if (inputChar == 0x8b) {\
                inputChar = s_char[pos++];\
                if (inputChar == 0x08) {\
                    match = 4;\
                }\
            }\
        /*"%PDF"*/\
        }else if (inputChar == '%'){ /*state # 62*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'P'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'D'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'F'){\
                        match = 5;\
                    }\
                }\
            }\
        /*"0x370x7a0xbc0xaf0x270x1c"*/\
        }else if (inputChar == 0x37){ /*state #65*/\
            inputChar = s_char[pos++];\
            if (inputChar == 0x7a){\
                inputChar = s_char[pos++];\
                if (inputChar == 0xbc) {\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0xaf){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 0x27) {\
                            inputChar = s_char[pos++];\
                            if (inputChar == 0x1c){\
                                match = 6;\
                            }\
                        }\
                    }\
                }\
            }\
        /*"8BPS"*/\
        }else if (inputChar == '8'){ /*state #70*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'B') {\
                inputChar = s_char[pos++];\
                if (inputChar == 'P'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'S') {\
                        match = 7;\
                    }\
                }\
            }\
        /*"<!doctyp"*/\
        }else if (inputChar == '<'){ /*state #73*/\
            inputChar = s_char[pos++];\
            if (inputChar == '!'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'd'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'o') {\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'c') {\
                            inputChar = s_char[pos++];\
                            if (inputChar == 't') {\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'y') {\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'p') {\
                                        match = 8;\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        /*"CWS"*/\
        }else if (inputChar == 'C'){/*state #80*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'W') {\
                inputChar = s_char[pos++];\
                if (inputChar == 'S'){\
                    match = 9;\
                }\
            }\
        /*FWS*/\
        }else if (inputChar == 'F'){/*state #82*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'W') {\
                inputChar = s_char[pos++];\
                if (inputChar == 'S'){\
                    match = 10;\
                }\
            }\
        }else if (inputChar == 'G'){ /*state #84*/\
            inputChar = s_char[pos++];\
            if (inputChar == 'I'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'F'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == '8'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == '7'){ /*state #88*/\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'a'){\
                                match = 11;\
                            }\
                        } else if (inputChar == '9'){ /*state #89*/\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'a'){\
                                match = 12;\
                            }\
                        }\
                    }\
                }\
            }\
        }else if(inputChar == 'I'){ /*"state #90"*/\
            inputChar = s_char[pos++];\
            if (inputChar == ' ') {\
                inputChar = s_char[pos++];\
                if (inputChar == 'I'){\
                    match = 13;\
                }\
            }else if(inputChar == 'D'){\
                inputChar = s_char[pos++];\
                if (inputChar == '3'){\
                    match = 14;\
                }\
            }\
        }else if(inputChar == 'M'){\
            inputChar = s_char[pos++];\
            if (inputChar == 'Z'){\
                match = 15;\
            /*Microsoft Visual Studio Solution File*/\
            }else if(inputChar == 'i'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'c'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'r'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'o'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 's'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'o'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'f'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 't'){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == ' '){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'V') {\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 'i'){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 's'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'u'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 'a'){\
                                                                    inputChar = s_char[pos++];\
                                                                    if (inputChar == 'l'){\
                                                                        inputChar = s_char[pos++];\
                                                                        if (inputChar == ' '){\
                                                                            inputChar = s_char[pos++];\
                                                                            if (inputChar == 'S'){\
                                                                                inputChar = s_char[pos++];\
                                                                                if (inputChar == 't'){\
                                                                                    inputChar = s_char[pos++];\
                                                                                    if (inputChar == 'u'){\
                                                                                        inputChar = s_char[pos++];\
                                                                                        if (inputChar == 'd'){\
                                                                                            inputChar = s_char[pos++];\
                                                                                            if (inputChar == 'i'){\
                                                                                                inputChar = s_char[pos++];\
                                                                                                if (inputChar == 'o'){\
                                                                                                    inputChar = s_char[pos++];\
                                                                                                    if (inputChar == ' '){\
                                                                                                        inputChar = s_char[pos++];\
                                                                                                        if (inputChar == 'S'){\
                                                                                                            inputChar = s_char[pos++];\
                                                                                                            if (inputChar == 'o'){\
                                                                                                                inputChar = s_char[pos++];\
                                                                                                                if (inputChar == 'l'){\
                                                                                                                    inputChar = s_char[pos++];\
                                                                                                                    if (inputChar == 'u'){\
                                                                                                                        inputChar = s_char[pos++];\
                                                                                                                        if (inputChar == 't'){\
                                                                                                                            inputChar = s_char[pos++];\
                                                                                                                            if (inputChar == 'i'){\
                                                                                                                                inputChar = s_char[pos++];\
                                                                                                                                if (inputChar == 'o'){\
                                                                                                                                    inputChar = s_char[pos++];\
                                                                                                                                    if (inputChar == 'n'){\
                                                                                                                                        inputChar = s_char[pos++];\
                                                                                                                                        if (inputChar == ' '){\
                                                                                                                                            inputChar = s_char[pos++];\
                                                                                                                                            if (inputChar == 'F'){\
                                                                                                                                                inputChar = s_char[pos++];\
                                                                                                                                                if (inputChar == 'i'){\
                                                                                                                                                    inputChar = s_char[pos++];\
                                                                                                                                                    if (inputChar == 'l'){\
                                                                                                                                                        inputChar = s_char[pos++];\
                                                                                                                                                        if (inputChar == 'e'){\
                                                                                                                                                            match = 16;\
                                                                                                                                                        }\
                                                                                                                                                    }\
                                                                                                                                                }\
                                                                                                                                            }\
                                                                                                                                        }\
                                                                                                                                   }\
                                                                                                                                }\
                                                                                                                            }\
                                                                                                                        }\
                                                                                                                    }\
                                                                                                                }\
                                                                                                            }\
                                                                                                        }\
                                                                                                    }\
                                                                                                }\
                                                                                            }\
                                                                                        }\
                                                                                    }\
                                                                                }\
                                                                            }\
                                                                        }\
                                                                    }\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 'O'){\
            inputChar = s_char[pos++];\
            if (inputChar == 'P'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'L'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'D'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'a'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 't'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'a'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'b'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 'a'){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 's'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'e'){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 'F'){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 'i'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'l'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 'e'){\
                                                                    match = 17;\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 'P'){\
            inputChar = s_char[pos++];\
            if (inputChar == 'A'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'G'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'E'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'D'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'U'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == '6'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == '4'){\
                                        match = 18;\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }else if (inputChar == 'K'){\
                inputChar = s_char[pos++];\
                if (inputChar == 0x03){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0x04){\
                        match = 19;\
                    }\
                }else if (inputChar == 0x05){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0x06){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'P'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'K'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 0x07){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 0x08){\
                                        match = 20;\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 'R'){\
            inputChar = s_char[pos++];\
            if (inputChar == 'e'){\
                inputChar = s_char[pos++];\
                if (inputChar == 't'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'u'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'r'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'n'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == '-'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'P'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 'a'){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 't'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'h'){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == ':'){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == ' '){\
                                                            match = 21;\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == '['){\
            inputChar = s_char[pos++];\
            if (inputChar == 'W'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'i'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'n'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'd'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'o'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'w'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 's'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == ' '){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 'L'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'a'){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 't'){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 'i'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'n'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == ' '){\
                                                                    match = 22;\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 'f'){\
            inputChar = s_char[pos++];\
            if (inputChar == 't'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'y'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'p'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'M'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'S'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'N'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'V'){\
                                        match = 23;\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
            /*0x7c0x4b0xc30x740xe10xc80x530xa40x790xb90x010x1d0xfc0x4f0xdd0x13*/\
        } else if (inputChar == 0x7c){\
            inputChar = s_char[pos++];\
            if (inputChar == 0x4b){\
                inputChar = s_char[pos++];\
                if (inputChar == 0xc3){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0x74){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 0xe1){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 0xc8){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 0x53){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 0xa4){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 0x79){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 0xb9){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 0x01){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 0x1d){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 0xfc){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 0x4f){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 0xdd){\
                                                                    inputChar = s_char[pos++];\
                                                                    if (inputChar == 0x13){\
                                                                        match = 24;\
                                                                    }\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 0x7e){\
            inputChar = s_char[pos++];\
            if (inputChar == 'E'){\
                inputChar = s_char[pos++];\
                if (inputChar == 'S'){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 'D'){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 'w'){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 0xf6){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 0x85){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == '>'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 0xbf){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 'j'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 0xd2){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == 0x11){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 'E'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'a'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 's'){\
                                                                    inputChar = s_char[pos++];\
                                                                    if (inputChar == 'y'){\
                                                                        inputChar = s_char[pos++];\
                                                                        if (inputChar == ' '){\
                                                                            inputChar = s_char[pos++];\
                                                                            if (inputChar == 'S'){\
                                                                                inputChar = s_char[pos++];\
                                                                                if (inputChar == 't'){\
                                                                                    inputChar = s_char[pos++];\
                                                                                    if (inputChar == 'r'){\
                                                                                        inputChar = s_char[pos++];\
                                                                                        if (inputChar == 'e'){\
                                                                                            inputChar = s_char[pos++];\
                                                                                            if (inputChar == 'e'){\
                                                                                                inputChar = s_char[pos++];\
                                                                                                if (inputChar == 't'){\
                                                                                                    inputChar = s_char[pos++];\
                                                                                                    if (inputChar == ' '){\
                                                                                                        inputChar = s_char[pos++];\
                                                                                                        if(inputChar == 'D'){\
                                                                                                            inputChar = s_char[pos++];\
                                                                                                            if (inputChar == 'r'){\
                                                                                                                inputChar = s_char[pos++];\
                                                                                                                if (inputChar == 'a'){\
                                                                                                                    inputChar = s_char[pos++];\
                                                                                                                    if (inputChar == 'w'){\
                                                                                                                        match = 25;\
                                                                                                                    }\
                                                                                                                }\
                                                                                                            }\
                                                                                                        }\
                                                                                                    }\
                                                                                                }\
                                                                                            }\
                                                                                        }\
                                                                                    }\
                                                                                }\
                                                                            }\
                                                                        }\
                                                                    }\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if(inputChar == 0xbe){\
            inputChar = s_char[pos++];\
            if (inputChar == 0xba){\
                inputChar = s_char[pos++];\
                if (inputChar == 0xfe){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0xca){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 0x0f){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 'P'){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 'a'){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 'l'){\
                                        inputChar = s_char[pos++];\
                                        if (inputChar == 'm'){\
                                            inputChar = s_char[pos++];\
                                            if (inputChar == 'S'){\
                                                inputChar = s_char[pos++];\
                                                if (inputChar == 'G'){\
                                                    inputChar = s_char[pos++];\
                                                    if (inputChar == ' '){\
                                                        inputChar = s_char[pos++];\
                                                        if (inputChar == 'D'){\
                                                            inputChar = s_char[pos++];\
                                                            if (inputChar == 'a'){\
                                                                inputChar = s_char[pos++];\
                                                                if (inputChar == 't'){\
                                                                    inputChar = s_char[pos++];\
                                                                    if (inputChar == 'a'){\
                                                                        match = 26;\
                                                                    }\
                                                                }\
                                                            }\
                                                        }\
                                                    }\
                                                }\
                                            }\
                                        }\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }else if (inputChar == 0xd0){\
            inputChar = s_char[pos++];\
            if (inputChar == 0xcf){\
                inputChar = s_char[pos++];\
                if (inputChar == 0x11){\
                    inputChar = s_char[pos++];\
                    if (inputChar == 0xe0){\
                        inputChar = s_char[pos++];\
                        if (inputChar == 0xa1){\
                            inputChar = s_char[pos++];\
                            if (inputChar == 0xb1){\
                                inputChar = s_char[pos++];\
                                if (inputChar == 0x1a){\
                                    inputChar = s_char[pos++];\
                                    if (inputChar == 0xe1){\
                                        match = 27;\
                                    }\
                                }\
                            }\
                        }\
                    }\
                }\
            }\
        }\
    }

__global__ void match_corasick_spec(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {
    
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    int start = gbid * THREAD_BLOCK_SIZE + t_id ;
    int pos;
    int inputChar;
    int match[4] = {0,0,0,0};
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    
    unsigned char *s_char;
    
    if ( gbid > num_blocks_minus1 ){
        return ; // whole block is outside input stream
    }

    s_char = (unsigned char *)s_input;

    // read global data to shared memory
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }

    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();

    int bdy = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    
    int j = 0 ;

    MANUAL_EXPAND_4( SUBSEG_MATCH_NOTEX(j, match[j]) ; j++ ;)
    

    // write 4 results  match[0:3] to global d_match_result[0:input_size)
    // one thread block processes (BLOCKSIZE * 4) substrings
    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;

    if ( gbid < num_blocks_minus1 ){
        #pragma unroll
        for (int j = 0 ; j < 4 ; j++ ){
            d_match_result[start] = match[j];
            start += THREAD_BLOCK_SIZE;
        }
    }else{
        int j = 0 ;
        MANUAL_EXPAND_4( if (start>=input_size) return ; d_match_result[start] = match[j]; \
        j++ ; start += THREAD_BLOCK_SIZE ; )
    } 


}

// void construct_automaton(PFAC_handle_t handle,std::stack<std::pair<int,int>>& pfac_table, std::string& automaton){
//     if(pfac_table.empty()) return;
//     std::pair<int,int> p = pfac_table.top();
//     pfac_table.pop();
//     for(int i = 0; i < CHAR_SET; i++){
//         int new_state = handle->h_PFAC_table[p.second * CHAR_SET + i];
//         if(new_state != TRAP_STATE){
//             pfac_table.emplace(i,new_state);
//         }
//     }
//     automaton += "if(inputChar == " + std::to_string(p.first) + "){\n";
//     if(p.second < handle->initial_state){
//         automaton += "  match = " + std::to_string(p.second) + ";\n";
//     }
//     automaton += "  pos = pos + 1;\n"
//                  "  if(pos < bdy){\n"
//                  "      inputChar = s_char[pos];\n";
//     construct_automaton(handle,pfac_table,automaton);
//     automaton += "}\n}\n";
// }

void build_automaton(int ch,int state,int depth,PFAC_handle_t handle,std::string& automaton){
    
    if(depth){
        automaton += "else if(inputChar == " + std::to_string(ch) + "){\n";
        if(state < handle->initial_state){
            automaton += "  match = " + std::to_string(state) + ";\n";
        }
    }else {
        automaton += "if(inputChar == " + std::to_string(ch) + "){\n";
        if(state < handle->initial_state){
            automaton += "  match = " + std::to_string(state) + ";\n";
        }
    }
    std::vector<std::pair<int,int>> states;
    for(int i = 0; i < CHAR_SET; i++){
        int new_state = handle->h_PFAC_table[state * CHAR_SET + i];
        if(new_state != TRAP_STATE){
            states.emplace_back(i,new_state);
            // build_automaton(i,new_state,handle,automaton);
        }
    }
    if(!states.empty()){
        automaton += "  if(++pos < bdy){\n"
                     "      inputChar = s_char[pos];\n";
    }

    int depth_ = 0;
    for(auto p : states){
        build_automaton(p.first,p.second,depth_++,handle,automaton);
    }
    
    if(!states.empty()){
        automaton += "}\n";
    }
    automaton += "}\n";
   
}

void matchCorasickSpecWrapper(PFAC_handle_t handle,dim3 grid, dim3 block,const int* d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result){
    std::vector<std::string> vpatterns;
    
    for (int i = 0; i < handle->numOfPatterns; i++) {
        vpatterns.push_back(std::string(handle->rowPtr[i],handle->patternLen_table[i+1]));
    }

    std::string kernel;
    
    kernel += "naive_spec_manual\n";
    kernel += "__global__\n";
    kernel += "void match_naive_opt_spec_manual_corasick_jit(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {\n";
    kernel += "    const int THREAD_BLOCK_SIZE = " + std::to_string(THREAD_BLOCK_SIZE) + ";\n";
    kernel += "    const int EXTRA_SIZE_PER_TB = " + std::to_string(EXTRA_SIZE_PER_TB) + ";\n";
    kernel += "    int t_id = threadIdx.x;\n"
               "    int gbid = blockIdx.y * gridDim.x + blockIdx.x;\n"
               "    int start = gbid * THREAD_BLOCK_SIZE + t_id;\n"
               "    int inputChar;\n"
               "    int pos;\n"
               "    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];\n"
               "    unsigned char *s_char;\n"
               "    if ( gbid > num_blocks_minus1 ){\n"
               "        return ;\n"
               "    }\n"
               "    s_char = (unsigned char *)s_input;\n"
               "    if ( start < n_hat ){\n"
               "        s_input[t_id] = d_input_string[start];\n"
               "    }\n"
               "    start += THREAD_BLOCK_SIZE ;\n"
               "    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){\n"
               "        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];\n"
               "    }\n"
               "    __syncthreads();\n"
               "    int bdy = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );\n"
            //    "    int legal_size = (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4 > bdy ? bdy : (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4;\n"
               "    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;\n"
               "    for (int j = 0; j < 4; j++) {\n"
               "        int match = 0;\n"
               "        pos = t_id + j * THREAD_BLOCK_SIZE;\n"
               "        if (pos < bdy){\n"
               "            inputChar = s_char[pos];\n";
    // std::stack<std::pair<int,int>> states;
    std::string automaton;
    
    int depth_ = 0;
    for(int i = 0;i < CHAR_SET; i++){
        if(handle->h_PFAC_table[handle->initial_state * CHAR_SET + i] != TRAP_STATE){
            // states.emplace(i,handle->h_PFAC_table[handle->initial_state * CHAR_SET + i]);
            build_automaton(i,handle->h_PFAC_table[handle->initial_state * CHAR_SET + i],depth_++,handle,automaton);
        }
    }

    kernel += automaton;

    kernel += "}\n"
              "if (gbid < num_blocks_minus1) {\n"
              "    d_match_result[start] = match;\n"
              "    start += THREAD_BLOCK_SIZE;\n"
              "}else {\n"
              "     if (start >= input_size){\n"
              "         return;\n"
              "     }\n"
              "     d_match_result[start] = match;\n"
              "     start += THREAD_BLOCK_SIZE;\n"
              "}\n";

    kernel += "}\n}\n";
    // std::cout << kernel << std::endl;

    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(kernel);
    using jitify::reflection::type_of;

    RUN((program.kernel("match_naive_opt_spec_manual_corasick_jit")
       .instantiate()
       .configure(grid, block)
       .launch(d_input_string,input_size,n_hat,num_blocks_minus1,d_match_result)))

    // RUN((match_corasick_spec<<<grid,block>>>(d_input_string,input_size,n_hat,num_blocks_minus1,d_match_result)))
}