#include "hip/hip_runtime.h"
#include "../spec_match.hpp"
#include "ImpalaKernels.hpp"


#define MANUAL_EXPAND_2( X )   { X ; X ; }
#define MANUAL_EXPAND_4( X )   { MANUAL_EXPAND_2( MANUAL_EXPAND_2( X ) )  }


#define  SUBSEG_MATCH_NOTEX( j, match ) \
    pos = t_id + j * THREAD_BLOCK_SIZE ;\
    if ( pos < bdy_ ){ \
        if (pos < bdy - 9 + 1 && s_char[pos] == 0x14\
                              && s_char[pos + 1] == 'f'\
                              && s_char[pos + 2] == 't'\
                              && s_char[pos + 3] == 'y'\
                              && s_char[pos + 4] == 'p'\
                              && s_char[pos + 5] == 'i'\
                              && s_char[pos + 6] == 's'\
                              && s_char[pos + 7] == 'o'\
                              && s_char[pos + 8] == 'm'){\
                              match = 1;}\
        if (pos < bdy - 9 + 1 && s_char[pos] == 0x18\
                              && s_char[pos + 1] == 'f'\
                              && s_char[pos + 2] == 't'\
                              && s_char[pos + 3] == 'y'\
                              && s_char[pos + 4] == 'p'\
                              && s_char[pos + 5] == '3'\
                              && s_char[pos + 6] == 'g'\
                              && s_char[pos + 7] == 'p'\
                              && s_char[pos + 8] == '5'){\
                              match = 2;}\
        if (pos < bdy - 16 + 1 && s_char[pos] == 0x1a\
                               && s_char[pos + 1] == 'E'\
                               && s_char[pos + 2] == 0xdf\
                               && s_char[pos + 3] == 0xa3\
                               && s_char[pos + 4] == 0x93\
                               && s_char[pos + 5] == 'B'\
                               && s_char[pos + 6] == 0x82\
                               && s_char[pos + 7] == 0x88\
                               && s_char[pos + 8] == 'm'\
                               && s_char[pos + 9] == 'a'\
                               && s_char[pos + 10] == 't'\
                               && s_char[pos + 11] == 'r'\
                               && s_char[pos + 12] == 'o'\
                               && s_char[pos + 13] == 's'\
                               && s_char[pos + 14] == 'k'\
                               && s_char[pos + 15] == 'a'){\
                               match = 3;}\
        if (pos < bdy - 3 + 1 && s_char[pos] == 0x1f\
                              && s_char[pos + 1] == 0x8b\
                              && s_char[pos + 2] == 0x08){\
                              match = 4;}\
        if (pos < bdy - 4 + 1 && s_char[pos] == '%'\
                              && s_char[pos + 1] == 'P'\
                              && s_char[pos + 2] == 'D'\
                              && s_char[pos + 3] == 'F'){\
                              match = 5;}\
        if (pos < bdy - 6 + 1 && s_char[pos] == 0x37\
                              && s_char[pos + 1] == 0x7a\
                              && s_char[pos + 2] == 0xbc\
                              && s_char[pos + 3] == 0xaf\
                              && s_char[pos + 4] == 0x27\
                              && s_char[pos + 5] == 0x1c){\
                              match = 6;}\
        if (pos < bdy - 4 + 1 && s_char[pos] == '8'\
                              && s_char[pos + 1] == 'B'\
                              && s_char[pos + 2] == 'P'\
                              && s_char[pos + 3] == 'S'){\
                              match = 7;}\
        if (pos < bdy - 8 + 1 && s_char[pos] == '<'\
                              && s_char[pos + 1] == '!'\
                              && s_char[pos + 2] == 'd'\
                              && s_char[pos + 3] == 'o'\
                              && s_char[pos + 4] == 'c'\
                              && s_char[pos + 5] == 't'\
                              && s_char[pos + 6] == 'y'\
                              && s_char[pos + 7] == 'p'){\
                              match = 8;}\
        if (pos < bdy - 3 + 1 && s_char[pos] == 'C'\
                              && s_char[pos + 1] == 'W'\
                              && s_char[pos + 2] == 'S'){\
                              match = 9;}\
        if (pos < bdy - 3 + 1 && s_char[pos] == 'F'\
                              && s_char[pos + 1] == 'W'\
                              && s_char[pos + 2] == 'S'){\
                              match = 10;}\
        if (pos < bdy - 6 + 1 && s_char[pos] == 'G'\
                              && s_char[pos + 1] == 'I'\
                              && s_char[pos + 2 ] == 'F'\
                              && s_char[pos + 3] == '8'\
                              && s_char[pos + 4] == '7'\
                              && s_char[pos + 5] == 'a'){\
                              match = 11;}\
        if (pos < bdy - 6 + 1 && s_char[pos] == 'G'\
                              && s_char[pos + 1] == 'I'\
                              && s_char[pos + 2 ] == 'F'\
                              && s_char[pos + 3] == '8'\
                              && s_char[pos + 4] == '9'\
                              && s_char[pos + 5] == 'a'){\
                              match = 12;}\
        if (pos < bdy - 3 + 1 && s_char[pos] == 'I'\
                              && s_char[pos + 1] == ' '\
                              && s_char[pos + 2] == 'I'){\
                              match = 13;}\
        if (pos < bdy - 3 + 1 && s_char[pos] == 'I'\
                              && s_char[pos + 1] == 'D'\
                              && s_char[pos + 2] == '3'){\
                              match = 14;}\
        if (pos < bdy - 2 + 1 && s_char[pos] == 'M'\
                              && s_char[pos + 1] == 'Z'){\
                              match = 15;}\
        if (pos < bdy - 37 + 1 && s_char[pos] == 'M'\
                               && s_char[pos + 1] == 'i'\
                               && s_char[pos + 2] == 'c'\
                               && s_char[pos + 3] == 'r'\
                               && s_char[pos + 4] == 'o'\
                               && s_char[pos + 5] == 's'\
                               && s_char[pos + 6] == 'o'\
                               && s_char[pos + 7] == 'f'\
                               && s_char[pos + 8] == 't'\
                               && s_char[pos + 9] == ' '\
                               && s_char[pos + 10] == 'V'\
                               && s_char[pos + 11] == 'i'\
                               && s_char[pos + 12] == 's'\
                               && s_char[pos + 13] == 'u'\
                               && s_char[pos + 14] == 'a'\
                               && s_char[pos + 15] == 'l'\
                               && s_char[pos + 16] == ' '\
                               && s_char[pos + 17] == 'S'\
                               && s_char[pos + 18] == 't'\
                               && s_char[pos + 19] == 'u'\
                               && s_char[pos + 20] == 'd'\
                               && s_char[pos + 21] == 'i'\
                               && s_char[pos + 22] == 'o'\
                               && s_char[pos + 23] == ' '\
                               && s_char[pos + 24] == 'S'\
                               && s_char[pos + 25] == 'o'\
                               && s_char[pos + 26] == 'l'\
                               && s_char[pos + 27] == 'u'\
                               && s_char[pos + 28] == 't'\
                               && s_char[pos + 29] == 'i'\
                               && s_char[pos + 30] == 'o'\
                               && s_char[pos + 31] == 'n'\
                               && s_char[pos + 32] == ' '\
                               && s_char[pos + 33] == 'F'\
                               && s_char[pos + 34] == 'i'\
                               && s_char[pos + 35] == 'l'\
                               && s_char[pos + 36] == 'e'){\
                               match = 16;}\
        if (pos < bdy - 15 + 1 && s_char[pos] == 'O'\
                               && s_char[pos + 1] == 'P'\
                               && s_char[pos + 2] == 'L'\
                               && s_char[pos + 3] == 'D'\
                               && s_char[pos + 4] == 'a'\
                               && s_char[pos + 5] == 't'\
                               && s_char[pos + 6] == 'a'\
                               && s_char[pos + 7] == 'b'\
                               && s_char[pos + 8] == 'a'\
                               && s_char[pos + 9] == 's'\
                               && s_char[pos + 10] == 'e'\
                               && s_char[pos + 11] == 'F'\
                               && s_char[pos + 12] == 'i'\
                               && s_char[pos + 13] == 'l'\
                               && s_char[pos + 14] == 'e'){\
                               match = 17;}\
        if (pos < bdy - 8 + 1 && s_char[pos] == 'P'\
                              && s_char[pos + 1] == 'A'\
                              && s_char[pos + 2] == 'G'\
                              && s_char[pos + 3]  == 'E'\
                              && s_char[pos + 4] == 'D'\
                              && s_char[pos + 5] == 'U'\
                              && s_char[pos + 6] == '6'\
                              && s_char[pos + 7] == '4'){\
                              match = 18;}\
        if (pos < bdy - 4 + 1 && s_char[pos] == 'P'\
                              && s_char[pos + 1] == 'K'\
                              && s_char[pos + 2] == 0x03\
                              && s_char[pos + 3] == 0x04){\
                              match = 19;}\
        if (pos < bdy - 8 + 1 && s_char[pos] == 'P'\
                              && s_char[pos + 1] == 'K'\
                              && s_char[pos + 2] == 0x05\
                              && s_char[pos + 3] == 0x06\
                              && s_char[pos + 4] == 'P'\
                              && s_char[pos + 5] == 'K'\
                              && s_char[pos + 6] == 0x07\
                              && s_char[pos + 7] == 0x08){\
                              match = 20;}\
        if (pos < bdy - 13 + 1 && s_char[pos] == 'R'\
                               && s_char[pos + 1] == 'e'\
                               && s_char[pos + 2] == 't'\
                               && s_char[pos + 3] == 'u'\
                               && s_char[pos + 4] == 'r'\
                               && s_char[pos + 5] == 'n'\
                               && s_char[pos + 6] == '-'\
                               && s_char[pos + 7] == 'P'\
                               && s_char[pos + 8] == 'a'\
                               && s_char[pos + 9] == 't'\
                               && s_char[pos + 10] == 'h'\
                               && s_char[pos + 11] == ':'\
                               && s_char[pos + 12] == ' '){\
                               match = 21;}\
        if (pos < bdy - 15 + 1 && s_char[pos] == '['\
                               && s_char[pos + 1] == 'W'\
                               && s_char[pos + 2] == 'i'\
                               && s_char[pos + 3] == 'n'\
                               && s_char[pos + 4] == 'd'\
                               && s_char[pos + 5] == 'o'\
                               && s_char[pos + 6] == 'w'\
                               && s_char[pos + 7] == 's'\
                               && s_char[pos + 8] == ' '\
                               && s_char[pos + 9] == 'L'\
                               && s_char[pos + 10] == 'a'\
                               && s_char[pos + 11] == 't'\
                               && s_char[pos + 12] == 'i'\
                               && s_char[pos + 13] == 'n'\
                               && s_char[pos + 14] == ' '){\
                               match = 22;}\
        if (pos < bdy - 8 + 1 && s_char[pos] == 'f'\
                        && s_char[pos + 1] == 't'\
                        && s_char[pos + 2] == 'y'\
                        && s_char[pos + 3] == 'p'\
                        && s_char[pos + 4] == 'M'\
                        && s_char[pos + 5] == 'S'\
                        && s_char[pos + 6] == 'N'\
                        && s_char[pos + 7] == 'V'){\
                        match = 23;}\
        if (pos < bdy - 16 + 1 && s_char[pos] == 0x7c\
                               && s_char[pos + 1] == 0x4b\
                               && s_char[pos + 2] == 0xc3\
                               && s_char[pos + 3] == 0x74\
                               && s_char[pos + 4] == 0xe1\
                               && s_char[pos + 5] == 0xc8\
                               && s_char[pos + 6] == 0x53\
                               && s_char[pos + 7] == 0xa4\
                               && s_char[pos + 8] == 0x79\
                               && s_char[pos + 9] == 0xb9\
                               && s_char[pos + 10] == 0x01\
                               && s_char[pos + 11] == 0x1d\
                               && s_char[pos + 12] == 0xfc\
                               && s_char[pos + 13] == 0x4f\
                               && s_char[pos + 14] == 0xdd\
                               && s_char[pos + 15] == 0x13){\
                               match = 24;}\
        if (pos < bdy - 28 + 1 && s_char[pos] == 0x7e\
                               && s_char[pos + 1] == 'E'\
                               && s_char[pos + 2] == 'S'\
                               && s_char[pos + 3] == 'D'\
                               && s_char[pos + 4] == 'w'\
                               && s_char[pos + 5] == 0xf6\
                               && s_char[pos + 6] == 0x85\
                               && s_char[pos + 7] == '>'\
                               && s_char[pos + 8] == 0xbf\
                               && s_char[pos + 9] == 'j'\
                               && s_char[pos + 10] == 0xd2\
                               && s_char[pos + 11] == 0x11\
                               && s_char[pos + 12] == 'E'\
                               && s_char[pos + 13] == 'a'\
                               && s_char[pos + 14] == 's'\
                               && s_char[pos + 15] == 'y'\
                               && s_char[pos + 16] == ' '\
                               && s_char[pos + 17] == 'S'\
                               && s_char[pos + 18] == 't'\
                               && s_char[pos + 19] == 'r'\
                               && s_char[pos + 20] == 'e'\
                               && s_char[pos + 21] == 'e'\
                               && s_char[pos + 22] == 't'\
                               && s_char[pos + 23] == ' '\
                               && s_char[pos + 24] == 'D'\
                               && s_char[pos + 25] == 'r'\
                               && s_char[pos + 26] == 'a'\
                               && s_char[pos + 27] == 'w'){\
                               match = 25;}\
        if (pos < bdy - 16 + 1 && s_char[pos] == 0xbe\
                               && s_char[pos + 1] == 0xba\
                               && s_char[pos + 2] == 0xfe\
                               && s_char[pos + 3] == 0xca\
                               && s_char[pos + 4] == 0x0f\
                               && s_char[pos + 5] == 'P'\
                               && s_char[pos + 6] == 'a'\
                               && s_char[pos + 7] == 'l'\
                               && s_char[pos + 8] == 'm'\
                               && s_char[pos + 9] == 'S'\
                               && s_char[pos + 10] == 'G'\
                               && s_char[pos + 11] == ' '\
                               && s_char[pos + 12] == 'D'\
                               && s_char[pos + 13] == 'a'\
                               && s_char[pos + 14] == 't'\
                               && s_char[pos + 15] == 'a'){\
                               match = 26;}\
        if (pos < bdy - 8 + 1 && s_char[pos] == 0xd0\
                              && s_char[pos + 1] == 0xcf\
                              && s_char[pos + 2] == 0x11\
                              && s_char[pos + 3] == 0xe0\
                              && s_char[pos + 4] == 0xa1\
                              && s_char[pos + 5] == 0xb1\
                              && s_char[pos + 6] == 0x1a\
                              && s_char[pos + 7] == 0xe1){\
                              match = 27;}\
    }

__global__ void match_naive_opt_spec_manual(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {
    
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    int start = gbid * THREAD_BLOCK_SIZE + t_id ;
    int pos;
    int inputChar;
    int match[4] = {0,0,0,0};
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    
    unsigned char *s_char;
    
    if ( gbid > num_blocks_minus1 ){
        return ; // whole block is outside input stream
    }

    s_char = (unsigned char *)s_input;

    // read global data to shared memory
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }

    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();

    int bdy_ = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    int bdy = (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4 > bdy_ ? bdy_ : (EXTRA_SIZE_PER_TB + THREAD_BLOCK_SIZE) * 4;


    int j = 0 ;

    MANUAL_EXPAND_4( SUBSEG_MATCH_NOTEX(j, match[j]) ; j++ ;)
    

    // write 4 results  match[0:3] to global d_match_result[0:input_size)
    // one thread block processes (BLOCKSIZE * 4) substrings
    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;

    if ( gbid < num_blocks_minus1 ){
        #pragma unroll
        for (int j = 0 ; j < 4 ; j++ ){
            d_match_result[start] = match[j];
            start += THREAD_BLOCK_SIZE;
        }
    }else{
        int j = 0 ;
        MANUAL_EXPAND_4( if (start>=input_size) return ; d_match_result[start] = match[j]; \
        j++ ; start += THREAD_BLOCK_SIZE ; )
    } 


}

void matchNaiveSpecManualOptWrapper(dim3 grid, dim3 block,const int* d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result){
    RUN((match_naive_opt_spec_manual<<<grid,block>>>(d_input_string,input_size,n_hat,num_blocks_minus1,d_match_result)))
}