#include "hip/hip_runtime.h"

// Patterns:
// AB
// ABE
// ABG
// BEDE
// ED

void match_naive_opt_spec_manual_corasick_jit(const int* __restrict__ d_input_string, int input_size, int n_hat, int num_blocks_minus1, int* d_match_result) {
    const int THREAD_BLOCK_SIZE = 256;
    const int EXTRA_SIZE_PER_TB = 128;
    int t_id = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;
    int start = gbid * THREAD_BLOCK_SIZE + t_id;
    int inputChar;
    int pos;
    __shared__ int s_input[ THREAD_BLOCK_SIZE + EXTRA_SIZE_PER_TB];
    unsigned char *s_char;
    if ( gbid > num_blocks_minus1 ){
        return ;
    }
    s_char = (unsigned char *)s_input;
    if ( start < n_hat ){
        s_input[t_id] = d_input_string[start];
    }
    start += THREAD_BLOCK_SIZE ;
    if ( (start < n_hat) && (t_id < EXTRA_SIZE_PER_TB) ){
        s_input[t_id + THREAD_BLOCK_SIZE] = d_input_string[start];
    }
    __syncthreads();
    int bdy = input_size - ( gbid * THREAD_BLOCK_SIZE * 4 );
    start = gbid * (THREAD_BLOCK_SIZE * 4) + t_id ;
    for (int j = 0; j < 4; j++) {
        int match = 0;
        pos = t_id + j * THREAD_BLOCK_SIZE;
        if (pos < bdy){
            inputChar = s_char[pos];
            switch(inputChar){
              case 65:
                if(++pos < bdy){
                  inputChar = s_char[pos];
                  switch(inputChar) {
                    case 66:
                      match = 1;
                      if(++pos < bdy){
                        inputChar = s_char[pos];
                        switch(inputChar) {
                          case 69:
                            match = 2;
                            break;
                          case 71:
                            match = 3;
                            break;
                        }
                      }
                      break; 
                    }
                }
                break;
              case 66:
                if(++pos < bdy){
                  inputChar = s_char[pos];
                  switch(inputChar) {
                    case 69:
                      if(++pos < bdy){
                        inputChar = s_char[pos];
                        switch(inputChar) {
                          case 68:
                            if(++pos < bdy){
                              inputChar = s_char[pos];
                              switch(inputChar) {
                                case 69:
                                match = 4;
                                break;
                              }
                            }
                            break;
                        }
                      }
                      break;
                    }
                }
                break;
              case 69:
                  if(++pos < bdy){
                    inputChar = s_char[pos];
                    switch(inputChar) {
                      case 68:
                      match = 5;
                      break;
                    }
                  }
                  break;
            }
}
if (gbid < num_blocks_minus1) {
    d_match_result[start] = match;
    start += THREAD_BLOCK_SIZE;
}else {
     if (start >= input_size){
         return;
     }
     d_match_result[start] = match;
     start += THREAD_BLOCK_SIZE;
}
}
}