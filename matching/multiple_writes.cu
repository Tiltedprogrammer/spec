
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

__global__ void write(int* a){

    a[0] = blockIdx.x + threadIdx.x;
}

int main(int argc, char** argv){

    int a[] = {0};
    int* d_a;
    hipMalloc(&d_a, 1 * sizeof(int));
    hipMemcpy(d_a,a,1 * sizeof(int),hipMemcpyHostToDevice);
    write<<<2,1>>>(d_a);
    hipMemcpy(a,d_a,1*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::cout << a[0] << "\n"; 


}
