#include "hip/hip_runtime.h"
#include "kernels.hpp"
#include "utils.hpp"
//CPU timer
#include "../include/timer.h"

#include <iostream>
#include <fstream>


typedef struct Template{

    char array[32] = {0};
    int size;

}Template;

#define block_size BLOCK_SIZE

__device__ long threadId(){
    
    long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;
    long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;
    return threadId;

}

__global__ void match(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {


    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}

__global__ void match_shared(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x < pattern_size) {
        spattern[threadIdx.x] = pattern[threadIdx.x];
    }
    __syncthreads();

    if(t_id < text_size){
        
        int matched = 1;
        result_buf[t_id] = 0;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != spattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     
    }
}

__global__ void match_multy(const char* __restrict__ patterns, int* p_sizes, int p_number,int max_len, const char* __restrict__ text, long text_size, int* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;

        if(t_id < text_size - max_len + 1){
        
            // result_buf[t_id] = 0;

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - p_sizes[i] + 1) {
                    int size = p_sizes[i];
                    for(int j = 0; j < size; j++) {
                    
                        if((int)text[t_id + j] != (int)patterns[j+p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += size;
            }
        }else {
                for(int i = 0; i < p_number; i++) {//for each pattern
                    matched = 1;
                    int size = p_sizes[i];
                    if(t_id < text_size - size + 1) {
                        for(int j = 0; j < size; j++) {
                        
                            if(text[t_id + j] != patterns[j+p_offset]) {
                                matched = -1;
                                break;
                            }
                        } 
                    
                        if(matched == 1) {
                            match_result = i+1; // 0 stands for missmatch
                        }
                    }
                    p_offset += size;                
            }
        }
        result_buf[t_id] = match_result;             
    }
}

//maximum 64 patterns with 8192 total length
__constant__ char mpatterns[1024]; //40 * 1024
__constant__ int cp_sizes[2248];

__global__ void match_multy_const(int p_number, int max_len, char* text, long text_size, int* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;
        // result_buf[t_id] = 0;

        if(t_id < text_size - max_len + 1){

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - cp_sizes[i] + 1){
                    for(int j = 0; j < cp_sizes[i]; j++) {
                
                        if(text[t_id + j] != mpatterns[j + p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += cp_sizes[i];
            }
        }else {
            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                if(t_id < text_size - cp_sizes[i] + 1){
                    for(int j = 0; j < cp_sizes[i]; j++) {
                
                        if(text[t_id + j] != mpatterns[j + p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                }
                p_offset += cp_sizes[i];
            }
        }
        result_buf[t_id] = match_result;             
    }
}


__global__ void match_multy_const_sizes(const char* __restrict__ patterns, int p_number,int max_len, const char* __restrict__ text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;

        if(t_id < text_size - max_len + 1){
        
            // result_buf[t_id] = 0;

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - p_sizes[i] + 1) {
                    for(int j = 0; j < cp_sizes[i]; j++) {
                    
                        if(text[t_id + j] != patterns[j+p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += cp_sizes[i];
            }
        }else {
                for(int i = 0; i < p_number; i++) {//for each pattern
                    matched = 1;
                    if(t_id < text_size - cp_sizes[i] + 1) {
                        for(int j = 0; j < cp_sizes[i]; j++) {
                        
                            if(text[t_id + j] != patterns[j+p_offset]) {
                                matched = -1;
                                break;
                            }
                        } 
                    
                        if(matched == 1) {
                            match_result = i+1; // 0 stands for missmatch
                        }
                    }
                    p_offset += cp_sizes[i];                
            }
        }
        result_buf[t_id] = match_result;             
    }
}

void multipattern_match_const_wrapper(std::vector<std::string> vpatterns, std::string file_name,size_t size, size_t offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];     
    }
    
    int loffset = 0;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cp_sizes), sizes, vpatterns.size() * sizeof(int)); 
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpyToSymbol(HIP_SYMBOL(mpatterns),vpatterns[i].c_str(),vpatterns[i].size(),loffset);
        loffset += sizes[i];
    }
    
    int* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    //nochunk only
    dim3 block(block_size);

    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    RUN((match_multy_const<<<grid,block>>>(vpatterns.size(),max,dtextptr,text_size,dresult_buf)))
    hipDeviceSynchronize();
    
    delete[](sizes);
    
    if(res_to_vec){
        int * h_match_result = new int[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,h_match_result[i]));
            }
        }

        delete[] (h_match_result);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}


void multipattern_match_const_unroll_wrapper(std::vector<std::string> vpatterns, std::string file_name,size_t size, size_t offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];     
    }
    
    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    int* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    //nochunk only
    dim3 block(block_size);
    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    std::string kernel;
    
    kernel += "multiple_match_const_unroll\n";
    kernel += "__device__ long threadId(){\n"
              "long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;\n"
              "long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;\n"
              "return threadId;\n}\n";

    kernel += "__constant__ char mpatterns[1024*40];\n";
    kernel += "__global__\n";
    kernel += "void multiple_match_const_unroll(char* text, long text_size, int* result_buf) {\n";
    kernel += "    long t_id = threadId();\n"

              "    if(t_id < text_size){\n"
            //   "       int p_offset = 0;\n"
              "       int matched = 1;\n"
              "       int match_result = 0;\n"
        // result_buf[t_id] = 0;

              "       if(t_id < text_size -" + std::to_string(max - 1) + "){\n";

    std::string cycle;
    std::string cycle_elem;
    int i = 0;
    int p_offset = 0;
    
    for(auto &vp : vpatterns){
        cycle += "matched = 1;\n";
        cycle += "      if(text[t_id + " + std::to_string(0) + "] != mpatterns[" + std::to_string(p_offset) + "]){\n" //+ std::to_string((int)vp[0]) + "){\n"
                     "      matched = -1;\n"
                     "  }\n"; 
        for(int j = 1; j < vp.size(); j++){
            cycle += "  else if(text[t_id + " + std::to_string(j) + "] != mpatterns[" + std::to_string(p_offset + j) + "]){\n" //+ std::to_string((int)vp[j]) + "){\n"
                     "      matched = -1;\n"
                     "  }\n";
        }
        cycle += "      if(matched == 1) {\n"
                 "          match_result = " + std::to_string(i + 1) + ";\n"
                 "      }\n";
        p_offset += vp.size();
        i++;                
    }
    cycle +="   }else{\n";

    i = 0;
    p_offset = 0;
    for(auto &vp : vpatterns){
        cycle += "matched = 1;\n";
        cycle += "   if(t_id < text_size - " + std::to_string(vp.size() - 1) + "){\n";
        cycle += "      if(text[t_id + " + std::to_string(0) + "] != mpatterns[" + std::to_string(p_offset) + "]){\n" //+ std::to_string((int)vp[0]) + "){\n"
                     "      matched = -1;\n"
                     "  }\n"; 
        for(int j = 1; j < vp.size(); j++){
            cycle += "  else if(text[t_id + " + std::to_string(j) + "] != mpatterns[" + std::to_string(p_offset + j) + "]){\n" //+ std::to_string((int)vp[j]) + "){\n"
                     "      matched = -1;\n"
                     "  }\n";
        }
        cycle += "      if(matched == 1) {\n"
                 "          match_result = " + std::to_string(i + 1) + ";\n"
                 "      }\n}\n";
        p_offset += vp.size();
        i++;                
    }
    cycle +="   }\n";

    kernel += cycle;
    kernel += "result_buf[t_id] = match_result;\n}\n}";

    //dump kernel

    // std::ofstream dump_file;
    
    // dump_file.open("const_kernel_dump");
    // dump_file << kernel;
    // dump_file.close();

    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(kernel);
    using jitify::reflection::type_of;

    auto kernel_instance = program.kernel("multiple_match_const_unroll").instantiate();

    char * cvpatterns = new char[len];
    
    int loffset = 0;

    for(auto &vp : vpatterns){
       for(int j = loffset, i = 0; j < loffset + vp.size(); j++, i++){
           cvpatterns[j] = vp[i];
        }
        loffset += vp.size();
        
    }

    // for(int i = 0; i < vpatterns.size(); i++){
    hipMemcpyHtoD(kernel_instance.get_constant_ptr("mpatterns"), cvpatterns, len);
    //    loffset += sizes[i];
    // }
    CudaCheckError();

    delete[](cvpatterns);


    RUN(kernel_instance.configure(grid, block)
       .launch(dtextptr,text_size,dresult_buf))
    hipDeviceSynchronize();
    
    delete[](sizes);
    
    if(res_to_vec){
        int * h_match_result = new int[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,h_match_result[i]));
            }
        }

        delete[] (h_match_result);
    }

    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}



__global__ void match_multy_shared(char* patterns, int* p_sizes, int p_number,int p_len, char* text, long text_size, char* result_buf){
    
    //assume that blockSize >= p_len
    extern __shared__ char sPatterns[];
    if (threadIdx.x < p_len){
        sPatterns[threadIdx.x] = patterns[threadIdx.x];
    }
    __syncthreads();

    long t_id = threadId();

    if(t_id < text_size){

        int p_offset = 0;
        int match_result = 0;
        int matched = 1;

        // result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1){
                for(int j = 0; j < p_sizes[i]; j++) {
            
                    if(text[t_id + j] != sPatterns[j + p_offset]) {
                        matched = -1;
                        break;
                    }
                } 
            
                if(matched == 1) {
                    match_result = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }
        result_buf[t_id] = match_result;             
    }


}

__global__ void match_chunk_shared(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x == 0) {
        for(int i = 0; i < pattern_size; i++){
            spattern[i] = pattern[i];
        }
    }
    __syncthreads();
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;
            if(i < text_size - left_bound - pattern_size + 1){

                for(int j = 0; j < pattern_size; j++) {

                    if(text[left_bound + i + j] != spattern[j]) {
                        matched = -1;
                        break;
                    }
                }

                if(matched == 1) {
                    result_buf[left_bound + i] = 1;
                }
            }
        }
                             
    }
}

__global__ void match_chunk(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            int matched = 1;
            
            if(left_bound + i + pattern_size - 1 >= text_size){
                matched = -1;
            } else {
                for(int j = 0; j < pattern_size ; j++) {

                    if(text[left_bound + i + j] != pattern[j]) {
                        matched = -1;
                        break;
                    }
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void kmp_chunk(int* prefix_table, char* pattern,int pattern_size,char* text, long text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}


__global__ void kmp_nochunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}

__constant__ char c_pattern[512]; //might be as fast as registers, but not in this case =)

__global__ void match_chunk_const(int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            int matched = 1;

            if(left_bound + i + pattern_size - 1 >= text_size){
                matched = -1;
            } else {
                for(int j = 0; j < pattern_size ; j++) {

                    if(text[left_bound + i + j] != c_pattern[j]) {
                        matched = -1;
                        break;
                    }
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_const(int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        if(t_id < text_size - pattern_size + 1){

            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != c_pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}


__constant__ int c_prefix[128];

__global__ void kmp_chunk_const(int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){

        char curChar = text[i];

        while(ams > 0 && c_pattern[ams] != curChar){
            ams = c_prefix[ams-1];
        }

        if(text[i] == c_pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = c_prefix[ams-1];
        }


    }
}

void multipattern_match_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    int* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    //nochunk only
    dim3 block(block_size);

    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    RUN((match_multy<<<grid,block>>>(dpatterns,dsizes,vpatterns.size(),max,dtextptr,text_size,dresult_buf)))
    hipDeviceSynchronize();

    
    delete[](sizes);

    if(res_to_vec){

        int * h_match_result = new int[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes); 
}


void multipattern_match_const_sizes_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cp_sizes), sizes, vpatterns.size() * sizeof(int)); 

    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);

    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    match_multy_const_sizes<<<grid,block>>>(dpatterns,vpatterns.size(),max,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    
    delete[](sizes);
        
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    if(res_to_vec){

        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns); 
}



void multipattern_match_shared_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_multy_shared<<<grid,block,len * sizeof(char)>>>(dpatterns,dsizes,vpatterns.size(),len,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    
    delete[](sizes);
    
    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}

void prefix(const char* pattern, int pattern_size, int* prefix_table){
    
    prefix_table[0] = 0;
    
    for (int i = 1; i < pattern_size; ++i) {
		
        int j = prefix_table[i-1];
		
        while (j > 0 && pattern[i] != pattern[j]){
			
            j = prefix_table[j-1];
        
        }
		
        if (pattern[i] == pattern[j])  ++j;
		
        prefix_table[i] = j;
	}
}

void match_naive_constant_wrapper(std::string pattern, std::string subject_string_filename, int nochunk, long size, long offset,std::vector<std::pair<int,int>> &res, int res_to_vec){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();

    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),pattern.c_str(),pattern_size);
    
    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    hipMemset(dresult_buf,0,text_size);
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        RUN((match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf)))
        hipDeviceSynchronize();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        RUN((match_chunk_const<<<grid,block>>>(pattern_size,chunk,dtextptr,text_size,dresult_buf)))
        hipDeviceSynchronize();
    }
    
    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(char),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);

}



void match_naive_wrapper(std::string pattern, std::string subject_string_filename, int nochunk, long size, long offset,std::vector<std::pair<int,int>> &res, int res_to_vec){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    hipMemset(dresult_buf,0,text_size);

    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;

    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        RUN((match<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf)))
        hipDeviceSynchronize();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        RUN((match_chunk<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf)))
        hipDeviceSynchronize();
        CudaCheckError();
    }
    
    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(char),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);

}

// Texture memory kernel
texture<int,1,hipReadModeElementType> patterns_tex; 

__global__ void match_tex(int* p_sizes, int p_number, char* text, long text_size, char* result_buf){
    
    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;
        
        // result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1) {
                for(int j = 0; j < p_sizes[i]; j++) {
                
                    if(text[t_id + j] != tex1Dfetch(patterns_tex,j+p_offset)) {
                        matched = -1;
                        break;
                    }
                } 
            
                if(matched == 1) {
                    match_result = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }
        result_buf[t_id] = match_result;             
    }
}

void multipattern_match_texture_wrapper(std::vector<std::string> vpatterns, std::string file_name, long size, long offset,int verbose){ //nochunk == 0 => nochunk

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    int* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(int));
    
    for(int i = 0; i < vpatterns.size(); i++){
        std::vector<int> pattern_int;
        // int* pattern_int = new int[vpatterns[i].size()];
        // int j = 0;
        for(auto ch: vpatterns[i]){
            // pattern_int[j]=(int)ch;
            pattern_int.push_back((int)ch);
            // j++;
        }
        hipMemcpy((void*)(dpatterns + loffset),&pattern_int[0],vpatterns[i].size()*sizeof(int),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }


    //tex mem
    textureReference *texRefTable ;
    hipGetTextureReference( (const struct textureReference**)&texRefTable, &patterns_tex);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
        // set texture parameters
    patterns_tex.addressMode[0] = hipAddressModeClamp;
    patterns_tex.addressMode[1] = hipAddressModeClamp;
    patterns_tex.filterMode     = hipFilterModePoint;
    patterns_tex.normalized     = 0;
        
    size_t offset_t ;
    hipBindTexture( &offset_t, (const struct textureReference*) texRefTable,
            (const void*) dpatterns, (const struct hipChannelFormatDesc*) &channelDesc, 
            len * sizeof(int));
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_tex<<<grid,block>>>(dsizes,vpatterns.size(),dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    
    delete[](sizes);
    
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    // if(res_to_vec){

    //     std::cout << "res to vec" << std::endl;

    //     char * h_match_result = new char[text_size];
    //     hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
    //     for (int i = 0; i < text_size; i++){
    //         if (h_match_result[i]){
    //             res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
    //         }
    //     }
    // }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);

    //unbind
    hipUnbindTexture(patterns_tex);
}



// void match_naive_shared(std::string pattern, std::string subject_string_filename, long nochunk,long size, int offset,int verbose){ //nochunk == 0 => nochunk

//     char* dtextptr;
//     long text_size;
//     if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
//         std::cout << "error opening file" << "\n";
//         return;
//     }

//     auto pattern_size = pattern.size();
//     char *dpattern;
//     hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
//     hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

//     char* dresult_buf;
//     hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
//     int chunk = 256;

//     dim3 block(block_size);
//     long grid_size;
//     long gsqrt;
//     am::timer time;
//     std::cout << "running ..." << "\n";
//     time.start();
    
//     if(nochunk){
//         grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         match_shared<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
//         hipDeviceSynchronize();
//         time.stop();
//     } else{
//         grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         match_chunk_shared<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
//         hipDeviceSynchronize();
//         time.stop();
//     }

//     std::cout << "running time " << time.milliseconds() << " ms" << std::endl;    

//     if(verbose){
//         write_from_device(&dresult_buf,text_size);
//     }

//     hipFree(dresult_buf);
//     hipFree(dtextptr);
//     hipFree(dpattern);
      
// }


void match_kmp(std::string pattern, std::string subject_string_filename, int constant, long size, long offset, int verbose, std::vector<std::pair<int,int>> &res, int res_to_vec){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    size_t text_size;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int* prefix_table = new int[pattern_size];
    
    prefix(pattern.c_str(),pattern_size,prefix_table);
    
    int* dprefix_table;

    hipMalloc((void**)&dprefix_table, pattern_size * sizeof(int));
    hipMemcpy((void*)dprefix_table,prefix_table,pattern_size * sizeof(int),hipMemcpyHostToDevice); 

    char* dresult_buf;
    //think about data transfer;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    hipMemset(dresult_buf,0,text_size);
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;

    if(constant){
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),pattern.c_str(),pattern.size());
        hipMemcpyToSymbol(HIP_SYMBOL(c_prefix),prefix_table,pattern.size() * sizeof(int));

        RUN((kmp_chunk_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf,chunk)))
        hipDeviceSynchronize();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        RUN((kmp_chunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk)))
        hipDeviceSynchronize();
    }

    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size * sizeof(char),hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
    }

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    delete[](prefix_table);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
    hipFree(dprefix_table); 
}
