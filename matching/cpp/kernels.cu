#include "hip/hip_runtime.h"

#include "kernels.hpp"
#include "utils.hpp"
//CPU timer
#include "../include/timer.h"

#include <iostream>


typedef struct Template{

    char array[32] = {0};
    int size;

}Template;

#define block_size BLOCK_SIZE

__device__ long threadId(){
    
    long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;
    long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;
    return threadId;

}

__global__ void match(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {


    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = 0;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}

__global__ void match_shared(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x < pattern_size) {
        spattern[threadIdx.x] = pattern[threadIdx.x];
    }
    __syncthreads();

    if(t_id < text_size){
        
        int matched = 1;
        result_buf[t_id] = 0;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != spattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     
    }
}

__global__ void match_multy(const char* __restrict__ patterns, int* p_sizes, int p_number,int max_len, const char* __restrict__ text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;

        if(t_id < text_size - max_len + 1){
        
            // result_buf[t_id] = 0;

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - p_sizes[i] + 1) {
                    for(int j = 0; j < p_sizes[i]; j++) {
                    
                        if(text[t_id + j] != patterns[j+p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += p_sizes[i];
            }
        }else {
                for(int i = 0; i < p_number; i++) {//for each pattern
                    matched = 1;
                    if(t_id < text_size - p_sizes[i] + 1) {
                        for(int j = 0; j < p_sizes[i]; j++) {
                        
                            if(text[t_id + j] != patterns[j+p_offset]) {
                                matched = -1;
                                break;
                            }
                        } 
                    
                        if(matched == 1) {
                            match_result = i+1; // 0 stands for missmatch
                        }
                    }
                    p_offset += p_sizes[i];                
            }
        }
        result_buf[t_id] = match_result;             
    }
}

//maximum 64 patterns with 8192 total length
__constant__ char mpatterns[128*64];
__constant__ int cp_sizes[64];

__global__ void match_multy_const(int p_number, int max_len, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;
        // result_buf[t_id] = 0;

        if(t_id < text_size - max_len + 1){

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - cp_sizes[i] + 1){
                    for(int j = 0; j < cp_sizes[i]; j++) {
                
                        if(text[t_id + j] != mpatterns[j + p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += cp_sizes[i];
            }
        }else {
            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                if(t_id < text_size - cp_sizes[i] + 1){
                    for(int j = 0; j < cp_sizes[i]; j++) {
                
                        if(text[t_id + j] != mpatterns[j + p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                }
                p_offset += cp_sizes[i];
            }
        }
        result_buf[t_id] = match_result;             
    }
}


__global__ void match_multy_const_sizes(const char* __restrict__ patterns, int p_number,int max_len, const char* __restrict__ text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;

        if(t_id < text_size - max_len + 1){
        
            // result_buf[t_id] = 0;

            for(int i = 0; i < p_number; i++) {//for each pattern
                matched = 1;
                // if(t_id < text_size - p_sizes[i] + 1) {
                    for(int j = 0; j < cp_sizes[i]; j++) {
                    
                        if(text[t_id + j] != patterns[j+p_offset]) {
                            matched = -1;
                            break;
                        }
                    } 
                
                    if(matched == 1) {
                        match_result = i+1; // 0 stands for missmatch
                    }
                // }
                p_offset += cp_sizes[i];
            }
        }else {
                for(int i = 0; i < p_number; i++) {//for each pattern
                    matched = 1;
                    if(t_id < text_size - cp_sizes[i] + 1) {
                        for(int j = 0; j < cp_sizes[i]; j++) {
                        
                            if(text[t_id + j] != patterns[j+p_offset]) {
                                matched = -1;
                                break;
                            }
                        } 
                    
                        if(matched == 1) {
                            match_result = i+1; // 0 stands for missmatch
                        }
                    }
                    p_offset += cp_sizes[i];                
            }
        }
        result_buf[t_id] = match_result;             
    }
}

void multipattern_match_const_wrapper(std::vector<std::string> vpatterns, std::string file_name,size_t size, size_t offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];     
    }
    
    int loffset = 0;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cp_sizes), sizes, vpatterns.size() * sizeof(int)); 
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpyToSymbol(HIP_SYMBOL(mpatterns),vpatterns[i].c_str(),vpatterns[i].size(),loffset);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    // grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    // gsqrt = (int)sqrt(grid_size) + 1;
    // dim3 grid(gsqrt,gsqrt);
    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    match_multy_const<<<grid,block>>>(vpatterns.size(),max,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }

        delete[] (h_match_result);
    }
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    // hipFree(dpatterns);
    // hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
}

__global__ void match_multy_shared(char* patterns, int* p_sizes, int p_number,int p_len, char* text, long text_size, char* result_buf){
    
    //assume that blockSize >= p_len
    extern __shared__ char sPatterns[];
    if (threadIdx.x < p_len){
        sPatterns[threadIdx.x] = patterns[threadIdx.x];
    }
    __syncthreads();

    long t_id = threadId();

    if(t_id < text_size){

        int p_offset = 0;
        int match_result = 0;
        int matched = 1;

        // result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1){
                for(int j = 0; j < p_sizes[i]; j++) {
            
                    if(text[t_id + j] != sPatterns[j + p_offset]) {
                        matched = -1;
                        break;
                    }
                } 
            
                if(matched == 1) {
                    match_result = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }
        result_buf[t_id] = match_result;             
    }


}

__global__ void match_chunk_shared(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x == 0) {
        for(int i = 0; i < pattern_size; i++){
            spattern[i] = pattern[i];
        }
    }
    __syncthreads();
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;
            if(i < text_size - left_bound - pattern_size + 1){

                for(int j = 0; j < pattern_size; j++) {

                    if(text[left_bound + i + j] != spattern[j]) {
                        matched = -1;
                        break;
                    }
                }

                if(matched == 1) {
                    result_buf[left_bound + i] = 1;
                }
            }
        }
                             
    }
}

__global__ void match_chunk(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;
            
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void kmp_chunk(int* prefix_table, char* pattern,int pattern_size,char* text, long text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}


__global__ void kmp_nochunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}

__constant__ char c_pattern[128*64]; //might be as fast as registers, but not in this case =)

__global__ void match_chunk_const(int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;

            for(int j = 0; j < pattern_size; j++) {
                
                if(text[left_bound + i + j] != c_pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_const(int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = 0;
        if(t_id < text_size - pattern_size + 1){

            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != c_pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}


__constant__ int c_prefix[128*64];

__global__ void kmp_chunk_const(int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && c_pattern[ams] != text[i]){
            ams = c_prefix[ams-1];
        }

        if(text[i] == c_pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = c_prefix[ams-1];
        }


    }
}

void multipattern_match_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    // grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    // gsqrt = (int)sqrt(grid_size) + 1;
    // dim3 grid(gsqrt,gsqrt);
    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    match_multy<<<grid,block>>>(dpatterns,dsizes,vpatterns.size(),max,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    if(res_to_vec){

        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}


void multipattern_match_const_sizes_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    int max = vpatterns[0].size();
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        max = sizes[i] > max ? sizes[i] : max;
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    // hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    // hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMemcpyToSymbol(HIP_SYMBOL(cp_sizes), sizes, vpatterns.size() * sizeof(int)); 

    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    // grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    // gsqrt = (int)sqrt(grid_size) + 1;
    // dim3 grid(gsqrt,gsqrt);
    int num_blocks = (text_size + block.x - 1) / block.x;
    int p = num_blocks / 32768;
    dim3 grid;
    if(p > 0) {
        grid.x = 32768;
        grid.y = p + 1;
    } else {
        grid.x = num_blocks;
    }
    match_multy_const_sizes<<<grid,block>>>(dpatterns,vpatterns.size(),max,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    if(res_to_vec){

        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    // hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}



void multipattern_match_shared_wrapper(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose,std::vector<std::pair<int,int>> &res ,int res_to_vec){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_multy_shared<<<grid,block,len * sizeof(char)>>>(dpatterns,dsizes,vpatterns.size(),len,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(res_to_vec){
        char * h_match_result = new char[text_size];
        hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
        for (int i = 0; i < text_size; i++){
            if (h_match_result[i]){
                res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
            }
        }
        delete[] (h_match_result);
    }
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}

// void prefix(const char* pattern, int pattern_size, int* prefix_table){
    
//     prefix_table[0] = 0;
    
//     for (int i = 1; i < pattern_size; ++i) {
		
//         int j = prefix_table[i-1];
		
//         while (j > 0 && pattern[i] != pattern[j]){
			
//             j = prefix_table[j-1];
        
//         }
		
//         if (pattern[i] == pattern[j])  ++j;
		
//         prefix_table[i] = j;
// 	}
// }

void match_naive_wrapper(std::string pattern, std::string subject_string_filename, int nochunk, long size, long offset,int verbose){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);

}

// Texture memory kernel
texture<int,1,hipReadModeElementType> patterns_tex; 

__global__ void match_tex(int* p_sizes, int p_number, char* text, long text_size, char* result_buf){
    
    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        int match_result = 0;
        
        // result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1) {
                for(int j = 0; j < p_sizes[i]; j++) {
                
                    if(text[t_id + j] != tex1Dfetch(patterns_tex,j+p_offset)) {
                        matched = -1;
                        break;
                    }
                } 
            
                if(matched == 1) {
                    match_result = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }
        result_buf[t_id] = match_result;             
    }
}

void multipattern_match_texture_wrapper(std::vector<std::string> vpatterns, std::string file_name, long size, long offset,int verbose){ //nochunk == 0 => nochunk

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    int* dpatterns;
    int* dsizes;

    char* dtextptr;
    size_t text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(int));
    
    for(int i = 0; i < vpatterns.size(); i++){
        std::vector<int> pattern_int;
        // int* pattern_int = new int[vpatterns[i].size()];
        // int j = 0;
        for(auto ch: vpatterns[i]){
            // pattern_int[j]=(int)ch;
            pattern_int.push_back((int)ch);
            // j++;
        }
        hipMemcpy((void*)(dpatterns + loffset),&pattern_int[0],vpatterns[i].size()*sizeof(int),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }


    //tex mem
    textureReference *texRefTable ;
    hipGetTextureReference( (const struct textureReference**)&texRefTable, &patterns_tex);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
        // set texture parameters
    patterns_tex.addressMode[0] = hipAddressModeClamp;
    patterns_tex.addressMode[1] = hipAddressModeClamp;
    patterns_tex.filterMode     = hipFilterModePoint;
    patterns_tex.normalized     = 0;
        
    size_t offset_t ;
    hipBindTexture( &offset_t, (const struct textureReference*) texRefTable,
            (const void*) dpatterns, (const struct hipChannelFormatDesc*) &channelDesc, 
            len * sizeof(int));
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_tex<<<grid,block>>>(dsizes,vpatterns.size(),dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    // if(res_to_vec){

    //     std::cout << "res to vec" << std::endl;

    //     char * h_match_result = new char[text_size];
    //     hipMemcpy(h_match_result,dresult_buf,text_size,hipMemcpyDeviceToHost);
    //     for (int i = 0; i < text_size; i++){
    //         if (h_match_result[i]){
    //             res.push_back(std::pair<int,int>(i,(int)h_match_result[i]));
    //         }
    //     }
    // }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);

    //unbind
    hipUnbindTexture(patterns_tex);
    // hipEventDestroy(start);
    // hipEventDestroy(stop); 

}



// void match_naive_shared(std::string pattern, std::string subject_string_filename, long nochunk,long size, int offset,int verbose){ //nochunk == 0 => nochunk

//     char* dtextptr;
//     long text_size;
//     if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
//         std::cout << "error opening file" << "\n";
//         return;
//     }

//     auto pattern_size = pattern.size();
//     char *dpattern;
//     hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
//     hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

//     char* dresult_buf;
//     hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
//     int chunk = 256;

//     dim3 block(block_size);
//     long grid_size;
//     long gsqrt;
//     am::timer time;
//     std::cout << "running ..." << "\n";
//     time.start();
    
//     if(nochunk){
//         grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         match_shared<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
//         hipDeviceSynchronize();
//         time.stop();
//     } else{
//         grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         match_chunk_shared<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
//         hipDeviceSynchronize();
//         time.stop();
//     }

//     std::cout << "running time " << time.milliseconds() << " ms" << std::endl;    

//     if(verbose){
//         write_from_device(&dresult_buf,text_size);
//     }

//     hipFree(dresult_buf);
//     hipFree(dtextptr);
//     hipFree(dpattern);
      
// }


// void match_kmp(std::string pattern, std::string subject_string_filename, int nochunk,long size,long offset,int verbose){ //nochunk == 0 => nochunk

//     auto pattern_size = pattern.size();
    
//     char* dtextptr;
//     long text_size;
//     if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
//         std::cout << "error opening file" << "\n";
//         return;
//     }
    
//     char *dpattern;
//     hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
//     hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

//     int* prefix_table = new int[pattern_size];
//     prefix(pattern.c_str(),pattern_size,prefix_table);
//     int* dprefix_table;

//     hipMalloc((void**)&dprefix_table, pattern_size * sizeof(int));
//     hipMemcpy((void*)dprefix_table,prefix_table,pattern_size * sizeof(int),hipMemcpyHostToDevice); 
//     delete[](prefix_table);

//     char* dresult_buf;
//     // std::cout << "text length : " << text_size << "\n";
//     //think about data transfer;
//     hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
//     int chunk = 256;

//     dim3 block(block_size);
//     long grid_size;
//     long gsqrt;
//     am::timer time;
//     std::cout << "running ..." << "\n";
//     time.start();
    
//     if(nochunk){
//         grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         kmp_nochunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
//         hipDeviceSynchronize();
//         time.stop();
//     } else{
//         grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
//         gsqrt = (int)sqrt(grid_size) + 1;
//         dim3 grid(gsqrt,gsqrt);
//         kmp_chunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
//         hipDeviceSynchronize();
//         time.stop();
//     }
//     std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

//     if(verbose){
//         write_from_device(&dresult_buf,text_size);
//     }
    
//     hipFree(dresult_buf);
//     hipFree(dtextptr);
//     hipFree(dpattern);
//     hipFree(dprefix_table); 
// }
