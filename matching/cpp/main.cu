#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <math.h>

//CPU timer
#include "../include/timer.h"
//arg parsing
#include "../include/cxxopts.hpp"


#define block_size BLOCK_SIZE

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct Template{

    char array[32] = {0};
    int size;

}Template;

__device__ long threadId(){
    
    long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;
    long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;
    return threadId;

}

long GetFileSize(std::string filename)
{
    // struct stat stat_buf;
    // int rc = stat(filename.c_str(), &stat_buf);
    // return rc == 0 ? stat_buf.st_size : -1;
    int fd = open(filename.c_str(),O_RDONLY);  //;
    long size = lseek(fd, 0, SEEK_END);
    close(fd);
    return size;
}

std::vector<std::string> read_pattern(std::string filename){
    
    std::ifstream file(filename,std::ios::binary);
    std::vector<std::string> res = std::vector<std::string>();
 
    if (!file) 
    {
        std::cout << "error openning pattern file" << "\n"; 
        return res;
    // TODO: assign item_name based on line (or if the entire line is 
    // the item name, replace line with item_name in the code above)
    }
    while(!file.eof()){

        std::string str;
        std::getline(file,str,'\0');
        res.push_back(str);
    }

    return res;

}

char* read_file(std::string filename,long &text_size,long size = 0, long offset = 0){
    
    long f_size = GetFileSize(filename);//TODO
    if(f_size == -1){
        std::cout << "bad_size" << "\n";
        return nullptr;
    }
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return nullptr;
    }

    if(size != 0 && size <= f_size){
        text_size = size;
    }else{
        text_size = f_size;
    }

    if(offset != 0){
        fseek(f,offset * sizeof(char),SEEK_CUR);
        if((f_size - offset) < size){
            text_size = f_size - offset;
        }
    }
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    long nbytes;

    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){//number of chunks
        
        if(feof(f)){
            std::cout << "premature end of file" << "\n";
            break;
        }

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;
        nbytes = fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,nbytes,hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(char** dresult_buf,long text_size){

    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    char* result_buf = new char[text_chunk];


    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(char),hipMemcpyDeviceToHost);
        
        for (long i = 0; i < (right_bound-left_bound); i++) {
            std::cout << (int)(result_buf[i]);
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}

__global__ void match(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {


    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = 0;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}

__global__ void match_shared(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x < pattern_size) {
        spattern[threadIdx.x] = pattern[threadIdx.x];
    }
    __syncthreads();

    if(t_id < text_size){
        
        int matched = 1;
        result_buf[t_id] = 0;

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != spattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     
    }
}

__global__ void match_multy(char* patterns, int* p_sizes, int p_number, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        
        result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1) {
                for(int j = 0; j < p_sizes[i]; j++) {
                
                    if(text[t_id + j] != patterns[j+p_offset]) {
                        matched = -1;
                        break;
                    }
                } 
            
                if(matched == 1) {
                    result_buf[t_id] = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }             
    }
}

__constant__ char mpatterns[128*64];
__constant__ int cp_sizes[64];
__global__ void match_multy_const(int p_number, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 1;
        result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - cp_sizes[i] + 1){
                for(int j = 0; j < cp_sizes[i]; j++) {
            
                    if(text[t_id + j] != mpatterns[j + p_offset]) {
                        matched = -1;
                        
                        break;
                    }
                } 
            
                if(matched == 1) {
                    result_buf[t_id] = i+1; // 0 stands for missmatch
                }
            }
            p_offset += cp_sizes[i];
        }             
    }
}

__global__ void match_multy_shared(char* patterns, int* p_sizes, int p_number,int p_len, char* text, long text_size, char* result_buf){
    
    //assume that blockSize >= p_len
    extern __shared__ char sPatterns[];
    if (threadIdx.x < p_len){
        sPatterns[threadIdx.x] = patterns[threadIdx.x];
    }
    __syncthreads();

    long t_id = threadId();

    if(t_id < text_size){

        int p_offset = 0;
        int matched = 1;

        result_buf[t_id] = 0;

        for(int i = 0; i < p_number; i++) {//for each pattern
            matched = 1;
            if(t_id < text_size - p_sizes[i] + 1){
                for(int j = 0; j < p_sizes[i]; j++) {
            
                    if(text[t_id + j] != sPatterns[j + p_offset]) {
                        matched = -1;
                        
                        break;
                    }
                } 
            
                if(matched == 1) {
                    result_buf[t_id] = i+1; // 0 stands for missmatch
                }
            }
            p_offset += p_sizes[i];
        }             
    }


}

__global__ void match_chunk_shared(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x == 0) {
        for(int i = 0; i < pattern_size; i++){
            spattern[i] = pattern[i];
        }
    }
    __syncthreads();
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;
            if(i < text_size - left_bound - pattern_size + 1){

                for(int j = 0; j < pattern_size; j++) {

                    if(text[left_bound + i + j] != spattern[j]) {
                        matched = -1;
                        break;
                    }
                }

                if(matched == 1) {
                    result_buf[left_bound + i] = 1;
                }
            }
        }
                             
    }
}

__global__ void match_chunk(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;
            
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

void prefix(const char* pattern, int pattern_size, int* prefix_table){
    
    prefix_table[0] = 0;
    
    for (int i = 1; i < pattern_size; ++i) {
		
        int j = prefix_table[i-1];
		
        while (j > 0 && pattern[i] != pattern[j]){
			
            j = prefix_table[j-1];
        
        }
		
        if (pattern[i] == pattern[j])  ++j;
		
        prefix_table[i] = j;
	}
}

__global__ void kmp_chunk(int* prefix_table, char* pattern,int pattern_size,char* text, long text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}


__global__ void kmp_nochunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}


void multipattern_match(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_multy<<<grid,block>>>(dpatterns,dsizes,vpatterns.size(),dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}

void multipattern_match_const(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];     
    }
    
    int loffset = 0;

    // char* dpatterns;
    // int* dsizes;

    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    // hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    
    hipMemcpyToSymbol(HIP_SYMBOL(cp_sizes), sizes, vpatterns.size() * sizeof(int)); 
    
    
    
    // hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpyToSymbol(HIP_SYMBOL(mpatterns),vpatterns[i].c_str(),vpatterns[i].size(),loffset);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_multy_const<<<grid,block>>>(vpatterns.size(),dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    // hipFree(dpatterns);
    // hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
}

void multipattern_match_shared(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    //nochunk only
    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();

    grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
    gsqrt = (int)sqrt(grid_size) + 1;
    dim3 grid(gsqrt,gsqrt);
    match_multy_shared<<<grid,block,len * sizeof(char)>>>(dpatterns,dsizes,vpatterns.size(),len,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}

void match_naive(std::string pattern, std::string subject_string_filename, int nochunk, long size, long offset,int verbose){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);

}



void match_naive_shared(std::string pattern, std::string subject_string_filename, long nochunk,long size, int offset,int verbose){ //nochunk == 0 => nochunk

    char* dtextptr;
    long text_size;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_shared<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk_shared<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;    

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
      
}


void match_kmp(std::string pattern, std::string subject_string_filename, int nochunk,long size,long offset,int verbose){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    long text_size;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int* prefix_table = new int[pattern_size];
    prefix(pattern.c_str(),pattern_size,prefix_table);
    int* dprefix_table;

    hipMalloc((void**)&dprefix_table, pattern_size * sizeof(int));
    hipMemcpy((void*)dprefix_table,prefix_table,pattern_size * sizeof(int),hipMemcpyHostToDevice); 
    delete[](prefix_table);

    char* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    //think about data transfer;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_nochunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_chunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    }
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
    hipFree(dprefix_table); 
}


__constant__ char c_pattern[128*64]; //might be as fast as registers, but not in this case =)

__global__ void match_chunk_const(int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = 0;
            int matched = 1;

            for(int j = 0; j < pattern_size; j++) {
                
                if(text[left_bound + i + j] != c_pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}

__global__ void match_const(int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = 0;
        if(t_id < text_size - pattern_size + 1){

            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != c_pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = 1;
            }             
        }
                     

    }
}

__constant__ int c_prefix[128*64];

__global__ void kmp_chunk_const(int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = 0;
        }

        while(ams > 0 && c_pattern[ams] != text[i]){
            ams = c_prefix[ams-1];
        }

        if(text[i] == c_pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = c_prefix[ams-1];
        }


    }
}

void match_naive_const(std::string pattern, std::string subject_string_filename, int nochunk,long size, long offset,int verbose){
    

    auto pattern_size = pattern.size(); // <= 128
    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),(void*)pattern.c_str(),pattern.size()*sizeof(char));

    // int* prefix_table = new int[pattern_size];
    // prefix(pattern.c_str(),pattern_size,prefix_table);
    // hipMemcpyToSymbol(HIP_SYMBOL(c_prefix),(void*)prefix_table,pattern.size()*sizeof(int));
    // delete[](prefix_table);

    long text_size;//TODO

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    //think about data transfer;
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    

    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk_const<<<grid,block>>>(pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void match_kmp_const(std::string pattern, std::string subject_string_filename, int nochunk,long size, long offset,int verbose){

    auto pattern_size = pattern.size(); // <= 128
    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),(void*)pattern.c_str(),pattern.size()*sizeof(char));

    int* prefix_table = new int[pattern_size];
    prefix(pattern.c_str(),pattern_size,prefix_table);
    hipMemcpyToSymbol(HIP_SYMBOL(c_prefix),(void*)prefix_table,pattern.size()*sizeof(int));
    delete[](prefix_table);

    long text_size;//TODO

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    //think about data transfer;
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    

    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_chunk_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    hipFree(dresult_buf);
    hipFree(dtextptr);
}

int main(int argc, char** argv) {

    
    long size = 0;
    long offset = 0;
    int type = 0;
    int verbose = 1;
    
    cxxopts::Options options("as", " - example command line options");

    options.add_options()("p,pattern","pattern to look for",cxxopts::value<std::string>())
                         ("f,filename","filename/devicename to look for in",cxxopts::value<std::string>())
                         ("s,size", "size of data to read",cxxopts::value<long>(size)->default_value("0"))
                         ("o,offset", "offset of data to read",cxxopts::value<long>(offset)->default_value("0"))
                         ("a,algorithm","algorithm to look for with",cxxopts::value<std::string>())
                         ("t,type","type of algorithm: 0 stands for nochunk-based and 1 for chunk-based",cxxopts::value<int>(type))
                         ("v,verbose","print result or not 0 stands for 'No' 1 for 'Yes'",cxxopts::value<int>(verbose));
    // std::string subject_string_filename("data/subject.txt");

    auto result = options.parse(argc, argv);

    if(result.count("algorithm") && result.count("type") && result.count("pattern") && result.count("filename") && result.count("verbose")){
        auto alg_name = result["algorithm"].as<std::string>();
        auto filename = result["filename"].as<std::string>();
        //patterns are separated with \x00
        auto patterns = read_pattern(result["pattern"].as<std::string>());
        std::string pattern; 
        if(patterns.size() == 1){
            pattern = patterns[0];
 //if contains \x00 --- considered empty
            if(type == 1 || type == 0){
                if(alg_name == "naive"){
                    match_naive(pattern,filename,type,size,offset,verbose);
                }else if(alg_name == "naivec"){
                    match_naive_const(pattern,filename,type,size,offset,verbose);
                }else if(alg_name == "naivesh"){
                    match_naive_shared(pattern,filename,type,size,offset,verbose);
                }else if(alg_name == "kmpc"){
                    match_kmp_const(pattern,filename,0,size,offset,verbose);
                }else if(alg_name == "kmp"){
                    match_kmp(pattern,filename,0,size,offset,verbose);
                }
            }else{
                std::cout << "type should be either 1 or 0" << "\n";
            }
        }else if(patterns.size() > 1){
            if(type == 1 || type == 0){
                if(alg_name == "mnaive"){
                    multipattern_match(patterns, filename, size, offset, verbose);
                    
                }else if(alg_name == "mnaivec"){
                    multipattern_match_const(patterns, filename, size, offset, verbose);
                }else if(alg_name == "mnaivesh"){
                    multipattern_match_shared(patterns,filename,size,offset,verbose);
                }
            }
        }else{
            std::cout << "bad patterns" << "\n";
        }
    }else{
        std::cout << "algorithm name shoud be specified with --algorithm=name and type with --type=type" << "\n";
        }    

    return 0;
}
