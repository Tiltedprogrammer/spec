#include "hip/hip_runtime.h"
multiple_match_const_unroll
__device__ long threadId(){
long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;
long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;
return threadId;
}
__constant__ int mpatterns[128*64];
__global__
void multiple_match_const_unroll(char* text, long text_size, char* result_buf) {
    long t_id = threadId();
    if(t_id < text_size){
       int p_offset = 0;
       int matched = 1;
       int match_result = 0;
       if(t_id < text_size -16 + 1){
        matched = 1;
        #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 0+1;
      }
      p_offset += 3;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 1+1;
      }
      p_offset += 3;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 2+1;
      }
      p_offset += 16;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 3+1;
      }
      p_offset += 16;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 4+1;
      }
      p_offset += 16;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 5+1;
      }
      p_offset += 5;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 6; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 6+1;
      }
      p_offset += 6;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 7+1;
      }
      p_offset += 3;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 8+1;
      }
      p_offset += 4;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 9+1;
      }
      p_offset += 4;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 10+1;
      }
      p_offset += 3;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 11+1;
      }
      p_offset += 5;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 12+1;
      }
      p_offset += 3;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 7; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 13+1;
      }
      p_offset += 7;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 14+1;
      }
      p_offset += 4;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 8; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 15+1;
      }
      p_offset += 8;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 16+1;
      }
      p_offset += 5;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 17+1;
      }
      p_offset += 5;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 18+1;
      }
      p_offset += 4;
      matched = 1;
      #pragma unroll
      for(int j = 0; j < 8; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
             matched = -1;
             break;
          }
      }
      if(matched == 1) {
          match_result = 19+1;
      }
      p_offset += 8;
   }else{
      matched = 1;
      if(t_id < text_size - 3 + 1){
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 0+1;
      }
}
      p_offset += 3;
      matched = 1;
      if(t_id < text_size - 3 + 1){
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 1+1;
      }
}
      p_offset += 3;
      matched = 1;
      if(t_id < text_size - 16 + 1){
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 2+1;
      }
}
      p_offset += 16;
      matched = 1;
      if(t_id < text_size - 16 + 1){
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 3+1;
      }
}
      p_offset += 16;
      matched = 1;
      if(t_id < text_size - 16 + 1){
      #pragma unroll
      for(int j = 0; j < 16; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 4+1;
      }
}
      p_offset += 16;
      matched = 1;
      if(t_id < text_size - 5 + 1){
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 5+1;
      }
}
      p_offset += 5;
      matched = 1;
      if(t_id < text_size - 6 + 1){
      #pragma unroll
      for(int j = 0; j < 6; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 6+1;
      }
}
      p_offset += 6;
      matched = 1;
      if(t_id < text_size - 3 + 1){
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 7+1;
      }
}
      p_offset += 3;
      matched = 1;
      if(t_id < text_size - 4 + 1){
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 8+1;
      }
}
      p_offset += 4;
      matched = 1;
      if(t_id < text_size - 4 + 1){
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 9+1;
      }
}
      p_offset += 4;
      matched = 1;
      if(t_id < text_size - 3 + 1){
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 10+1;
      }
}
      p_offset += 3;
      matched = 1;
      if(t_id < text_size - 5 + 1){
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 11+1;
      }
}
      p_offset += 5;
      matched = 1;
      if(t_id < text_size - 3 + 1){
      #pragma unroll
      for(int j = 0; j < 3; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 12+1;
      }
}
      p_offset += 3;
      matched = 1;
      if(t_id < text_size - 7 + 1){
      #pragma unroll
      for(int j = 0; j < 7; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 13+1;
      }
}
      p_offset += 7;
      matched = 1;
      if(t_id < text_size - 4 + 1){
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 14+1;
      }
}
      p_offset += 4;
      matched = 1;
      if(t_id < text_size - 8 + 1){
      #pragma unroll
      for(int j = 0; j < 8; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 15+1;
      }
}
      p_offset += 8;
      matched = 1;
      if(t_id < text_size - 5 + 1){
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 16+1;
      }
}
      p_offset += 5;
      matched = 1;
      if(t_id < text_size - 5 + 1){
      #pragma unroll
      for(int j = 0; j < 5; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 17+1;
      }
}
      p_offset += 5;
      matched = 1;
      if(t_id < text_size - 4 + 1){
      #pragma unroll
      for(int j = 0; j < 4; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 18+1;
      }
}
      p_offset += 4;
      matched = 1;
      if(t_id < text_size - 8 + 1){
      #pragma unroll
      for(int j = 0; j < 8; j++) {
          if(text[t_id + j] != mpatterns[j + p_offset]) {
              matched = -1;
              break;
          }
      }
      if(matched == 1) {
         match_result = 19+1;
      }
}
      p_offset += 8;
   }
result_buf[t_id] = match_result;
}
}