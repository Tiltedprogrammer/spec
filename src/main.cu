#include <string>
#include <iostream>

#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"

void println(int a) {
    printf("%i\n",a);
}


int main(int argc, char** argv) {
    
    std::string pattern = "abcdefg00";
    // pattern.resize(32);
    auto pattern_size = 7; 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";
    //chage get42_cuda to get42 for cpu version
    dummy_fun += "  string_match(Pattern { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "}, text, text_size,result_buf,32);}";

    std::string program = std::string((char*)fun_impala) + dummy_fun;
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    int* result_buf;
    std::string text;
    std::cin >> text;
    auto text_size = text.length();
    std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size + 1);
    hipMemcpy((void*)textptr,text.c_str(),text_size + 1,hipMemcpyHostToDevice);
    hipMallocManaged((void**)&result_buf, text_size);
    
    hipMemset((void*)result_buf, -1, text_size * sizeof(int));
    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    call(textptr,text_size,result_buf);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(result_buf);

    return 0;
}