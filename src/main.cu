#include <string>
#include <iostream>

#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"

void println(int a) {
    printf("%i\n",a);
}


int main(int argc, char** argv) {
    
    std::string pattern = "abcdefga";

    pattern.resize(31,'0');
    auto pattern_size = 8; 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";
    //chage get42_cuda to get42 for cpu version
    dummy_fun += "  string_match_pseudoKMP(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return 0;
    }
    
    std::string text;
    std::cin >> text;
    auto text_size = text.length();
    int* result_buf = new int[text_size];
    std::cout << "text length : " << text_size << "\n";
    // const char* textptr = text.c_str();
    //think about data transfer;
    // hipMalloc((void**)&textptr, text_size + 1);
    // hipMemcpy((void*)textptr,text.c_str(),text_size + 1,hipMemcpyHostToDevice);
    // hipMallocManaged((void**)&result_buf, text_size * sizeof(int));
    
    for(int i = 0; i < text_size; i++) {
        result_buf[i] = -1;
    }
    // hipMemset((void*)result_buf, -1, text_size * sizeof(int));
    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    call(text.c_str(),text_size,result_buf);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    // hipFree(result_buf);
    delete[] (result_buf);

    return 0;
}