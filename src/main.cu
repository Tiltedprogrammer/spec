#include <string>
#include <iostream>

#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"


int main(int argc, char** argv) {
    
    // if (argc != 2 ) {
        // std::cout << "pattern string required\n";
        // return 0;
    // }
    std::string pattern = std::string(argv[1]);

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return 0;
    }
    auto pattern_size = pattern.size();
    pattern.resize(31,'0'); 
    std::cout << pattern_size << "\n";
    std::cout << pattern << "\n";
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";
    
    dummy_fun += "  string_match(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return 0;
    }
    
    std::string text = std::string(argv[2]);
    // std::cin >> text;
    auto text_size = text.length();
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,text.c_str(),text_size * sizeof(char),hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    call(dtext,text_size,dresult_buf);
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);

    return 0;
}