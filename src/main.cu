#include <cstdio>
#include <string>
#include <iostream>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <stdlib.h>



#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"
#include "timer.h"


long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

char* read_file(std::string filename,  int &text_size){
    
    text_size = GetFileSize(filename) - 1;//TODO
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return 0;
    }

    
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    for(int i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;
        fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,right_bound-(left_bound),hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(int** dresult_buf,int text_size){

    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    int* result_buf = new int[text_chunk];


    for(int i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(int),hipMemcpyDeviceToHost);
        
        for (int i = 0; i < (right_bound-left_bound); i++) {
            std::cout << result_buf[i];
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}


void match_pe_(std::string pattern, std::string subject_string_filename) {
    
    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }
    auto pattern_size = pattern.size();
    pattern.resize(31,'0'); 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    dummy_fun += "  string_match_pseudoKMP(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}

void match_pe(std::string subject_string_filename,std::string program_) {
    
        
    std::string program = std::string((char*)fun_impala) + program_;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    int text_size;
    char* dtextptr = read_file(subject_string_filename,text_size);
    
    int* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtextptr,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void prefix_f(std::string pattern, int index){

     std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy()-> i32{\n";

    dummy_fun += "  prefix_function(\"" + pattern + "\","
              + std::to_string(index) + ")}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef int (*function) ();
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }
    std::cout << call() << "\n";

}

void match_pe_pointer(std::string pattern, std::string subject_string_filename) {
    
    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }
    auto pattern_size = pattern.size();
    // pattern.resize(31,'0'); 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    dummy_fun += "  string_match_pseudoKMP_pointer(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f; 
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	std::cout << "can not oppen file" << subject_string_filename << "\n";
	    return;
    }
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}

void match_pe_pointer_multipattern(int p_number,char** argv_patterns, std::string subject_string_filename) {
    
    int* sizes = new int[p_number];
    int len = 0;
    for(int i = 1; i < p_number+1; i++) {
        auto str = std::string(argv_patterns[i]);
        sizes[i-1] = str.length();
        len += str.length();    
    }

    char* patterns = new char[len];
    
    int offset = 0;

    for(int i = 0; i < p_number; i++){

        for(int j = 0; j < sizes[i]; j++){
            patterns[offset+j] = argv_patterns[i+1][j];
        }
        offset+=sizes[i];    
    } 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

//TODO max_pattern_size
    dummy_fun += "  string_match_pseudoKMP_pointer_multiple( \"" + std::string(patterns) + "\", "
              + "["+ std::to_string(sizes[1]) + "]" + "," + std::to_string(p_number)+",3, text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}


//TODO template method

//TODO refactor with fun that takes impala program as input

int main(int argc, char** argv) {
    
    // if (argc != 2 ) {
        // std::cout << "pattern string required\n";
        // return 0;
    // }
    std::string pattern = std::string(argv[1]);

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return 0;
    }
    
    auto pattern_size = pattern.size();
    pattern.resize(31,'0'); 
    
    std::string subject = std::string("data/subject.txt");

    std::string match_pseudoKMP;

    match_pseudoKMP += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_pseudoKMP += "  string_match_pseudoKMP(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string match_pseudoKMP_nochunk;

    match_pseudoKMP_nochunk += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_pseudoKMP_nochunk += "  string_match_pseudoKMP_nochunk(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string match_pseudoKMP_nochunk_nope;

    match_pseudoKMP_nochunk_nope += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_pseudoKMP_nochunk_nope += "  string_match_pseudoKMP_nochunk_nope(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string match_pseudoKMP_chunk_nope;

    match_pseudoKMP_chunk_nope += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_pseudoKMP_chunk_nope += "  string_match_pseudoKMP_nope(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;
    
    std::string match_KMP_chunk;
    match_KMP_chunk += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_KMP_chunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf,256,256,0)}"; //;

    std::string match_KMP_nochunk;
    match_KMP_nochunk += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_KMP_nochunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf,256,256,1)}"; //;

    std::string match_naive_nochunk;
    match_naive_nochunk += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_naive_nochunk += "  string_match(Template {array :\"" + pattern + "\", size :"
              + std::to_string(pattern_size) + "},"+ std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf,256,256,1)}"; //;

    std::string match_naive_chunk;
    match_naive_chunk += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    match_naive_chunk += "  string_match(Template {array :\"" + pattern + "\", size :"
              + std::to_string(pattern_size) + "},"+ std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf,256,256,0)}"; //;
    
    // match_pe_pointer(pattern,subject);
    // match_pe_pointer_multipattern(argc-1,argv,subject);
    // match_pe(subject,match_pseudoKMP);
    auto arg = std::string(argv[2]) + std::string(argv[3]);
    if(arg == "dirty0"){
        match_pe(subject,match_pseudoKMP);
    }else if(arg == "dirty1") {
        match_pe(subject,match_pseudoKMP_nochunk);
    }else if(arg == "kmp0") {
        match_pe(subject,match_KMP_chunk);
    }else if(arg == "kmp1") {
        match_pe(subject,match_KMP_nochunk);
    }else if(arg == "dirtynaive0") {
        match_pe(subject,match_pseudoKMP_chunk_nope);
    }else if(arg == "dirtynaive1") {
        match_pe(subject,match_pseudoKMP_nochunk_nope);
    }else if(arg == "clean0") {
        match_pe(subject,match_naive_chunk);
    }else if(arg == "clean1") {
        match_pe(subject,match_naive_nochunk);
    }else {

    }
    // prefix(pattern,subject);
    // match_pe_nochunk(pattern,subject);
    // std::cout << GetFileSize(std::string("subject.txt"));

    return 0;
}