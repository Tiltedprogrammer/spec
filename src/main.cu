#include <cstdio>
#include <string>
#include <iostream>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <stdlib.h>



#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"
#include "timer.h"

void match_naive_cuda(std::string pattern, std::string text) {
    
    if (pattern.size() > 31) { //actual maximum is 32 for now
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    pattern.resize(31,'0'); 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    dummy_fun += "  string_match(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    
    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    }
    
    auto text_size = text.length();
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,text.c_str(),text_size * sizeof(char),hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    call(dtext,text_size,dresult_buf);
    
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);

}

long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}


void match_pe(std::string pattern, std::string subject_string_filename) {
    
    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }
    auto pattern_size = pattern.size();
    pattern.resize(31,'0'); 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    dummy_fun += "  string_match_pseudoKMP(Template { array : \"" + pattern + "\", size : "
              + std::to_string(pattern_size) + "},32i8 ,text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}


void match_pe_pointer(std::string pattern, std::string subject_string_filename) {
    
    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }
    auto pattern_size = pattern.size();
    // pattern.resize(31,'0'); 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

    dummy_fun += "  string_match_pseudoKMP_pointer( \"" + pattern + "\", "
              + std::to_string(pattern_size) + ",text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}



int main(int argc, char** argv) {
    
    // if (argc != 2 ) {
        // std::cout << "pattern string required\n";
        // return 0;
    // }
    std::string pattern = std::string(argv[1]);
    std::string subject = std::string("data/subject.txt");
    
    // match_naive_cuda(pattern,text);
    match_pe_pointer(pattern,subject);
    // std::cout << GetFileSize(std::string("subject.txt"));

    return 0;
}