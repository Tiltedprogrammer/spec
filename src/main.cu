#include <cstdio>
#include <string>
#include <iostream>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <stdlib.h>

#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"
#include "timer.h"
#include "cxxopts.hpp"

#define block_size BLOCK_SIZE

extern "C" void string_match_nope(const char*,int,short int,char*,long,char*,int,int,int);
extern "C" void string_match_multiple(const char *, const int *,short,const char*, long,char *,int);
// extern "C" void match_kmp(const char*,int,short int,char*,long,char*,int,int,int);


long GetFileSize(std::string filename)
{
    // struct stat stat_buf;
    // int rc = stat(filename.c_str(), &stat_buf);
    // return rc == 0 ? stat_buf.st_size : -1;
    int fd = open(filename.c_str(),O_RDONLY);  //;
    long size = lseek(fd, 0, SEEK_END);
    close(fd);
    return size;
}

std::vector<std::string> read_pattern(std::string filename){
    
    std::ifstream file(filename,std::ios::binary);
    std::vector<std::string> res = std::vector<std::string>();
 
    if (!file) 
    {
        std::cout << "error openning pattern file" << "\n"; 
        return res;
    // TODO: assign item_name based on line (or if the entire line is 
    // the item name, replace line with item_name in the code above)
    }
    while(!file.eof()){

        std::string str;
        std::getline(file,str,'\0');
        res.push_back(str);
    }
    // std::getline(file, str);
    return res;

}

char* read_file(std::string filename,long &text_size,long size = 0, long offset = 0){
    
    long f_size = GetFileSize(filename);//TODO
    if(f_size == -1){
        std::cout << "bad_size" << "\n";
        return nullptr;
    }
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return nullptr;
    }

    if(size != 0 && size <= f_size){
        text_size = size;
    }else{
        text_size = f_size;
    }

    if(offset != 0){
        fseek(f,offset * sizeof(char),SEEK_CUR);
        if((f_size - offset) < size){
            text_size = f_size - offset;
        }
    }
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    long nbytes;

    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){//number of chunks
        
        if(feof(f)){
            std::cout << "premature end of file" << "\n";
            break;
        }

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;
        nbytes = fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,nbytes,hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(char** dresult_buf,long text_size){

    int text_chunk = 128 * 1024 * 1024;
    
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    char* result_buf = new char[text_chunk];


    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(char),hipMemcpyDeviceToHost);
        
        for (long i = 0; i < (right_bound-left_bound); i++) {
            std::cout << (int)result_buf[i];
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}


void match_pe(std::string subject_string_filename,long size, long offset,std::string program_, int verbose) {
    
        
    std::string program = std::string((char*)fun_impala) + program_;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, long, char *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    long text_size;
    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset))==nullptr){
        std::cout << "error reading file" << "\n";
        return;
    }

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
       
        write_from_device(&dresult_buf,text_size);
    
    }
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void match_pe_pat(std::string subject_string_filename,std::string program_,std::string pattern, int pattern_size,long size,long offset) {
    
        
    std::string program = std::string((char*)fun_impala) + program_;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*,int,const char*, long, char *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }
    
    long text_size;
    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        return;
    }
    
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";

    char* d_pat;
    hipMalloc((void**)&d_pat,pattern_size * sizeof(char));
    hipMemcpy(d_pat,pattern.c_str(),pattern_size*sizeof(char),hipMemcpyHostToDevice);
    
    time.start();

    call(d_pat,pattern_size,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(d_pat);
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

template<typename Function>
void match_nope(std::string subject_string_filename,std::string pattern,int pattern_size, int nochunk, Function f,long size,long offset,int verbose) {
    
    am::timer time;

    long text_size;

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";
    

    char* d_pat;
    hipMalloc((void**)&d_pat,pattern_size);
    hipMemcpy(d_pat,pattern.c_str(),pattern_size,hipMemcpyHostToDevice);
    
    time.start();

    if(nochunk){
        f(d_pat,pattern_size,32,dtextptr,text_size,dresult_buf,512,256,1);
    }else {
        f(d_pat,pattern_size,32,dtextptr,text_size,dresult_buf,512,256,0);
    }
    hipDeviceSynchronize();
    time.stop();

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }

    hipFree(d_pat);
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void prefix_f(std::string pattern, int index){

     std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy()-> i32{\n";

    dummy_fun += "  prefix_function(\"" + pattern + "\","
              + std::to_string(index) + ",0,0)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef int (*function) ();
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }
    std::cout << call() << "\n";

}

void multipattern_match(std::vector<std::string> vpatterns, std::string file_name,long size, long offset,int verbose){

    int* sizes = new int[vpatterns.size()];

    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].size();
        len += sizes[i];    
    }
    
    int loffset = 0;

    char* dpatterns;
    int* dsizes;

    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(file_name,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + loffset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        loffset += sizes[i];
    }
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));

    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(patterns : &[u8], sizes : &[i32], size : u8, text : &[u8], text_size : i64, result_buf : &mut[u8],block_size : i32) -> (){\n";

    dummy_fun += "  string_match_multiple_nope(patterns, sizes,size,text,text_size,result_buf,block_size);}"; //;
    

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    time.stop();
    std::cout << "compilation time " << time.milliseconds() << std::endl;
    time.reset();
    typedef void (*function) (const char*,const int*,short,const char*, long, const char *,int);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    std::cout << "\n";


    std::cout << "running ..." << "\n";
    time.start();
    call(dpatterns,dsizes,vpatterns.size(),dtextptr,text_size,dresult_buf,block_size);
    hipDeviceSynchronize();
    time.stop();

    
    delete[](sizes);
    
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpatterns);
    hipFree(dsizes);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);  
}

void match_pe_pointer_multipattern(std::vector<std::string> vpatterns, std::string subject_string_filename, long size, long offset, int verbose) {
    
    std::string sizes;

    int len = 0;
    sizes = std::to_string(vpatterns[0].size());
    for(int i = 1; i < vpatterns.size(); i++) {
        sizes += "," + std::to_string(vpatterns[i].size());
        len +=  vpatterns[i].size();   
    }
    
    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }

    std::string dummy_fun;

    std::string patterns;
    for (auto &vp : vpatterns) patterns += vp;
    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

    dummy_fun += "  string_match_multiple(\"" + patterns + "\","
              + "["+ sizes + "]" + "," + std::to_string(vpatterns.size()) + "u8,text, text_size,result_buf,"+std::to_string(block_size)+")}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    time.stop();
    std::cout << "compilation time " << time.milliseconds() << std::endl;
    time.reset();
    typedef void (*function) (const char*, long, const char *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    std::cout << "\n";
    
    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    //think about data transfer;
  
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    if(verbose){
        write_from_device(&dresult_buf,text_size);
    }
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}


//TODO template method

//TODO refactor with fun that takes impala program as input

int main(int argc, char** argv) {
    
   
    long size = 0;
    long offset = 0;
    int type = 0;
    int verbose = 1;
    
    cxxopts::Options options("as", " - example command line options");

    options.add_options()("p,pattern","pattern to look for",cxxopts::value<std::string>())
                         ("f,filename","filename/devicename to look for in",cxxopts::value<std::string>())
                         ("s,size", "size of data to read",cxxopts::value<long>(size)->default_value("0"))
                         ("o,offset", "offset of data to read",cxxopts::value<long>(offset)->default_value("0"))
                         ("a,algorithm","algorithm to look for with",cxxopts::value<std::string>())
                         ("t,type","type of algorithm: 0 stands for nochunk-based and 1 for chunk-based",cxxopts::value<int>(type))
                         ("v,verbose","print result or not 0 stands for 'No' 1 for 'Yes'",cxxopts::value<int>(verbose));
    // std::string subject_string_filename("data/subject.txt");

    auto result = options.parse(argc, argv);

    if(result.count("algorithm") && result.count("type") && result.count("pattern") && result.count("filename") && result.count("verbose")){
        auto alg_name = result["algorithm"].as<std::string>();
        auto filename = result["filename"].as<std::string>();
        auto patterns = read_pattern(result["pattern"].as<std::string>());
        std::string pattern;
        if(patterns.size() >= 1){
            pattern = patterns[0];
        
        auto pattern_size = pattern.size();
    // pattern.resize(31,'0'); 

        std::string r_naive_spec;

        r_naive_spec += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        r_naive_spec += "  string_match_pseudoKMP(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_nochunk;

        match_pseudoKMP_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk += "  string_match_pseudoKMP_nochunk(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_nochunk_nope;

        match_pseudoKMP_nochunk_nope += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk_nope += "  string_match_pseudoKMP_nochunk_nope(\"" + pattern + "\", "
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_chunk_nope;

        match_pseudoKMP_chunk_nope += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_chunk_nope += "  string_match_pseudoKMP_nope(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;
    
        std::string match_KMP_chunk;
        match_KMP_chunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_KMP_chunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;

        std::string match_KMP_nochunk;
        match_KMP_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_KMP_nochunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_nochunk;
        match_naive_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_nochunk += "  string_match(\"" + pattern + "\","
              + std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_chunk;
        match_naive_chunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_chunk += "  string_match(\"" + pattern + "\","
              + std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;
    


        std::string match_naive_nochunk_nope;
        match_naive_nochunk_nope += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_nochunk_nope += "  $string_match_nope($pattern,p_size, 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_chunk_nope;
        match_naive_chunk_nope += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_chunk_nope += "  $string_match_nope($pattern, p_size, 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;



        std::string match_pseudoKMP_chunk_nope_annotated;

        match_pseudoKMP_chunk_nope_annotated += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_chunk_nope_annotated += "  string_match_pseudoKMP_nope(pattern,p_size,32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;


        std::string match_pseudoKMP_nochunk_nope_annotated;

        match_pseudoKMP_nochunk_nope_annotated += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk_nope_annotated += "  string_match_pseudoKMP_nochunk_nope(pattern,p_size,32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;         

        if(type == 0){
            if(alg_name == "r_naive_spec"){
                match_pe(filename,size,offset,r_naive_spec,verbose);
            }else if(alg_name == "kmp"){
                match_pe(filename,size,offset,match_KMP_chunk,verbose);
            }else if(alg_name == "cleankmp"){
                // match_nope(filename,pattern,pattern_size,0,match_kmp,size,offset);
            }else if(alg_name == "cleanpe"){
                match_pe(filename,size,offset,match_naive_chunk,verbose);
            }else if(alg_name == "mcleanpe"){
                match_pe_pointer_multipattern(patterns,filename,size,offset,verbose);
            }else{
                std::cout << "no such algorithm" << "\n";
            }
        }else if(type == 1){
            if(alg_name == "r_naive_spec"){
                match_pe(filename,size,offset,match_pseudoKMP_nochunk,verbose);
            }else if(alg_name == "kmp"){
                match_pe(filename,size,offset,match_KMP_nochunk,verbose);
            }else if(alg_name == "cleankmp"){
                match_nope(filename,pattern,pattern_size,1,string_match_nope,size,offset,verbose);
            }else if(alg_name == "cleanpe"){
                match_pe(filename,size,offset,match_naive_nochunk,verbose);
            }else if(alg_name == "mcleanpe"){
                match_pe_pointer_multipattern(patterns,filename,size,offset,verbose);
            }else if(alg_name == "mcleannope"){
                multipattern_match(patterns,filename,size,offset,verbose);
            }else{
                std::cout << "no such algorithm" << "\n";
            }
        }else {
            std::cout << "type should be either 1 or 0" << "\n";
        }
        }
        }else{
            std::cout << "bad patterns" <<"\n";
        }

    return 0;
}