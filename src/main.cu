#include <cstdio>
#include <string>
#include <iostream>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <stdlib.h>

#define RUNTIME_ENABLE_JIT
#include <anydsl_runtime.h>

// Generated from fun.impala
#include "fun.inc"
#include "timer.h"
#include "cxxopts.hpp"

#define block_size 1024

extern "C" void string_match_nope(const char*,int,short int,char*,long,char*,int,int,int);
// extern "C" void match_kmp(const char*,int,short int,char*,long,char*,int,int,int);


long GetFileSize(std::string filename)
{
    // struct stat stat_buf;
    // int rc = stat(filename.c_str(), &stat_buf);
    // return rc == 0 ? stat_buf.st_size : -1;
    int fd = open(filename.c_str(),O_RDONLY);  //;
    long size = lseek(fd, 0, SEEK_END);
    close(fd);
    return size;
}

std::vector<std::string> read_pattern(std::string filename){
    
    std::ifstream file(filename,std::ios::binary);
    std::vector<std::string> res = std::vector<std::string>();
 
    if (!file) 
    {
        std::cout << "error openning pattern file" << "\n"; 
        return res;
    // TODO: assign item_name based on line (or if the entire line is 
    // the item name, replace line with item_name in the code above)
    }
    while(!file.eof()){

        std::string str;
        std::getline(file,str,'\0');
        res.push_back(str);
    }
    // std::getline(file, str);
    return res;

}

char* read_file(std::string filename,long &text_size,long size = 0, long offset = 0){
    
    long f_size = GetFileSize(filename) - 1;//TODO
    if(f_size == -1){
        std::cout << "bad_size" << "\n";
        return nullptr;
    }
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return nullptr;
    }

    if(size != 0 && size <= f_size){
        text_size = size;
    }else{
        text_size = f_size;
    }

    if(offset != 0){
        fseek(f,offset * sizeof(char),SEEK_CUR);
        if((f_size - offset) < size){
            text_size = f_size - offset;
        }
    }
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    long nbytes;

    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){//number of chunks
        
        if(feof(f)){
            std::cout << "premature end of file" << "\n";
            break;
        }

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;
        nbytes = fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,nbytes,hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(char** dresult_buf,long text_size){

    int text_chunk = 128 * 1024 * 1024;
    
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    char* result_buf = new char[text_chunk];


    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(char),hipMemcpyDeviceToHost);
        
        for (long i = 0; i < (right_bound-left_bound); i++) {
            std::cout << result_buf[i];
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}


void match_pe(std::string subject_string_filename,long size, long offset,std::string program_) {
    
        
    std::string program = std::string((char*)fun_impala) + program_;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, long, char *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    long text_size;
    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset))==nullptr){
        std::cout << "error reading file" << "\n";
        return;
    }

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void match_pe_pat(std::string subject_string_filename,std::string program_,std::string pattern, int pattern_size,long size,long offset) {
    
        
    std::string program = std::string((char*)fun_impala) + program_;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*,int,const char*, long, char *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }
    
    long text_size;
    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        return;
    }
    
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";

    char* d_pat;
    hipMalloc((void**)&d_pat,pattern_size * sizeof(char));
    hipMemcpy(d_pat,pattern.c_str(),pattern_size*sizeof(char),hipMemcpyHostToDevice);
    
    time.start();

    call(d_pat,pattern_size,dtextptr,text_size,dresult_buf);
    hipDeviceSynchronize();

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(d_pat);
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

template<typename Function>
void match_nope(std::string subject_string_filename,std::string pattern,int pattern_size, int nochunk, Function f,long size,long offset) {
    
    am::timer time;

    long text_size;

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    std::cout << "running ... " << "\n";
    

    char* d_pat;
    hipMalloc((void**)&d_pat,pattern_size);
    hipMemcpy(d_pat,pattern.c_str(),pattern_size,hipMemcpyHostToDevice);
    
    time.start();

    if(nochunk){
        f(d_pat,pattern_size,32,dtextptr,text_size,dresult_buf,512,256,1);
    }else {
        f(d_pat,pattern_size,32,dtextptr,text_size,dresult_buf,512,256,0);
    }
    hipDeviceSynchronize();
    time.stop();

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(d_pat);
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void prefix_f(std::string pattern, int index){

     std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy()-> i32{\n";

    dummy_fun += "  prefix_function(\"" + pattern + "\","
              + std::to_string(index) + ",0,0)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef int (*function) ();
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }
    std::cout << call() << "\n";

}

void match_pe_pointer_multipattern(int p_number,std::vector<std::string> vpatterns, std::string subject_string_filename) {
    
    int* sizes = new int[p_number];
    int len = 0;
    for(int i = 1; i < p_number+1; i++) {
        auto str = std::string(vpatterns[i]);
        sizes[i-1] = str.length();
        len += str.length();    
    }

    char* patterns = new char[len];
    
    int offset = 0;

    for(int i = 0; i < p_number; i++){

        for(int j = 0; j < sizes[i]; j++){
            patterns[offset+j] = vpatterns[i+1][j];
        }
        offset+=sizes[i];    
    } 
        
    std::string dummy_fun;

    //maybe asyncronous read from disk and jit;
    dummy_fun += "extern fn dummy(text : &[u8], text_size : i32, result_buf : &mut[i32]) -> (){\n";

//TODO max_pattern_size
    dummy_fun += "  string_match_pseudoKMP_pointer_multiple( \"" + std::string(patterns) + "\", "
              + "["+ std::to_string(sizes[1]) + "]" + "," + std::to_string(p_number)+",3, text, text_size,result_buf,256,256)}"; //;

    std::string program = std::string((char*)fun_impala) + dummy_fun;

    std::cout << "compiling ... " << std::flush;
    am::timer time;
    // time.start();

    auto key = anydsl_compile(program.c_str(),program.size(),0);
    typedef void (*function) (const char*, int, const int *);
    auto call = reinterpret_cast<function>(anydsl_lookup_function(key,"dummy"));
    
    if (call == nullptr) {
        std::cout << "compiliacion failed\n";
        return;
    } else {
        std::cout << "succesfully compiled\n";
    }

    // time.stop();
    // std::cout << "compilation time " << time.milliseconds() << " ms" << std::endl;
    

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);

    // string[fsize] = 0;
    // int fdin,fdout;
    // if (fdin = open(subject_string_filename.c_str(),O_RDONLY) < 0) {
        // std::cout << "can't open file" << subject_string_filename << "\n";
        // return;
    // }
    // char *subject_string = new char[text_size];
    // read(fdin,(void*)subject_string,text_size);
    
    std::cout << "\n";
    int* result_buf = new int[text_size];
    int* dresult_buf;
    char* dtext;
    //think about data transfer;
    hipMalloc((void**)&dtext, text_size * sizeof(char));
    hipMemcpy((void*)dtext,subject_string,text_size * sizeof(char),hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // hipMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    
    // call(text.c_str(),text_size,result_buf);
    // time.reset();
    std::cout << "running ... " << "\n";
    time.start();
    
    call(dtext,text_size,dresult_buf);

    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(dtext);
    delete[] (result_buf);
}


//TODO template method

//TODO refactor with fun that takes impala program as input

int main(int argc, char** argv) {
    
   
    long size = 0;
    long offset = 0;
    int type = 0;
    
    cxxopts::Options options("as", " - example command line options");

    options.add_options()("p,pattern","pattern to look for",cxxopts::value<std::string>())
                         ("f,filename","filename/devicename to look for in",cxxopts::value<std::string>())
                         ("s,size", "size of data to read",cxxopts::value<long>(size)->default_value("0"))
                         ("o,offset", "offset of data to read",cxxopts::value<long>(offset)->default_value("0"))
                         ("a,algorithm","algorithm to look for with",cxxopts::value<std::string>())
                         ("t,type","type of algorithm: 0 stands for nochunk-based and 1 for chunk-based",cxxopts::value<int>(type));
    // std::string subject_string_filename("data/subject.txt");

    auto result = options.parse(argc, argv);

    if(result.count("algorithm") && result.count("type") && result.count("pattern") && result.count("filename")){
        auto alg_name = result["algorithm"].as<std::string>();
        auto filename = result["filename"].as<std::string>();
        auto patterns = read_pattern(result["pattern"].as<std::string>());
        std::string pattern;
        if(patterns.size() == 1){
            pattern = patterns[0];
        }else{
            std::cout << "bad pattern/failed to read file" << "\n";
            return 0;
        }

        auto pattern_size = pattern.size();
    // pattern.resize(31,'0'); 

        std::string r_naive_spec;

        r_naive_spec += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        r_naive_spec += "  string_match_pseudoKMP(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_nochunk;

        match_pseudoKMP_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk += "  string_match_pseudoKMP_nochunk(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_nochunk_nope;

        match_pseudoKMP_nochunk_nope += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk_nope += "  string_match_pseudoKMP_nochunk_nope(\"" + pattern + "\", "
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;

        std::string match_pseudoKMP_chunk_nope;

        match_pseudoKMP_chunk_nope += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_chunk_nope += "  string_match_pseudoKMP_nope(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;
    
        std::string match_KMP_chunk;
        match_KMP_chunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_KMP_chunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;

        std::string match_KMP_nochunk;
        match_KMP_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_KMP_nochunk += "  match_kmp(\"" + pattern + "\","
              + std::to_string(pattern_size) + ",32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_nochunk;
        match_naive_nochunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_nochunk += "  string_match(\"" + pattern + "\","
              + std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_chunk;
        match_naive_chunk += "extern fn dummy(text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_chunk += "  string_match(\"" + pattern + "\","
              + std::to_string(pattern_size) + ", 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;
    


        std::string match_naive_nochunk_nope;
        match_naive_nochunk_nope += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_nochunk_nope += "  $string_match_nope($pattern,p_size, 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,1)}"; //;

        std::string match_naive_chunk_nope;
        match_naive_chunk_nope += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_naive_chunk_nope += "  $string_match_nope($pattern, p_size, 32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256,0)}"; //;



        std::string match_pseudoKMP_chunk_nope_annotated;

        match_pseudoKMP_chunk_nope_annotated += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_chunk_nope_annotated += "  string_match_pseudoKMP_nope(pattern,p_size,32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;


        std::string match_pseudoKMP_nochunk_nope_annotated;

        match_pseudoKMP_nochunk_nope_annotated += "extern fn dummy(pattern : &[u8],p_size : i32,text : &[u8], text_size : i64, result_buf : &mut[u8]) -> (){\n";

        match_pseudoKMP_nochunk_nope_annotated += "  string_match_pseudoKMP_nochunk_nope(pattern,p_size,32i8 ,text, text_size,result_buf," + std::to_string(block_size) + ",256)}"; //;         

        if (pattern.size() > 128) {
            std::cout << "pattern should be less then or eq 128 bytes\n";
            return 0;
        }
        if(type == 0){
            if(alg_name == "r_naive_spec"){
                match_pe(filename,size,offset,r_naive_spec);
            }else if(alg_name == "kmp"){
                match_pe(filename,size,offset,match_KMP_chunk);
            }else if(alg_name == "cleankmp"){
                // match_nope(filename,pattern,pattern_size,0,match_kmp,size,offset);
            }else if(alg_name == "cleanpe"){
                match_pe(filename,size,offset,match_naive_chunk);
            }
        }else if(type == 1){
            if(alg_name == "r_naive_spec"){
                match_pe(filename,size,offset,match_pseudoKMP_nochunk);
            }else if(alg_name == "kmp"){
                match_pe(filename,size,offset,match_KMP_nochunk);
            }else if(alg_name == "cleankmp"){
                match_nope(filename,pattern,pattern_size,1,string_match_nope,size,offset);
            }else if(alg_name == "cleanpe"){
                match_pe(filename,size,offset,match_naive_nochunk);
            }
        }else {
            std::cout << pattern << " " << pattern.size() << "\n";
            std::cout << "type should be either 1 or 0" << "\n";
        }
    }

    return 0;
}