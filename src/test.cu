
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>

__global__ void match(char* pattern, int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != pattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}


int main(int argc, char** argv) {

    std::string pattern = std::string(argv[1]);

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return 0;
    }
    auto pattern_size = pattern.size();
    pattern.resize(31,'0');
    char* dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size,hipMemcpyHostToDevice); 
    std::cout << pattern_size << "\n";
    std::cout << pattern << "\n";

    std::string text = std::string(argv[2]);
    // std::cin >> text;
    auto text_size = text.length();
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,text.c_str(),text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    // for(int i = 0; i < text_size; i++) {
        // result_buf[i] = -1;
    // }
    // cudaMemset((void*)dresult_buf, -1, text_size*sizeof(int));
    // for (int i = 0; i < text_size; i++) {
        // std::cout << result_buf[i];
    // }
    // std::cout << "\n";
    
    // call(text.c_str(),text_size,result_buf);
    match<<<6,2>>>(dpattern,pattern_size,textptr,text_size,dresult_buf);
    hipDeviceSynchronize();
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    delete[] (result_buf);

    return 0;
}