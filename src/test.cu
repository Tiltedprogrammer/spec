#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include "timer.h"

typedef struct Template{
    char array[32] = {0};
    int size;
}Template;

__global__ void match(char* pattern, int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != pattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void match_multy(char* pattern, int* p_sizes, int p_number, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 0;
        result_buf[t_id] = -1;

        for(int i = 0; i < p_number; i++) {
            matched = 0;
            for(int j = 0; j < p_sizes[i]; j++){
                
                if(text[t_id + j] != pattern[j+p_offset]) {
                    matched = -1;
                }
            }

            p_offset += p_sizes[i]; 
            
            if(matched != -1) {
                result_buf[t_id] = i;
            }
        }             
    }
}

__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

void multipattern_match(int p_number,char** argv_patterns, char* file_name){

    int* sizes = new int[p_number];
    int len = 0;
    for(int i = 1; i < p_number+1; i++) {
        auto str = std::string(argv_patterns[i]);
        sizes[i-1] = str.length();
        len += str.length();    
    }

    char* patterns = new char[len];
    
    int offset = 0;

    for(int i = 0; i < p_number; i++){

        for(int j = 0; j < sizes[i]; j++){
            patterns[offset+j] = argv_patterns[i+1][j];
        }
        offset+=sizes[i];    
    }

    char* dpatterns;
    int* dsizes;
    hipMalloc((void**)&dsizes, (p_number)*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (p_number)*sizeof(int), hipMemcpyHostToDevice); 
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    hipMemcpy((void*)dpatterns, patterns, len*sizeof(char), hipMemcpyHostToDevice);

    std::string subject_string_filename(file_name);

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f;
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	std::cout << "can not oppen file" << subject_string_filename << "\n";
	    return;
    }

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(1024);
    int grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    match_multy<<<grid,block>>>(dpatterns,dsizes,p_number,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    delete[](sizes);
    delete[](patterns);
    delete[](subject_string);
    hipDeviceSynchronize();
    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    // hipFree(dpattern);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete[] (result_buf);  

}


int main(int argc, char** argv) {

    
    // std::string pattern = std::string(argv[1]);

    // if (pattern.size() > 31) {
        // std::cout << "pattern should be less then or eq 31 bytes\n";
        // return 0;
    // }
    // auto pattern_size = pattern.size();
    // pattern.resize(31,'0');
    // Template dpattern_s;
    // dpattern_s.size = pattern_size;
    // for(int i = 0; i < pattern_size; i++) {
        // dpattern_s.array[i] = pattern[i];
    // }
    // char* dpatterns;
    // int* dsizes;
    // hipMalloc((void**)&dsizes, (argc-1)*sizeof(int));
    // hipMemcpy((void*)dsizes, sizes, (argc-1)*sizeof(int), hipMemcpyHostToDevice); 
    // hipMalloc((void**)&dpatterns, len * sizeof(char));
    // hipMemcpy((void*)dpatterns, patterns, len*sizeof(char), hipMemcpyHostToDevice); 
    

    /*std::string subject_string_filename("data/subject.txt");

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f;
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	std::cout << "can not oppen file" << subject_string_filename << "\n";
	return 0;
    }

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(1024);
    int grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    // match_struct<<<grid,block>>>(dpattern_s,textptr,text_size,dresult_buf);
    // match<<<grid,block>>>(dpattern,pattern_size,textptr,text_size,dresult_buf);
    match_multy<<<grid,block>>>(dpatterns,dsizes,argc-1,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    delete[](sizes);
    delete[](patterns);
    delete[](subject_string);
    hipDeviceSynchronize();
    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    // hipFree(dpattern);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] (result_buf);*/

    multipattern_match(argc-1,argv,"data/subject.txt");

    return 0;
}
