#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include "timer.h"

typedef struct Template{

    char array[32] = {0};
    int size;

}Template;

long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

char* read_file(std::string filename,  int &text_size){
    
    text_size = GetFileSize(filename) - 1;//TODO
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return 0;
    }

    
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    for(int i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;
        fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,right_bound-(left_bound),hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(int** dresult_buf,int text_size){

    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    int* result_buf = new int[text_chunk];


    for(int i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(int),hipMemcpyDeviceToHost);
        
        for (int i = 0; i < (right_bound-left_bound); i++) {
            std::cout << result_buf[i];
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}

__global__ void match(char* pattern, int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != pattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void match_shared(char* pattern, int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ char spattern [32];
    if(threadIdx.x < pattern_size) {
        spattern[threadIdx.x] = pattern[threadIdx.x];
    }
    __syncthreads();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != spattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void match_multy(char* pattern, int* p_sizes, int p_number, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 0;
        result_buf[t_id] = -1;

        for(int i = 0; i < p_number; i++) {
            matched = 0;
            for(int j = 0; j < p_sizes[i]; j++){
                
                if(text[t_id + j] != pattern[j+p_offset]) {
                    matched = -1;
                }
            }

            p_offset += p_sizes[i]; 
            
            if(matched != -1) {
                result_buf[t_id] = i;
            }
        }             
    }
}

__global__ void match_chunk_shared(char* pattern, int pattern_size, int chunk_size ,char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ char spattern [32];
    if(threadIdx.x == 0) {
        for(int i = 0; i < pattern_size; i++){
            spattern[i] = pattern[i];
        }
    }
    __syncthreads();
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = -1;
            int matched = 1;
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != spattern[j]) {
                    matched = -1;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}

__global__ void match_chunk(char* pattern, int pattern_size, int chunk_size ,char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = -1;
            int matched = 1;
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != pattern[j]) {
                    matched = -1;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}


__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

void prefix(const char* pattern, int pattern_size, int* prefix_table){
    
    prefix_table[0] = 0;
    
    for (int i = 1; i < pattern_size; ++i) {
		
        int j = prefix_table[i-1];
		
        while (j > 0 && pattern[i] != pattern[j]){
			
            j = prefix_table[j-1];
        
        }
		
        if (pattern[i] == pattern[j])  ++j;
		
        prefix_table[i] = j;
	}
}

__global__ void kmp_chunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, int* result_buf,int chunk){
    
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    int left_bound = t_id * chunk;
    int right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(int i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = -1;
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = 1;
            ams = prefix_table[ams-1];
        }


    }
}

__global__ void kmp_nochunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, int* result_buf,int chunk){
    
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    int ams = 0;

    for(int i = 0; i < pattern_size; i++){
        
        result_buf[t_id] = -1;

        while(ams > 0 && pattern[ams] != text[t_id + i]){
            ams = prefix_table[ams-1];
        }

        if(text[t_id + i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[t_id] = 1;
            ams = prefix_table[ams-1];
        }


    }
}


void multipattern_match(int p_number,char** argv_patterns, char* file_name){

    int* sizes = new int[p_number];
    int len = 0;
    for(int i = 1; i < p_number+1; i++) {
        auto str = std::string(argv_patterns[i]);
        sizes[i-1] = str.length();
        len += str.length();    
    }

    char* patterns = new char[len];
    
    int offset = 0;

    for(int i = 0; i < p_number; i++){

        for(int j = 0; j < sizes[i]; j++){
            patterns[offset+j] = argv_patterns[i+1][j];
        }
        offset+=sizes[i];    
    }

    char* dpatterns;
    int* dsizes;
    hipMalloc((void**)&dsizes, (p_number)*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (p_number)*sizeof(int), hipMemcpyHostToDevice); 
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    hipMemcpy((void*)dpatterns, patterns, len*sizeof(char), hipMemcpyHostToDevice);

    std::string subject_string_filename(file_name);

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f;
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	std::cout << "can not oppen file" << subject_string_filename << "\n";
	    return;
    }

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(1024);
    int grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    match_multy<<<grid,block>>>(dpatterns,dsizes,p_number,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    delete[](sizes);
    delete[](patterns);
    delete[](subject_string);
    hipDeviceSynchronize();
    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    // hipFree(dpattern);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete[] (result_buf);  

}

void match_naive(std::string pattern, std::string subject_string_filename, int nochunk){ //nochunk == 0 => nochunk

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int text_size;
    char* dtextptr = read_file(subject_string_filename,text_size);

    int* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    int chunk = 256;

    dim3 block(1024);
    int grid_size;
    if(nochunk){
        grid_size = (text_size + block.x - 1) / block.x;
    } else{
        grid_size = (((text_size + chunk - 1) / chunk) + block.x - 1) / block.x;
    }
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    if(nochunk){
        match<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
    }else{
        match_chunk<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        }  
    hipDeviceSynchronize();
    time.stop();

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);

}



void match_naive_shared(std::string pattern, std::string subject_string_filename, int nochunk){ //nochunk == 0 => nochunk

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int text_size;
    char* dtextptr = read_file(subject_string_filename,text_size);
    
    int* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    int chunk = 256;

    dim3 block(1024);
    int grid_size;
    if(nochunk){
        grid_size = (text_size + block.x - 1) / block.x;
    } else{
        grid_size = (((text_size + chunk - 1) / chunk) + block.x - 1) / block.x;
    }
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    if(nochunk){
        match_shared<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
    }else{
        match_chunk_shared<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        }  

    hipDeviceSynchronize();
    time.stop();

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
      
}

void match_naive_pointer(std::string pattern, std::string subject_string_filename){ //nochunk == 0 => nochunk

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f;
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << subject_string_filename << "\n";
	    return;
    }

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(1024);
    int grid_size;
    grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    match<<<grid,block>>>(dpattern,pattern_size,textptr,text_size,dresult_buf);
    // match_struct<<<grid,block>>>(dpattern_s,textptr,text_size,dresult_buf);
    // match<<<grid,block>>>(dpattern,pattern_size,textptr,text_size,dresult_buf);
    // match_multy<<<grid,block>>>(dpatterns,dsizes,argc-1,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    // delete[](sizes);
    hipDeviceSynchronize();
    time.stop();

    // delete[](pattern);
    delete[](subject_string);
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    hipFree(dpattern);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete[] (result_buf);   
}

void match_kmp(std::string pattern, std::string subject_string_filename, int nochunk){ //nochunk == 0 => nochunk

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int* prefix_table = new int[pattern_size];
    prefix(pattern.c_str(),pattern_size,prefix_table);
    int* dprefix_table;

    hipMalloc((void**)&dprefix_table, pattern_size * sizeof(int));
    hipMemcpy((void*)dprefix_table,prefix_table,pattern_size * sizeof(int),hipMemcpyHostToDevice); 
    delete[](prefix_table);

    int text_size;//TODO

    char* dtextptr = read_file(subject_string_filename,text_size);
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    //think about data transfer;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    int chunk = 256;

    dim3 block(1024);
    int grid_size;
    if(nochunk){
        grid_size = (text_size + block.x - 1) / block.x;
    } else{
        grid_size = (((text_size + chunk - 1) / chunk) + block.x - 1) / block.x;
    }
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    if(nochunk){
        kmp_nochunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
    }else{
        kmp_chunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
        }  

    hipDeviceSynchronize();
    time.stop();

    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
    hipFree(dprefix_table);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);   
}


__constant__ char c_pattern[32]; //might be as fast as registers, but not in this case =)

__global__ void match_chunk_const(int pattern_size, int chunk_size ,char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = -1;
            int matched = 1;
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != c_pattern[j]) {
                    matched = -1;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = 1;
            }
        }
                             
    }
}

__global__ void match_const(int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != c_pattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

void match_const(std::string pattern, std::string subject_string_filename, int nochunk){
    
    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),(void*)pattern.c_str(),pattern.size()*sizeof(char));

    int text_size;//TODO

    char* dtextptr = read_file(subject_string_filename,text_size);

    //think about data transfer;
    int* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    

    int chunk = 256;

    dim3 block(1024);
    int grid_size;
    if(nochunk){
        grid_size = (text_size + block.x - 1) / block.x;
    } else{
        grid_size = (((text_size + chunk - 1) / chunk) + block.x - 1) / block.x;
    }
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    if(nochunk){
        match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf);
    }else{
        match_chunk_const<<<grid,block>>>(pattern_size,chunk,dtextptr,text_size,dresult_buf);
        }  
    //move results back;

    hipDeviceSynchronize();
    time.stop();

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

int main(int argc, char** argv) {

    
    std::string pattern = std::string(argv[1]);
    std::string subject_string_filename("data/subject.txt");
    auto arg = std::string(argv[2]) + std::string(argv[3]);
    
    if(arg == "naive1"){
    
            match_naive(pattern,subject_string_filename,1);
    
    }else if(arg == "naive0"){
            
            match_naive(pattern,subject_string_filename,0);

    }else if(arg == "kmp1") {
            
            match_kmp(pattern,subject_string_filename,1);
    
    }else if(arg == "kmp0") {
            match_kmp(pattern,subject_string_filename,0);
            

    }else if(arg == "const1") {
            match_const(pattern,subject_string_filename,1);
    }else if(arg == "const0"){
            match_const(pattern,subject_string_filename,0);
    }else if(arg == "naivesh1"){
            match_naive_shared(pattern,subject_string_filename,1);
    }else if(arg == "naivesh0"){
            match_naive_shared(pattern,subject_string_filename,0);
    }else{

    }
    // match_naive(pattern,subject_string_filename,atoi(argv[2]));
    // match_naive_pointer(pattern,subject_string_filename);
    // match_kmp(pattern,subject_string_filename,1);

    return 0;
}
