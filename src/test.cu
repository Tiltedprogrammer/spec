#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <stdlib.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include "timer.h"

typedef struct Template{
    char array[32] = {0};
    int size;
}Template;

__global__ void match(char* pattern, int pattern_size, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern_size; i++) {
            if(text[t_id + i] != pattern[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

long GetFileSize(std::string filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}


int main(int argc, char** argv) {

    std::string pattern = std::string(argv[1]);

    if (pattern.size() > 31) {
        std::cout << "pattern should be less then or eq 31 bytes\n";
        return 0;
    }
    auto pattern_size = pattern.size();
    pattern.resize(31,'0');
    Template dpattern_s;
    dpattern_s.size = pattern_size;
    for(int i = 0; i < pattern_size; i++) {
        dpattern_s.array[i] = pattern[i];
    }
    // char* dpattern;
    // hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    // hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size,hipMemcpyHostToDevice); 
    std::cout << dpattern_s.size << "\n";
    for(int i = 0; i < pattern_size; i++) {
        std::cout << dpattern_s.array[i] << "\n";
    }
    // std::cout << dpatt << "\n";

    std::string subject_string_filename("data/subject.txt");

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f = fopen(subject_string_filename.c_str(), "rb");
    // fseek(f, 0, SEEK_END);
    // long fsize = ftell(f);
    // fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    delete[](subject_string);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(1024);
    int grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    // hipEventRecord(start);
    match_struct<<<grid,block>>>(dpattern_s,textptr,text_size,dresult_buf);
    // match<<<grid,block>>>(dpattern,pattern_size,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    hipDeviceSynchronize();
    time.stop();
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    // hipFree(dpattern);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] (result_buf);

    return 0;
}