#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <fstream>
#include <math.h>
#include "timer.h"
#include "cxxopts.hpp"


#define block_size 1024

typedef struct Template{

    char array[32] = {0};
    int size;

}Template;

__device__ long threadId(){
    
    long blockId = (long)blockIdx.y * (long)gridDim.x + (long)blockIdx.x;
    long threadId = blockId * (long)blockDim.x + (long)threadIdx.x;
    return threadId;

}

long GetFileSize(std::string filename)
{
    // struct stat stat_buf;
    // int rc = stat(filename.c_str(), &stat_buf);
    // return rc == 0 ? stat_buf.st_size : -1;
    int fd = open(filename.c_str(),O_RDONLY);  //;
    long size = lseek(fd, 0, SEEK_END);
    close(fd);
    return size;
}

std::vector<std::string> read_pattern(std::string filename){
    
    std::ifstream file(filename);
    std::vector<std::string> res = std::vector<std::string>();
 
    if (!file) 
    {
        std::cout << "error openning pattern file" << "\n"; 
    // TODO: assign item_name based on line (or if the entire line is 
    // the item name, replace line with item_name in the code above)
    }
    while(!file.eof()){
        std::string str;
        std::getline(file, str);
        res.push_back(str);
    }
    // std::getline(file, str);
    return res;

}

char* read_file(std::string filename,long &text_size,long size = 0, long offset = 0){
    
    long f_size = GetFileSize(filename) - 1;//TODO
    if(f_size == -1){
        std::cout << "bad_size" << "\n";
        return nullptr;
    }
    //read file
    FILE *f;
    if((f = fopen(filename.c_str(), "rb")) == NULL){
	    std::cout << "can not oppen file" << filename << "\n";
	    return nullptr;
    }

    if(size != 0 && size <= f_size){
        text_size = size;
    }else{
        text_size = f_size;
    }

    if(offset != 0){
        fseek(f,offset * sizeof(char),SEEK_CUR);
        if((f_size - offset) < size){
            text_size = f_size - offset;
        }
    }
    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }
    char *subject_string = new char[text_chunk];

    char* dtextptr;
    
    hipMalloc((void**)&dtextptr, text_size * sizeof(char));

    long nbytes;

    for(long i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){//number of chunks
        
        if(feof(f)){
            std::cout << "premature end of file" << "\n";
            break;
        }

        long right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        long left_bound = i * text_chunk;
        nbytes = fread(subject_string,sizeof(char),right_bound-(left_bound),f);
        hipMemcpy((void*)(dtextptr + left_bound),subject_string,nbytes,hipMemcpyHostToDevice);

    }

    delete[](subject_string);
    fclose(f);

    return dtextptr;
}

void write_from_device(char** dresult_buf,int text_size){

    int text_chunk = 128 * 1024 * 1024;
    if(text_size < text_chunk) {
        text_chunk = text_size;
    }

    char* result_buf = new char[text_chunk];


    for(int i = 0; i < (text_size + text_chunk - 1) / text_chunk; i++){ //number of chunks

        int right_bound = (i+1) * text_chunk < text_size ? (i+1) * text_chunk : text_size;
        int left_bound = i * text_chunk;

        hipMemcpy((void*)(result_buf),((*dresult_buf)+left_bound),(right_bound-(left_bound))*sizeof(char),hipMemcpyDeviceToHost);
        
        for (int i = 0; i < (right_bound-left_bound); i++) {
            std::cout << result_buf[i];
        }

    }
    std::cout << "\n";
    delete[] (result_buf);

}

__global__ void match(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {


    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = '0';

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = '1';
            }             
        }
                     

    }
}

__global__ void match_shared(char* pattern, int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x < pattern_size) {
        spattern[threadIdx.x] = pattern[threadIdx.x];
    }
    __syncthreads();

    if(t_id < text_size){
        
        int matched = 1;
        result_buf[t_id] = '0';

        if(t_id < text_size - pattern_size + 1){
            
            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != spattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = '1';
            }             
        }
                     
    }
}

__global__ void match_multy(char* pattern, int* p_sizes, int p_number, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int p_offset = 0;
        int matched = 0;
        result_buf[t_id] = -1;

        for(int i = 0; i < p_number; i++) {
            matched = 0;
            for(int j = 0; j < p_sizes[i]; j++){
                
                if(text[t_id + j] != pattern[j+p_offset]) {
                    matched = -1;
                    break;
                }
            }

            p_offset += p_sizes[i]; 
            
            if(matched != -1) {
                result_buf[t_id] = i;
            }
        }             
    }
}

__global__ void match_chunk_shared(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    __shared__ char spattern [128];
    if(threadIdx.x == 0) {
        for(int i = 0; i < pattern_size; i++){
            spattern[i] = pattern[i];
        }
    }
    __syncthreads();
    int left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (int i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = '0';
            int matched = 1;
            if(i < text_size - left_bound - pattern_size + 1){
                #pragma unroll
                for(int j = 0; j < pattern_size; j++) {

                    if(text[left_bound + i + j] != spattern[j]) {
                        matched = -1;
                        break;
                    }
                }

                if(matched == 1) {
                    result_buf[left_bound + i] = '1';
                }
            }
        }
                             
    }
}

__global__ void match_chunk(char* pattern, int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = '0';
            int matched = 1;
            
            for(int j = 0; j < pattern_size; j++) {

                if(text[left_bound + i + j] != pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = '1';
            }
        }
                             
    }
}


__global__ void match_struct(Template pattern, char* text, int text_size, int* result_buf) {

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = -1;

        for(int i = 0; i < pattern.size; i++) {
            if(text[t_id + i] != pattern.array[i]) {
                matched = -1;
            }
        }
        if(matched == 1) {
            result_buf[t_id] = 1;
        }             
                     

    }
}

void prefix(const char* pattern, int pattern_size, int* prefix_table){
    
    prefix_table[0] = 0;
    
    for (int i = 1; i < pattern_size; ++i) {
		
        int j = prefix_table[i-1];
		
        while (j > 0 && pattern[i] != pattern[j]){
			
            j = prefix_table[j-1];
        
        }
		
        if (pattern[i] == pattern[j])  ++j;
		
        prefix_table[i] = j;
	}
}

__global__ void kmp_chunk(int* prefix_table, char* pattern,int pattern_size,char* text, long text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = '0';
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = '1';
            ams = prefix_table[ams-1];
        }


    }
}


__global__ void kmp_nochunk(int* prefix_table, char* pattern,int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = '0';
        }

        while(ams > 0 && pattern[ams] != text[i]){
            ams = prefix_table[ams-1];
        }

        if(text[i] == pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = '1';
            ams = prefix_table[ams-1];
        }


    }
}


void multipattern_match(int p_number,std::vector<std::string> vpatterns, char* file_name){

    int* sizes = new int[vpatterns.size()];
    int len = 0;
    for(int i = 0; i < vpatterns.size(); i++) {
        sizes[i] = vpatterns[i].length();
        len += sizes[i];    
    }

    char* patterns = new char[len];
    
    int offset = 0;

    char* dpatterns;
    int* dsizes;
    
    hipMalloc((void**)&dsizes, (vpatterns.size())*sizeof(int));
    hipMemcpy((void*)dsizes, sizes, (vpatterns.size())*sizeof(int), hipMemcpyHostToDevice); 
    
    hipMalloc((void**)&dpatterns, len * sizeof(char));
    
    for(int i = 0; i < vpatterns.size(); i++){
        hipMemcpy((void*)(dpatterns + offset*sizeof(char)),vpatterns[i].c_str(),vpatterns[i].size(),hipMemcpyHostToDevice);
        offset+=sizes[i];
    }

    std::string subject_string_filename(file_name);

    auto text_size = GetFileSize(subject_string_filename) - 1;//TODO
    
    //read file
    FILE *f;
    if((f = fopen(subject_string_filename.c_str(), "rb")) == NULL){
	std::cout << "can not oppen file" << subject_string_filename << "\n";
	    return;
    }

    char *subject_string = new char[text_size];
    fread(subject_string, 1, text_size, f);
    fclose(f);
    // std::cin >> text;
    int* result_buf = new int[text_size];
    int* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    char* textptr;
    //think about data transfer;
    hipMalloc((void**)&textptr, text_size * sizeof(char));
    hipMemcpy((void*)textptr,subject_string,text_size,hipMemcpyHostToDevice);
    hipMalloc((void**)&dresult_buf, text_size * sizeof(int));
    
    dim3 block(block_size);
    int grid_size = (text_size + block.x - 1) / block.x;
    dim3 grid(grid_size);

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    
    std::cout << "running ..." << "\n";

    am::timer time;
    time.start();
    match_multy<<<grid,block>>>(dpatterns,dsizes,p_number,textptr,text_size,dresult_buf);
    // hipEventRecord(stop);
    hipDeviceSynchronize();
    time.stop();
    delete[](sizes);
    delete[](patterns);
    delete[](subject_string);
    
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    hipMemcpy((void*)result_buf,dresult_buf,text_size*sizeof(int),hipMemcpyDeviceToHost);
    // hipEventSynchronize(stop);
    hipDeviceSynchronize();

    float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);

    // std::cout << "running time " << milliseconds << "ms" << "\n";



    for (int i = 0; i < text_size; i++) {
        std::cout << result_buf[i];
    }
    std::cout << "\n";
    
    hipFree(dresult_buf);
    hipFree(textptr);
    // hipFree(dpattern);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
    delete[] (result_buf);  

}

void match_naive(std::string pattern, std::string subject_string_filename, int nochunk, int size, int offset){ //nochunk == 0 => nochunk

    if (pattern.size() > 128) {
        std::cout << "pattern should be less then or eq 128 bytes\n";
        return;
    }

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    long text_size;

    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;
    
    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);

}



void match_naive_shared(std::string pattern, std::string subject_string_filename, int nochunk,int size, int offset){ //nochunk == 0 => nochunk

    if (pattern.size() > 128) {
        std::cout << "pattern should be less then or eq 128 bytes\n";
        return;
    }

    char* dtextptr;
    long text_size;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }

    auto pattern_size = pattern.size();
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_shared<<<grid,block>>>(dpattern,pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk_shared<<<grid,block>>>(dpattern,pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;    

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
      
}


void match_kmp(std::string pattern, std::string subject_string_filename, int nochunk,int size,int offset){ //nochunk == 0 => nochunk

    auto pattern_size = pattern.size();
    
    char* dtextptr;
    long text_size;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    
    char *dpattern;
    hipMalloc((void**)&dpattern, pattern_size * sizeof(char));
    hipMemcpy((void*)dpattern,pattern.c_str(),pattern_size * sizeof(char),hipMemcpyHostToDevice); 

    int* prefix_table = new int[pattern_size];
    prefix(pattern.c_str(),pattern_size,prefix_table);
    int* dprefix_table;

    hipMalloc((void**)&dprefix_table, pattern_size * sizeof(int));
    hipMemcpy((void*)dprefix_table,prefix_table,pattern_size * sizeof(int),hipMemcpyHostToDevice); 
    delete[](prefix_table);

    char* dresult_buf;
    // std::cout << "text length : " << text_size << "\n";
    //think about data transfer;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    
    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_nochunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_chunk<<<grid,block>>>(dprefix_table,dpattern,pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    }
    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
    hipFree(dpattern);
    hipFree(dprefix_table); 
}


__constant__ char c_pattern[128]; //might be as fast as registers, but not in this case =)

__global__ void match_chunk_const(int pattern_size, int chunk_size ,char* text, long text_size, char* result_buf) {

    long t_id = threadId();
    long left_bound = t_id * chunk_size;
    // int right_bound = left_bound + chunk_size + pattern_size - 1 >= text_size ? text_size  
                                                                        //  : left_bound + chunk_size + pattern_size - 1;

    if(left_bound < text_size){
        for (long i = 0; i < chunk_size && left_bound + i < text_size; i++) {

            result_buf[left_bound + i] = '0';
            int matched = 1;

            for(int j = 0; j < pattern_size; j++) {
                
                if(text[left_bound + i + j] != c_pattern[j]) {
                    matched = -1;
                    break;
                }
            }

            if(matched == 1) {
                result_buf[left_bound + i] = '1';
            }
        }
                             
    }
}

__global__ void match_const(int pattern_size, char* text, long text_size, char* result_buf) {

    long t_id = threadId();

    if(t_id < text_size){
        int matched = 1;
        result_buf[t_id] = '0';
        if(t_id < text_size - pattern_size + 1){

            for(int i = 0; i < pattern_size; i++) {
                if(text[t_id + i] != c_pattern[i]) {
                    matched = -1;
                    return;
                }
            }
            if(matched == 1) {
                result_buf[t_id] = '1';
            }             
        }
                     

    }
}

__constant__ int c_prefix[128];

__global__ void kmp_chunk_const(int pattern_size,char* text, int text_size, char* result_buf,int chunk){
    
    long t_id = threadId();

    long left_bound = t_id * chunk;
    long right_bound = left_bound + chunk + pattern_size - 1 < text_size ? left_bound + chunk + pattern_size - 1 : text_size;

    int ams = 0;

    for(long i = left_bound; i < right_bound; i++){
        
        if (i < left_bound + chunk) {
            result_buf[i] = '0';
        }

        while(ams > 0 && c_pattern[ams] != text[i]){
            ams = c_prefix[ams-1];
        }

        if(text[i] == c_pattern[ams]){
            ams += 1;
        }
        if(ams == pattern_size) {
            result_buf[i-pattern_size + 1] = '1';
            ams = c_prefix[ams-1];
        }


    }
}

void match_naive_const(std::string pattern, std::string subject_string_filename, int nochunk,int size, int offset){
    
    if (pattern.size() > 128) {
        std::cout << "pattern should be less then or eq 128 bytes\n";
        return;
    }

    auto pattern_size = pattern.size(); // <= 128
    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),(void*)pattern.c_str(),pattern.size()*sizeof(char));

    // int* prefix_table = new int[pattern_size];
    // prefix(pattern.c_str(),pattern_size,prefix_table);
    // hipMemcpyToSymbol(HIP_SYMBOL(c_prefix),(void*)prefix_table,pattern.size()*sizeof(int));
    // delete[](prefix_table);

    long text_size;//TODO

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    //think about data transfer;
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    

    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_chunk_const<<<grid,block>>>(pattern_size,chunk,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

void match_kmp_const(std::string pattern, std::string subject_string_filename, int nochunk,int size, int offset){
    
    if (pattern.size() > 128) {
        std::cout << "pattern should be less then or eq 128 bytes\n";
        return;
    }

    auto pattern_size = pattern.size(); // <= 128
    hipMemcpyToSymbol(HIP_SYMBOL(c_pattern),(void*)pattern.c_str(),pattern.size()*sizeof(char));

    int* prefix_table = new int[pattern_size];
    prefix(pattern.c_str(),pattern_size,prefix_table);
    hipMemcpyToSymbol(HIP_SYMBOL(c_prefix),(void*)prefix_table,pattern.size()*sizeof(int));
    delete[](prefix_table);

    long text_size;//TODO

    char* dtextptr;
    if((dtextptr = read_file(subject_string_filename,text_size,size,offset)) == nullptr){
        std::cout << "error opening file" << "\n";
        return;
    }
    //think about data transfer;
    char* dresult_buf;
    hipMalloc((void**)&dresult_buf, text_size * sizeof(char));
    

    int chunk = 256;

    dim3 block(block_size);
    long grid_size;
    long gsqrt;
    am::timer time;
    std::cout << "running ..." << "\n";
    time.start();
    
    if(nochunk){
        grid_size = (text_size + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        match_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf);
        hipDeviceSynchronize();
        time.stop();
    } else{
        grid_size = (((text_size + (long)chunk - (long)1) / (long)chunk) + (long)block.x - 1L) / (long)block.x;
        gsqrt = (int)sqrt(grid_size) + 1;
        dim3 grid(gsqrt,gsqrt);
        kmp_chunk_const<<<grid,block>>>(pattern_size,dtextptr,text_size,dresult_buf,chunk);
        hipDeviceSynchronize();
        time.stop();
    }

    std::cout << "running time " << time.milliseconds() << " ms" << std::endl;

    write_from_device(&dresult_buf,text_size);
    
    hipFree(dresult_buf);
    hipFree(dtextptr);
}

int main(int argc, char** argv) {

    
    int size = 0;
    int offset = 0;
    int type = 0;
    
    cxxopts::Options options("as", " - example command line options");

    options.add_options()("p,pattern","pattern to look for",cxxopts::value<std::string>())
                         ("f,filename","filename/devicename to look for in",cxxopts::value<std::string>())
                         ("s,size", "size of data to read",cxxopts::value<int>(size)->default_value("0"))
                         ("o,offset", "offset of data to read",cxxopts::value<int>(offset)->default_value("0"))
                         ("a,algorithm","algorithm to look for with",cxxopts::value<std::string>())
                         ("t,type","type of algorithm: 0 stands for nochunk-based and 1 for chunk-based",cxxopts::value<int>(type));
    // std::string subject_string_filename("data/subject.txt");

    auto result = options.parse(argc, argv);

    if(result.count("algorithm") && result.count("type") && result.count("pattern") && result.count("filename")){
        auto alg_name = result["algorithm"].as<std::string>();
        auto filename = result["filename"].as<std::string>();
        auto patterns = read_pattern(result["pattern"].as<std::string>());
        std::string pattern; 
        if(patterns.size() == 1){
            pattern = patterns[0];
        } //if contains \x00 --- considered empty
        if(type == 1 || type == 0){
            if(alg_name == "naive"){
                match_naive(pattern,filename,type,size,offset);
            }else if(alg_name == "naivec"){
                match_naive_const(pattern,filename,type,size,offset);
            }else if(alg_name == "naivesh"){
                match_naive_shared(pattern,filename,type,size,offset);
            }else if(alg_name == "kmp"){
                match_kmp_const(pattern,filename,0,size,offset);
            }
        }else{
            std::cout << "type should be either 1 or 0" << "\n";
        }
    }else{
        std::cout << "algorithm name shoud be specified with --algorithm=name and type with --type=type" << "\n";
    }    

    return 0;
}
